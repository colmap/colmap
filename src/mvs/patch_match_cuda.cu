#include "hip/hip_runtime.h"
// Copyright (c) 2022, ETH Zurich and UNC Chapel Hill.
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//
//     * Neither the name of ETH Zurich and UNC Chapel Hill nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDERS OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// Author: Johannes L. Schoenberger (jsch-at-demuc-dot-de)

#define _USE_MATH_DEFINES

#include "mvs/patch_match_cuda.h"

#include <algorithm>
#include <cfloat>
#include <cmath>
#include <cstdint>
#include <sstream>

#include "util/hip/hip_runtime.h"
#include "util/cudacc.h"
#include "util/logging.h"

// The number of threads per Cuda thread. Warning: Do not change this value,
// since the templated window sizes rely on this value.
#define THREADS_PER_BLOCK 32

// We must not include "util/math.h" to avoid any Eigen includes here,
// since Visual Studio cannot compile some of the Eigen/Boost expressions.
#ifndef DEG2RAD
#define DEG2RAD(deg) deg * 0.0174532925199432
#endif

namespace colmap {
namespace mvs {

texture<uint8_t, hipTextureType2D, hipReadModeNormalizedFloat>
    ref_image_texture;
texture<uint8_t, hipTextureType2DLayered, hipReadModeNormalizedFloat>
    src_images_texture;
texture<float, hipTextureType2DLayered, hipReadModeElementType>
    src_depth_maps_texture;
texture<float, hipTextureType2D, hipReadModeElementType> poses_texture;

// Calibration of reference image as {fx, cx, fy, cy}.
__constant__ float ref_K[4];
// Calibration of reference image as {1/fx, -cx/fx, 1/fy, -cy/fy}.
__constant__ float ref_inv_K[4];

__device__ inline void Mat33DotVec3(const float mat[9], const float vec[3],
                                    float result[3]) {
  result[0] = mat[0] * vec[0] + mat[1] * vec[1] + mat[2] * vec[2];
  result[1] = mat[3] * vec[0] + mat[4] * vec[1] + mat[5] * vec[2];
  result[2] = mat[6] * vec[0] + mat[7] * vec[1] + mat[8] * vec[2];
}

__device__ inline void Mat33DotVec3Homogeneous(const float mat[9],
                                               const float vec[2],
                                               float result[2]) {
  const float inv_z = 1.0f / (mat[6] * vec[0] + mat[7] * vec[1] + mat[8]);
  result[0] = inv_z * (mat[0] * vec[0] + mat[1] * vec[1] + mat[2]);
  result[1] = inv_z * (mat[3] * vec[0] + mat[4] * vec[1] + mat[5]);
}

__device__ inline float DotProduct3(const float vec1[3], const float vec2[3]) {
  return vec1[0] * vec2[0] + vec1[1] * vec2[1] + vec1[2] * vec2[2];
}

__device__ inline float GenerateRandomDepth(const float depth_min,
                                            const float depth_max,
                                            hiprandState* rand_state) {
  return hiprand_uniform(rand_state) * (depth_max - depth_min) + depth_min;
}

__device__ inline void GenerateRandomNormal(const int row, const int col,
                                            hiprandState* rand_state,
                                            float normal[3]) {
  // Unbiased sampling of normal, according to George Marsaglia, "Choosing a
  // Point from the Surface of a Sphere", 1972.
  float v1 = 0.0f;
  float v2 = 0.0f;
  float s = 2.0f;
  while (s >= 1.0f) {
    v1 = 2.0f * hiprand_uniform(rand_state) - 1.0f;
    v2 = 2.0f * hiprand_uniform(rand_state) - 1.0f;
    s = v1 * v1 + v2 * v2;
  }

  const float s_norm = sqrt(1.0f - s);
  normal[0] = 2.0f * v1 * s_norm;
  normal[1] = 2.0f * v2 * s_norm;
  normal[2] = 1.0f - 2.0f * s;

  // Make sure normal is looking away from camera.
  const float view_ray[3] = {ref_inv_K[0] * col + ref_inv_K[1],
                             ref_inv_K[2] * row + ref_inv_K[3], 1.0f};
  if (DotProduct3(normal, view_ray) > 0) {
    normal[0] = -normal[0];
    normal[1] = -normal[1];
    normal[2] = -normal[2];
  }
}

__device__ inline float PerturbDepth(const float perturbation,
                                     const float depth,
                                     hiprandState* rand_state) {
  const float depth_min = (1.0f - perturbation) * depth;
  const float depth_max = (1.0f + perturbation) * depth;
  return GenerateRandomDepth(depth_min, depth_max, rand_state);
}

__device__ inline void PerturbNormal(const int row, const int col,
                                     const float perturbation,
                                     const float normal[3],
                                     hiprandState* rand_state,
                                     float perturbed_normal[3],
                                     const int num_trials = 0) {
  // Perturbation rotation angles.
  const float a1 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
  const float a2 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;
  const float a3 = (hiprand_uniform(rand_state) - 0.5f) * perturbation;

  const float sin_a1 = sin(a1);
  const float sin_a2 = sin(a2);
  const float sin_a3 = sin(a3);
  const float cos_a1 = cos(a1);
  const float cos_a2 = cos(a2);
  const float cos_a3 = cos(a3);

  // R = Rx * Ry * Rz
  float R[9];
  R[0] = cos_a2 * cos_a3;
  R[1] = -cos_a2 * sin_a3;
  R[2] = sin_a2;
  R[3] = cos_a1 * sin_a3 + cos_a3 * sin_a1 * sin_a2;
  R[4] = cos_a1 * cos_a3 - sin_a1 * sin_a2 * sin_a3;
  R[5] = -cos_a2 * sin_a1;
  R[6] = sin_a1 * sin_a3 - cos_a1 * cos_a3 * sin_a2;
  R[7] = cos_a3 * sin_a1 + cos_a1 * sin_a2 * sin_a3;
  R[8] = cos_a1 * cos_a2;

  // Perturb the normal vector.
  Mat33DotVec3(R, normal, perturbed_normal);

  // Make sure the perturbed normal is still looking in the same direction as
  // the viewing direction, otherwise try again but with smaller perturbation.
  const float view_ray[3] = {ref_inv_K[0] * col + ref_inv_K[1],
                             ref_inv_K[2] * row + ref_inv_K[3], 1.0f};
  if (DotProduct3(perturbed_normal, view_ray) >= 0.0f) {
    const int kMaxNumTrials = 3;
    if (num_trials < kMaxNumTrials) {
      PerturbNormal(row, col, 0.5f * perturbation, normal, rand_state,
                    perturbed_normal, num_trials + 1);
      return;
    } else {
      perturbed_normal[0] = normal[0];
      perturbed_normal[1] = normal[1];
      perturbed_normal[2] = normal[2];
      return;
    }
  }

  // Make sure normal has unit norm.
  const float inv_norm = rsqrt(DotProduct3(perturbed_normal, perturbed_normal));
  perturbed_normal[0] *= inv_norm;
  perturbed_normal[1] *= inv_norm;
  perturbed_normal[2] *= inv_norm;
}

__device__ inline void ComputePointAtDepth(const float row, const float col,
                                           const float depth, float point[3]) {
  point[0] = depth * (ref_inv_K[0] * col + ref_inv_K[1]);
  point[1] = depth * (ref_inv_K[2] * row + ref_inv_K[3]);
  point[2] = depth;
}

// Transfer depth on plane from viewing ray at row1 to row2. The returned
// depth is the intersection of the viewing ray through row2 with the plane
// at row1 defined by the given depth and normal.
__device__ inline float PropagateDepth(const float depth1,
                                       const float normal1[3], const float row1,
                                       const float row2) {
  // Point along first viewing ray.
  const float x1 = depth1 * (ref_inv_K[2] * row1 + ref_inv_K[3]);
  const float y1 = depth1;
  // Point on plane defined by point along first viewing ray and plane normal1.
  const float x2 = x1 + normal1[2];
  const float y2 = y1 - normal1[1];

  // Origin of second viewing ray.
  // const float x3 = 0.0f;
  // const float y3 = 0.0f;
  // Point on second viewing ray.
  const float x4 = ref_inv_K[2] * row2 + ref_inv_K[3];
  // const float y4 = 1.0f;

  // Intersection of the lines ((x1, y1), (x2, y2)) and ((x3, y3), (x4, y4)).
  const float denom = x2 - x1 + x4 * (y1 - y2);
  constexpr float kEps = 1e-5f;
  if (abs(denom) < kEps) {
    return depth1;
  }
  const float nom = y1 * x2 - x1 * y2;
  return nom / denom;
}

// First, compute triangulation angle between reference and source image for 3D
// point. Second, compute incident angle between viewing direction of source
// image and normal direction of 3D point. Both angles are cosine distances.
__device__ inline void ComputeViewingAngles(const float point[3],
                                            const float normal[3],
                                            const int image_idx,
                                            float* cos_triangulation_angle,
                                            float* cos_incident_angle) {
  *cos_triangulation_angle = 0.0f;
  *cos_incident_angle = 0.0f;

  // Projection center of source image.
  float C[3];
  for (int i = 0; i < 3; ++i) {
    C[i] = tex2D(poses_texture, i + 16, image_idx);
  }

  // Ray from point to camera.
  const float SX[3] = {C[0] - point[0], C[1] - point[1], C[2] - point[2]};

  // Length of ray from reference image to point.
  const float RX_inv_norm = rsqrt(DotProduct3(point, point));

  // Length of ray from source image to point.
  const float SX_inv_norm = rsqrt(DotProduct3(SX, SX));

  *cos_incident_angle = DotProduct3(SX, normal) * SX_inv_norm;
  *cos_triangulation_angle = DotProduct3(SX, point) * RX_inv_norm * SX_inv_norm;
}

__device__ inline void ComposeHomography(const int image_idx, const int row,
                                         const int col, const float depth,
                                         const float normal[3], float H[9]) {
  // Calibration of source image.
  float K[4];
  for (int i = 0; i < 4; ++i) {
    K[i] = tex2D(poses_texture, i, image_idx);
  }

  // Relative rotation between reference and source image.
  float R[9];
  for (int i = 0; i < 9; ++i) {
    R[i] = tex2D(poses_texture, i + 4, image_idx);
  }

  // Relative translation between reference and source image.
  float T[3];
  for (int i = 0; i < 3; ++i) {
    T[i] = tex2D(poses_texture, i + 13, image_idx);
  }

  // Distance to the plane.
  const float dist =
      depth * (normal[0] * (ref_inv_K[0] * col + ref_inv_K[1]) +
               normal[1] * (ref_inv_K[2] * row + ref_inv_K[3]) + normal[2]);
  const float inv_dist = 1.0f / dist;

  const float inv_dist_N0 = inv_dist * normal[0];
  const float inv_dist_N1 = inv_dist * normal[1];
  const float inv_dist_N2 = inv_dist * normal[2];

  // Homography as H = K * (R - T * n' / d) * Kref^-1.
  H[0] = ref_inv_K[0] * (K[0] * (R[0] + inv_dist_N0 * T[0]) +
                         K[1] * (R[6] + inv_dist_N0 * T[2]));
  H[1] = ref_inv_K[2] * (K[0] * (R[1] + inv_dist_N1 * T[0]) +
                         K[1] * (R[7] + inv_dist_N1 * T[2]));
  H[2] = K[0] * (R[2] + inv_dist_N2 * T[0]) +
         K[1] * (R[8] + inv_dist_N2 * T[2]) +
         ref_inv_K[1] * (K[0] * (R[0] + inv_dist_N0 * T[0]) +
                         K[1] * (R[6] + inv_dist_N0 * T[2])) +
         ref_inv_K[3] * (K[0] * (R[1] + inv_dist_N1 * T[0]) +
                         K[1] * (R[7] + inv_dist_N1 * T[2]));
  H[3] = ref_inv_K[0] * (K[2] * (R[3] + inv_dist_N0 * T[1]) +
                         K[3] * (R[6] + inv_dist_N0 * T[2]));
  H[4] = ref_inv_K[2] * (K[2] * (R[4] + inv_dist_N1 * T[1]) +
                         K[3] * (R[7] + inv_dist_N1 * T[2]));
  H[5] = K[2] * (R[5] + inv_dist_N2 * T[1]) +
         K[3] * (R[8] + inv_dist_N2 * T[2]) +
         ref_inv_K[1] * (K[2] * (R[3] + inv_dist_N0 * T[1]) +
                         K[3] * (R[6] + inv_dist_N0 * T[2])) +
         ref_inv_K[3] * (K[2] * (R[4] + inv_dist_N1 * T[1]) +
                         K[3] * (R[7] + inv_dist_N1 * T[2]));
  H[6] = ref_inv_K[0] * (R[6] + inv_dist_N0 * T[2]);
  H[7] = ref_inv_K[2] * (R[7] + inv_dist_N1 * T[2]);
  H[8] = R[8] + ref_inv_K[1] * (R[6] + inv_dist_N0 * T[2]) +
         ref_inv_K[3] * (R[7] + inv_dist_N1 * T[2]) + inv_dist_N2 * T[2];
}

// Each thread in the current warp / thread block reads in 3 columns of the
// reference image. The shared memory holds 3 * THREADS_PER_BLOCK columns and
// kWindowSize rows of the reference image. Each thread copies every
// THREADS_PER_BLOCK-th column from global to shared memory offset by its ID.
// For example, if THREADS_PER_BLOCK = 32, then thread 0 reads columns 0, 32, 64
// and thread 1 columns 1, 33, 65. When computing the photoconsistency, which is
// shared among each thread block, each thread can then read the reference image
// colors from shared memory. Note that this limits the window radius to a
// maximum of THREADS_PER_BLOCK.
template <int kWindowSize>
struct LocalRefImage {
  const static int kWindowRadius = kWindowSize / 2;
  const static int kThreadBlockRadius = 1;
  const static int kThreadBlockSize = 2 * kThreadBlockRadius + 1;
  const static int kNumRows = kWindowSize;
  const static int kNumColumns = kThreadBlockSize * THREADS_PER_BLOCK;
  const static int kDataSize = kNumRows * kNumColumns;

  float* data = nullptr;

  __device__ inline void Read(const int row) {
    // For the first row, read the entire block into shared memory. For all
    // consecutive rows, it is only necessary to shift the rows in shared memory
    // up by one element and then read in a new row at the bottom of the shared
    // memory. Note that this assumes that the calling loop starts with the
    // first row and then consecutively reads in the next row.

    const int thread_id = threadIdx.x;
    const int thread_block_first_id = blockDim.x * blockIdx.x;

    const int local_col_start = thread_id;
    const int global_col_start = thread_block_first_id -
                                 kThreadBlockRadius * THREADS_PER_BLOCK +
                                 thread_id;

    if (row == 0) {
      int global_row = row - kWindowRadius;
      for (int local_row = 0; local_row < kNumRows; ++local_row, ++global_row) {
        int local_col = local_col_start;
        int global_col = global_col_start;
#pragma unroll
        for (int block = 0; block < kThreadBlockSize; ++block) {
          data[local_row * kNumColumns + local_col] =
              tex2D(ref_image_texture, global_col, global_row);
          local_col += THREADS_PER_BLOCK;
          global_col += THREADS_PER_BLOCK;
        }
      }
    } else {
      // Move rows in shared memory up by one row.
      for (int local_row = 1; local_row < kNumRows; ++local_row) {
        int local_col = local_col_start;
#pragma unroll
        for (int block = 0; block < kThreadBlockSize; ++block) {
          data[(local_row - 1) * kNumColumns + local_col] =
              data[local_row * kNumColumns + local_col];
          local_col += THREADS_PER_BLOCK;
        }
      }

      // Read next row into the last row of shared memory.
      const int local_row = kNumRows - 1;
      const int global_row = row + kWindowRadius;
      int local_col = local_col_start;
      int global_col = global_col_start;
#pragma unroll
      for (int block = 0; block < kThreadBlockSize; ++block) {
        data[local_row * kNumColumns + local_col] =
            tex2D(ref_image_texture, global_col, global_row);
        local_col += THREADS_PER_BLOCK;
        global_col += THREADS_PER_BLOCK;
      }
    }
  }
};

// The return values is 1 - NCC, so the range is [0, 2], the smaller the
// value, the better the color consistency.
template <int kWindowSize, int kWindowStep>
struct PhotoConsistencyCostComputer {
  const static int kWindowRadius = kWindowSize / 2;

  __device__ PhotoConsistencyCostComputer(const float sigma_spatial,
                                          const float sigma_color)
      : bilateral_weight_computer_(sigma_spatial, sigma_color) {}

  // Maximum photo consistency cost as 1 - min(NCC).
  const float kMaxCost = 2.0f;

  // Thread warp local reference image data around current patch.
  typedef LocalRefImage<kWindowSize> LocalRefImageType;
  LocalRefImageType local_ref_image;

  // Precomputed sum of raw and squared image intensities.
  float local_ref_sum = 0.0f;
  float local_ref_squared_sum = 0.0f;

  // Index of source image.
  int src_image_idx = -1;

  // Center position of patch in reference image.
  int row = -1;
  int col = -1;

  // Depth and normal for which to warp patch.
  float depth = 0.0f;
  const float* normal = nullptr;

  __device__ inline void Read(const int row) {
    local_ref_image.Read(row);
    __syncthreads();
  }

  __device__ inline float Compute() const {
    float tform[9];
    ComposeHomography(src_image_idx, row, col, depth, normal, tform);

    float tform_step[8];
    for (int i = 0; i < 8; ++i) {
      tform_step[i] = kWindowStep * tform[i];
    }

    const int thread_id = threadIdx.x;
    const int row_start = row - kWindowRadius;
    const int col_start = col - kWindowRadius;

    float col_src = tform[0] * col_start + tform[1] * row_start + tform[2];
    float row_src = tform[3] * col_start + tform[4] * row_start + tform[5];
    float z = tform[6] * col_start + tform[7] * row_start + tform[8];
    float base_col_src = col_src;
    float base_row_src = row_src;
    float base_z = z;

    int ref_image_idx = THREADS_PER_BLOCK - kWindowRadius + thread_id;
    int ref_image_base_idx = ref_image_idx;

    const float ref_center_color =
        local_ref_image
            .data[ref_image_idx + kWindowRadius * 3 * THREADS_PER_BLOCK +
                  kWindowRadius];
    const float ref_color_sum = local_ref_sum;
    const float ref_color_squared_sum = local_ref_squared_sum;
    float src_color_sum = 0.0f;
    float src_color_squared_sum = 0.0f;
    float src_ref_color_sum = 0.0f;
    float bilateral_weight_sum = 0.0f;

    for (int row = -kWindowRadius; row <= kWindowRadius; row += kWindowStep) {
      for (int col = -kWindowRadius; col <= kWindowRadius; col += kWindowStep) {
        const float inv_z = 1.0f / z;
        const float norm_col_src = inv_z * col_src + 0.5f;
        const float norm_row_src = inv_z * row_src + 0.5f;
        const float ref_color = local_ref_image.data[ref_image_idx];
        const float src_color = tex2DLayered(src_images_texture, norm_col_src,
                                             norm_row_src, src_image_idx);

        const float bilateral_weight = bilateral_weight_computer_.Compute(
            row, col, ref_center_color, ref_color);

        const float bilateral_weight_src = bilateral_weight * src_color;

        src_color_sum += bilateral_weight_src;
        src_color_squared_sum += bilateral_weight_src * src_color;
        src_ref_color_sum += bilateral_weight_src * ref_color;
        bilateral_weight_sum += bilateral_weight;

        ref_image_idx += kWindowStep;

        // Accumulate warped source coordinates per row to reduce numerical
        // errors. Note that this is necessary since coordinates usually are in
        // the order of 1000s as opposed to the color values which are
        // normalized to the range [0, 1].
        col_src += tform_step[0];
        row_src += tform_step[3];
        z += tform_step[6];
      }

      ref_image_base_idx += kWindowStep * 3 * THREADS_PER_BLOCK;
      ref_image_idx = ref_image_base_idx;

      base_col_src += tform_step[1];
      base_row_src += tform_step[4];
      base_z += tform_step[7];

      col_src = base_col_src;
      row_src = base_row_src;
      z = base_z;
    }

    const float inv_bilateral_weight_sum = 1.0f / bilateral_weight_sum;
    src_color_sum *= inv_bilateral_weight_sum;
    src_color_squared_sum *= inv_bilateral_weight_sum;
    src_ref_color_sum *= inv_bilateral_weight_sum;

    const float ref_color_var =
        ref_color_squared_sum - ref_color_sum * ref_color_sum;
    const float src_color_var =
        src_color_squared_sum - src_color_sum * src_color_sum;

    // Based on Jensen's Inequality for convex functions, the variance
    // should always be larger than 0. Do not make this threshold smaller.
    constexpr float kMinVar = 1e-5f;
    if (ref_color_var < kMinVar || src_color_var < kMinVar) {
      return kMaxCost;
    } else {
      const float src_ref_color_covar =
          src_ref_color_sum - ref_color_sum * src_color_sum;
      const float src_ref_color_var = sqrt(ref_color_var * src_color_var);
      return max(0.0f,
                 min(kMaxCost, 1.0f - src_ref_color_covar / src_ref_color_var));
    }
  }

 private:
  const BilateralWeightComputer bilateral_weight_computer_;
};

__device__ inline float ComputeGeomConsistencyCost(const float row,
                                                   const float col,
                                                   const float depth,
                                                   const int image_idx,
                                                   const float max_cost) {
  // Extract projection matrices for source image.
  float P[12];
  for (int i = 0; i < 12; ++i) {
    P[i] = tex2D(poses_texture, i + 19, image_idx);
  }
  float inv_P[12];
  for (int i = 0; i < 12; ++i) {
    inv_P[i] = tex2D(poses_texture, i + 31, image_idx);
  }

  // Project point in reference image to world.
  float forward_point[3];
  ComputePointAtDepth(row, col, depth, forward_point);

  // Project world point to source image.
  const float inv_forward_z =
      1.0f / (P[8] * forward_point[0] + P[9] * forward_point[1] +
              P[10] * forward_point[2] + P[11]);
  float src_col =
      inv_forward_z * (P[0] * forward_point[0] + P[1] * forward_point[1] +
                       P[2] * forward_point[2] + P[3]);
  float src_row =
      inv_forward_z * (P[4] * forward_point[0] + P[5] * forward_point[1] +
                       P[6] * forward_point[2] + P[7]);

  // Extract depth in source image.
  const float src_depth = tex2DLayered(src_depth_maps_texture, src_col + 0.5f,
                                       src_row + 0.5f, image_idx);

  // Projection outside of source image.
  if (src_depth == 0.0f) {
    return max_cost;
  }

  // Project point in source image to world.
  src_col *= src_depth;
  src_row *= src_depth;
  const float backward_point_x =
      inv_P[0] * src_col + inv_P[1] * src_row + inv_P[2] * src_depth + inv_P[3];
  const float backward_point_y =
      inv_P[4] * src_col + inv_P[5] * src_row + inv_P[6] * src_depth + inv_P[7];
  const float backward_point_z = inv_P[8] * src_col + inv_P[9] * src_row +
                                 inv_P[10] * src_depth + inv_P[11];
  const float inv_backward_point_z = 1.0f / backward_point_z;

  // Project world point back to reference image.
  const float backward_col =
      inv_backward_point_z *
      (ref_K[0] * backward_point_x + ref_K[1] * backward_point_z);
  const float backward_row =
      inv_backward_point_z *
      (ref_K[2] * backward_point_y + ref_K[3] * backward_point_z);

  // Return truncated reprojection error between original observation and
  // the forward-backward projected observation.
  const float diff_col = col - backward_col;
  const float diff_row = row - backward_row;
  return min(max_cost, sqrt(diff_col * diff_col + diff_row * diff_row));
}

// Find index of minimum in given values.
template <int kNumCosts>
__device__ inline int FindMinCost(const float costs[kNumCosts]) {
  float min_cost = costs[0];
  int min_cost_idx = 0;
  for (int idx = 1; idx < kNumCosts; ++idx) {
    if (costs[idx] <= min_cost) {
      min_cost = costs[idx];
      min_cost_idx = idx;
    }
  }
  return min_cost_idx;
}

__device__ inline void TransformPDFToCDF(float* probs, const int num_probs) {
  float prob_sum = 0.0f;
  for (int i = 0; i < num_probs; ++i) {
    prob_sum += probs[i];
  }
  const float inv_prob_sum = 1.0f / prob_sum;

  float cum_prob = 0.0f;
  for (int i = 0; i < num_probs; ++i) {
    const float prob = probs[i] * inv_prob_sum;
    cum_prob += prob;
    probs[i] = cum_prob;
  }
}

class LikelihoodComputer {
 public:
  __device__ LikelihoodComputer(const float ncc_sigma,
                                const float min_triangulation_angle,
                                const float incident_angle_sigma)
      : cos_min_triangulation_angle_(cos(min_triangulation_angle)),
        inv_incident_angle_sigma_square_(
            -0.5f / (incident_angle_sigma * incident_angle_sigma)),
        inv_ncc_sigma_square_(-0.5f / (ncc_sigma * ncc_sigma)),
        ncc_norm_factor_(ComputeNCCCostNormFactor(ncc_sigma)) {}

  // Compute forward message from current cost and forward message of
  // previous / neighboring pixel.
  __device__ float ComputeForwardMessage(const float cost,
                                         const float prev) const {
    return ComputeMessage<true>(cost, prev);
  }

  // Compute backward message from current cost and backward message of
  // previous / neighboring pixel.
  __device__ float ComputeBackwardMessage(const float cost,
                                          const float prev) const {
    return ComputeMessage<false>(cost, prev);
  }

  // Compute the selection probability from the forward and backward message.
  __device__ inline float ComputeSelProb(const float alpha, const float beta,
                                         const float prev,
                                         const float prev_weight) const {
    const float zn0 = (1.0f - alpha) * (1.0f - beta);
    const float zn1 = alpha * beta;
    const float curr = zn1 / (zn0 + zn1);
    return prev_weight * prev + (1.0f - prev_weight) * curr;
  }

  // Compute NCC probability. Note that cost = 1 - NCC.
  __device__ inline float ComputeNCCProb(const float cost) const {
    return exp(cost * cost * inv_ncc_sigma_square_) * ncc_norm_factor_;
  }

  // Compute the triangulation angle probability.
  __device__ inline float ComputeTriProb(
      const float cos_triangulation_angle) const {
    const float abs_cos_triangulation_angle = abs(cos_triangulation_angle);
    if (abs_cos_triangulation_angle > cos_min_triangulation_angle_) {
      const float scaled = 1.0f - (1.0f - abs_cos_triangulation_angle) /
                                      (1.0f - cos_min_triangulation_angle_);
      const float likelihood = 1.0f - scaled * scaled;
      return min(1.0f, max(0.0f, likelihood));
    } else {
      return 1.0f;
    }
  }

  // Compute the incident angle probability.
  __device__ inline float ComputeIncProb(const float cos_incident_angle) const {
    const float x = 1.0f - max(0.0f, cos_incident_angle);
    return exp(x * x * inv_incident_angle_sigma_square_);
  }

  // Compute the warping/resolution prior probability.
  template <int kWindowSize>
  __device__ inline float ComputeResolutionProb(const float H[9],
                                                const float row,
                                                const float col) const {
    const int kWindowRadius = kWindowSize / 2;

    // Warp corners of patch in reference image to source image.
    float src1[2];
    const float ref1[2] = {col - kWindowRadius, row - kWindowRadius};
    Mat33DotVec3Homogeneous(H, ref1, src1);
    float src2[2];
    const float ref2[2] = {col - kWindowRadius, row + kWindowRadius};
    Mat33DotVec3Homogeneous(H, ref2, src2);
    float src3[2];
    const float ref3[2] = {col + kWindowRadius, row + kWindowRadius};
    Mat33DotVec3Homogeneous(H, ref3, src3);
    float src4[2];
    const float ref4[2] = {col + kWindowRadius, row - kWindowRadius};
    Mat33DotVec3Homogeneous(H, ref4, src4);

    // Compute area of patches in reference and source image.
    const float ref_area = kWindowSize * kWindowSize;
    const float src_area =
        abs(0.5f * (src1[0] * src2[1] - src2[0] * src1[1] - src1[0] * src4[1] +
                    src2[0] * src3[1] - src3[0] * src2[1] + src4[0] * src1[1] +
                    src3[0] * src4[1] - src4[0] * src3[1]));

    if (ref_area > src_area) {
      return src_area / ref_area;
    } else {
      return ref_area / src_area;
    }
  }

 private:
  // The normalization for the likelihood function, i.e. the normalization for
  // the prior on the matching cost.
  __device__ static inline float ComputeNCCCostNormFactor(
      const float ncc_sigma) {
    // A = sqrt(2pi)*sigma/2*erf(sqrt(2)/sigma)
    // erf(x) = 2/sqrt(pi) * integral from 0 to x of exp(-t^2) dt
    return 2.0f / (sqrt(2.0f * M_PI) * ncc_sigma *
                   erff(2.0f / (ncc_sigma * 1.414213562f)));
  }

  // Compute the forward or backward message.
  template <bool kForward>
  __device__ inline float ComputeMessage(const float cost,
                                         const float prev) const {
    constexpr float kUniformProb = 0.5f;
    constexpr float kNoChangeProb = 0.99999f;
    const float kChangeProb = 1.0f - kNoChangeProb;
    const float emission = ComputeNCCProb(cost);

    float zn0;  // Message for selection probability = 0.
    float zn1;  // Message for selection probability = 1.
    if (kForward) {
      zn0 = (prev * kChangeProb + (1.0f - prev) * kNoChangeProb) * kUniformProb;
      zn1 = (prev * kNoChangeProb + (1.0f - prev) * kChangeProb) * emission;
    } else {
      zn0 = prev * emission * kChangeProb +
            (1.0f - prev) * kUniformProb * kNoChangeProb;
      zn1 = prev * emission * kNoChangeProb +
            (1.0f - prev) * kUniformProb * kChangeProb;
    }

    return zn1 / (zn0 + zn1);
  }

  float cos_min_triangulation_angle_;
  float inv_incident_angle_sigma_square_;
  float inv_ncc_sigma_square_;
  float ncc_norm_factor_;
};

// Rotate normals by 90deg around z-axis in counter-clockwise direction.
__global__ void InitNormalMap(GpuMat<float> normal_map,
                              GpuMat<hiprandState> rand_state_map) {
  const int row = blockDim.y * blockIdx.y + threadIdx.y;
  const int col = blockDim.x * blockIdx.x + threadIdx.x;
  if (col < normal_map.GetWidth() && row < normal_map.GetHeight()) {
    hiprandState rand_state = rand_state_map.Get(row, col);
    float normal[3];
    GenerateRandomNormal(row, col, &rand_state, normal);
    normal_map.SetSlice(row, col, normal);
    rand_state_map.Set(row, col, rand_state);
  }
}

// Rotate normals by 90deg around z-axis in counter-clockwise direction.
__global__ void RotateNormalMap(GpuMat<float> normal_map) {
  const int row = blockDim.y * blockIdx.y + threadIdx.y;
  const int col = blockDim.x * blockIdx.x + threadIdx.x;
  if (col < normal_map.GetWidth() && row < normal_map.GetHeight()) {
    float normal[3];
    normal_map.GetSlice(row, col, normal);
    float rotated_normal[3];
    rotated_normal[0] = normal[1];
    rotated_normal[1] = -normal[0];
    rotated_normal[2] = normal[2];
    normal_map.SetSlice(row, col, rotated_normal);
  }
}

template <int kWindowSize, int kWindowStep>
__global__ void ComputeInitialCost(GpuMat<float> cost_map,
                                   const GpuMat<float> depth_map,
                                   const GpuMat<float> normal_map,
                                   const GpuMat<float> ref_sum_image,
                                   const GpuMat<float> ref_squared_sum_image,
                                   const float sigma_spatial,
                                   const float sigma_color) {
  const int col = blockDim.x * blockIdx.x + threadIdx.x;

  typedef PhotoConsistencyCostComputer<kWindowSize, kWindowStep>
      PhotoConsistencyCostComputerType;
  PhotoConsistencyCostComputerType pcc_computer(sigma_spatial, sigma_color);
  pcc_computer.col = col;

  __shared__ float local_ref_image_data
      [PhotoConsistencyCostComputerType::LocalRefImageType::kDataSize];
  pcc_computer.local_ref_image.data = &local_ref_image_data[0];

  float normal[3] = {0};
  pcc_computer.normal = normal;

  for (int row = 0; row < cost_map.GetHeight(); ++row) {
    // Note that this must be executed even for pixels outside the borders,
    // since pixels are used in the local neighborhood of the current pixel.
    pcc_computer.Read(row);

    if (col < cost_map.GetWidth()) {
      pcc_computer.depth = depth_map.Get(row, col);
      normal_map.GetSlice(row, col, normal);

      pcc_computer.row = row;
      pcc_computer.local_ref_sum = ref_sum_image.Get(row, col);
      pcc_computer.local_ref_squared_sum = ref_squared_sum_image.Get(row, col);

      for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
        pcc_computer.src_image_idx = image_idx;
        cost_map.Set(row, col, image_idx, pcc_computer.Compute());
      }
    }
  }
}

struct SweepOptions {
  float perturbation = 1.0f;
  float depth_min = 0.0f;
  float depth_max = 1.0f;
  int num_samples = 15;
  float sigma_spatial = 3.0f;
  float sigma_color = 0.3f;
  float ncc_sigma = 0.6f;
  float min_triangulation_angle = 0.5f;
  float incident_angle_sigma = 0.9f;
  float prev_sel_prob_weight = 0.0f;
  float geom_consistency_regularizer = 0.1f;
  float geom_consistency_max_cost = 5.0f;
  float filter_min_ncc = 0.1f;
  float filter_min_triangulation_angle = 3.0f;
  int filter_min_num_consistent = 2;
  float filter_geom_consistency_max_cost = 1.0f;
};

template <int kWindowSize, int kWindowStep, bool kGeomConsistencyTerm = false,
          bool kFilterPhotoConsistency = false,
          bool kFilterGeomConsistency = false>
__global__ void SweepFromTopToBottom(
    GpuMat<float> global_workspace, GpuMat<hiprandState> rand_state_map,
    GpuMat<float> cost_map, GpuMat<float> depth_map, GpuMat<float> normal_map,
    GpuMat<uint8_t> consistency_mask, GpuMat<float> sel_prob_map,
    const GpuMat<float> prev_sel_prob_map, const GpuMat<float> ref_sum_image,
    const GpuMat<float> ref_squared_sum_image, const SweepOptions options) {
  const int col = blockDim.x * blockIdx.x + threadIdx.x;

  // Probability for boundary pixels.
  constexpr float kUniformProb = 0.5f;

  LikelihoodComputer likelihood_computer(options.ncc_sigma,
                                         options.min_triangulation_angle,
                                         options.incident_angle_sigma);

  float* forward_message =
      &global_workspace.GetPtr()[col * global_workspace.GetHeight()];
  float* sampling_probs =
      &global_workspace.GetPtr()[global_workspace.GetWidth() *
                                     global_workspace.GetHeight() +
                                 col * global_workspace.GetHeight()];

  //////////////////////////////////////////////////////////////////////////////
  // Compute backward message for all rows. Note that the backward messages are
  // temporarily stored in the sel_prob_map and replaced row by row as the
  // updated forward messages are computed further below.
  //////////////////////////////////////////////////////////////////////////////

  if (col < cost_map.GetWidth()) {
    for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
      // Compute backward message.
      float beta = kUniformProb;
      for (int row = cost_map.GetHeight() - 1; row >= 0; --row) {
        const float cost = cost_map.Get(row, col, image_idx);
        beta = likelihood_computer.ComputeBackwardMessage(cost, beta);
        sel_prob_map.Set(row, col, image_idx, beta);
      }

      // Initialize forward message.
      forward_message[image_idx] = kUniformProb;
    }
  }

  //////////////////////////////////////////////////////////////////////////////
  // Estimate parameters for remaining rows and compute selection probabilities.
  //////////////////////////////////////////////////////////////////////////////

  typedef PhotoConsistencyCostComputer<kWindowSize, kWindowStep>
      PhotoConsistencyCostComputerType;
  PhotoConsistencyCostComputerType pcc_computer(options.sigma_spatial,
                                                options.sigma_color);
  pcc_computer.col = col;

  __shared__ float local_ref_image_data
      [PhotoConsistencyCostComputerType::LocalRefImageType::kDataSize];
  pcc_computer.local_ref_image.data = &local_ref_image_data[0];

  struct ParamState {
    float depth = 0.0f;
    float normal[3] = {0};
  };

  // Parameters of previous pixel in column.
  ParamState prev_param_state;
  // Parameters of current pixel in column.
  ParamState curr_param_state;
  // Randomly sampled parameters.
  ParamState rand_param_state;
  // Cuda PRNG state for random sampling.
  hiprandState rand_state;

  if (col < cost_map.GetWidth()) {
    // Read random state for current column.
    rand_state = rand_state_map.Get(0, col);
    // Parameters for first row in column.
    prev_param_state.depth = depth_map.Get(0, col);
    normal_map.GetSlice(0, col, prev_param_state.normal);
  }

  for (int row = 0; row < cost_map.GetHeight(); ++row) {
    // Note that this must be executed even for pixels outside the borders,
    // since pixels are used in the local neighborhood of the current pixel.
    pcc_computer.Read(row);

    if (col >= cost_map.GetWidth()) {
      continue;
    }

    pcc_computer.row = row;
    pcc_computer.local_ref_sum = ref_sum_image.Get(row, col);
    pcc_computer.local_ref_squared_sum = ref_squared_sum_image.Get(row, col);

    // Propagate the depth at which the current ray intersects with the plane
    // of the normal of the previous ray. This helps to better estimate
    // the depth of very oblique structures, i.e. pixels whose normal direction
    // is significantly different from their viewing direction.
    prev_param_state.depth = PropagateDepth(
        prev_param_state.depth, prev_param_state.normal, row - 1, row);

    // Read parameters for current pixel from previous sweep.
    curr_param_state.depth = depth_map.Get(row, col);
    normal_map.GetSlice(row, col, curr_param_state.normal);

    // Generate random parameters.
    rand_param_state.depth =
        PerturbDepth(options.perturbation, curr_param_state.depth, &rand_state);
    PerturbNormal(row, col, options.perturbation * M_PI,
                  curr_param_state.normal, &rand_state,
                  rand_param_state.normal);

    // Read in the backward message, compute selection probabilities and
    // modulate selection probabilities with priors.

    float point[3];
    ComputePointAtDepth(row, col, curr_param_state.depth, point);

    for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
      const float cost = cost_map.Get(row, col, image_idx);
      const float alpha = likelihood_computer.ComputeForwardMessage(
          cost, forward_message[image_idx]);
      const float beta = sel_prob_map.Get(row, col, image_idx);
      const float prev_prob = prev_sel_prob_map.Get(row, col, image_idx);
      const float sel_prob = likelihood_computer.ComputeSelProb(
          alpha, beta, prev_prob, options.prev_sel_prob_weight);

      float cos_triangulation_angle;
      float cos_incident_angle;
      ComputeViewingAngles(point, curr_param_state.normal, image_idx,
                           &cos_triangulation_angle, &cos_incident_angle);
      const float tri_prob =
          likelihood_computer.ComputeTriProb(cos_triangulation_angle);
      const float inc_prob =
          likelihood_computer.ComputeIncProb(cos_incident_angle);

      float H[9];
      ComposeHomography(image_idx, row, col, curr_param_state.depth,
                        curr_param_state.normal, H);
      const float res_prob =
          likelihood_computer.ComputeResolutionProb<kWindowSize>(H, row, col);

      sampling_probs[image_idx] = sel_prob * tri_prob * inc_prob * res_prob;
    }

    TransformPDFToCDF(sampling_probs, cost_map.GetDepth());

    // Compute matching cost using Monte Carlo sampling of source images. Images
    // with higher selection probability are more likely to be sampled. Hence,
    // if only very few source images see the reference image pixel, the same
    // source image is likely to be sampled many times. Instead of taking
    // the best K probabilities, this sampling scheme has the advantage of
    // being adaptive to any distribution of selection probabilities.

    constexpr int kNumCosts = 5;
    float costs[kNumCosts] = {0};
    const float depths[kNumCosts] = {
        curr_param_state.depth, prev_param_state.depth, rand_param_state.depth,
        curr_param_state.depth, rand_param_state.depth};
    const float* normals[kNumCosts] = {
        curr_param_state.normal, prev_param_state.normal,
        rand_param_state.normal, rand_param_state.normal,
        curr_param_state.normal};

    for (int sample = 0; sample < options.num_samples; ++sample) {
      const float rand_prob = hiprand_uniform(&rand_state) - FLT_EPSILON;

      pcc_computer.src_image_idx = -1;
      for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
        const float prob = sampling_probs[image_idx];
        if (prob > rand_prob) {
          pcc_computer.src_image_idx = image_idx;
          break;
        }
      }

      if (pcc_computer.src_image_idx == -1) {
        continue;
      }

      costs[0] += cost_map.Get(row, col, pcc_computer.src_image_idx);
      if (kGeomConsistencyTerm) {
        costs[0] += options.geom_consistency_regularizer *
                    ComputeGeomConsistencyCost(
                        row, col, depths[0], pcc_computer.src_image_idx,
                        options.geom_consistency_max_cost);
      }

      for (int i = 1; i < kNumCosts; ++i) {
        pcc_computer.depth = depths[i];
        pcc_computer.normal = normals[i];
        costs[i] += pcc_computer.Compute();
        if (kGeomConsistencyTerm) {
          costs[i] += options.geom_consistency_regularizer *
                      ComputeGeomConsistencyCost(
                          row, col, depths[i], pcc_computer.src_image_idx,
                          options.geom_consistency_max_cost);
        }
      }
    }

    // Find the parameters of the minimum cost.
    const int min_cost_idx = FindMinCost<kNumCosts>(costs);
    const float best_depth = depths[min_cost_idx];
    const float* best_normal = normals[min_cost_idx];

    // Save best new parameters.
    depth_map.Set(row, col, best_depth);
    normal_map.SetSlice(row, col, best_normal);

    // Use the new cost to recompute the updated forward message and
    // the selection probability.
    pcc_computer.depth = best_depth;
    pcc_computer.normal = best_normal;
    for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
      // Determine the cost for best depth.
      float cost;
      if (min_cost_idx == 0) {
        cost = cost_map.Get(row, col, image_idx);
      } else {
        pcc_computer.src_image_idx = image_idx;
        cost = pcc_computer.Compute();
        cost_map.Set(row, col, image_idx, cost);
      }

      const float alpha = likelihood_computer.ComputeForwardMessage(
          cost, forward_message[image_idx]);
      const float beta = sel_prob_map.Get(row, col, image_idx);
      const float prev_prob = prev_sel_prob_map.Get(row, col, image_idx);
      const float prob = likelihood_computer.ComputeSelProb(
          alpha, beta, prev_prob, options.prev_sel_prob_weight);
      forward_message[image_idx] = alpha;
      sel_prob_map.Set(row, col, image_idx, prob);
    }

    if (kFilterPhotoConsistency || kFilterGeomConsistency) {
      int num_consistent = 0;

      float best_point[3];
      ComputePointAtDepth(row, col, best_depth, best_point);

      const float min_ncc_prob =
          likelihood_computer.ComputeNCCProb(1.0f - options.filter_min_ncc);
      const float cos_min_triangulation_angle =
          cos(options.filter_min_triangulation_angle);

      for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
        float cos_triangulation_angle;
        float cos_incident_angle;
        ComputeViewingAngles(best_point, best_normal, image_idx,
                             &cos_triangulation_angle, &cos_incident_angle);
        if (cos_triangulation_angle > cos_min_triangulation_angle ||
            cos_incident_angle <= 0.0f) {
          continue;
        }

        if (!kFilterGeomConsistency) {
          if (sel_prob_map.Get(row, col, image_idx) >= min_ncc_prob) {
            consistency_mask.Set(row, col, image_idx, 1);
            num_consistent += 1;
          }
        } else if (!kFilterPhotoConsistency) {
          if (ComputeGeomConsistencyCost(row, col, best_depth, image_idx,
                                         options.geom_consistency_max_cost) <=
              options.filter_geom_consistency_max_cost) {
            consistency_mask.Set(row, col, image_idx, 1);
            num_consistent += 1;
          }
        } else {
          if (sel_prob_map.Get(row, col, image_idx) >= min_ncc_prob &&
              ComputeGeomConsistencyCost(row, col, best_depth, image_idx,
                                         options.geom_consistency_max_cost) <=
                  options.filter_geom_consistency_max_cost) {
            consistency_mask.Set(row, col, image_idx, 1);
            num_consistent += 1;
          }
        }
      }

      if (num_consistent < options.filter_min_num_consistent) {
        depth_map.Set(row, col, 0.0f);
        normal_map.Set(row, col, 0, 0.0f);
        normal_map.Set(row, col, 1, 0.0f);
        normal_map.Set(row, col, 2, 0.0f);
        for (int image_idx = 0; image_idx < cost_map.GetDepth(); ++image_idx) {
          consistency_mask.Set(row, col, image_idx, 0);
        }
      }
    }

    // Update previous depth for next row.
    prev_param_state.depth = best_depth;
    for (int i = 0; i < 3; ++i) {
      prev_param_state.normal[i] = best_normal[i];
    }
  }

  if (col < cost_map.GetWidth()) {
    rand_state_map.Set(0, col, rand_state);
  }
}

PatchMatchCuda::PatchMatchCuda(const PatchMatchOptions& options,
                               const PatchMatch::Problem& problem)
    : options_(options),
      problem_(problem),
      ref_width_(0),
      ref_height_(0),
      rotation_in_half_pi_(0) {
  SetBestCudaDevice(std::stoi(options_.gpu_index));
  InitRefImage();
  InitSourceImages();
  InitTransforms();
  InitWorkspaceMemory();
}

PatchMatchCuda::~PatchMatchCuda() {
  for (size_t i = 0; i < 4; ++i) {
    poses_device_[i].reset();
  }
}

void PatchMatchCuda::Run() {
#define CASE_WINDOW_RADIUS(window_radius, window_step)              \
  case window_radius:                                               \
    RunWithWindowSizeAndStep<2 * window_radius + 1, window_step>(); \
    break;

#define CASE_WINDOW_STEP(window_step)                                 \
  case window_step:                                                   \
    switch (options_.window_radius) {                                 \
      CASE_WINDOW_RADIUS(1, window_step)                              \
      CASE_WINDOW_RADIUS(2, window_step)                              \
      CASE_WINDOW_RADIUS(3, window_step)                              \
      CASE_WINDOW_RADIUS(4, window_step)                              \
      CASE_WINDOW_RADIUS(5, window_step)                              \
      CASE_WINDOW_RADIUS(6, window_step)                              \
      CASE_WINDOW_RADIUS(7, window_step)                              \
      CASE_WINDOW_RADIUS(8, window_step)                              \
      CASE_WINDOW_RADIUS(9, window_step)                              \
      CASE_WINDOW_RADIUS(10, window_step)                             \
      CASE_WINDOW_RADIUS(11, window_step)                             \
      CASE_WINDOW_RADIUS(12, window_step)                             \
      CASE_WINDOW_RADIUS(13, window_step)                             \
      CASE_WINDOW_RADIUS(14, window_step)                             \
      CASE_WINDOW_RADIUS(15, window_step)                             \
      CASE_WINDOW_RADIUS(16, window_step)                             \
      CASE_WINDOW_RADIUS(17, window_step)                             \
      CASE_WINDOW_RADIUS(18, window_step)                             \
      CASE_WINDOW_RADIUS(19, window_step)                             \
      CASE_WINDOW_RADIUS(20, window_step)                             \
      default: {                                                      \
        std::cerr << "Error: Window size not supported" << std::endl; \
        break;                                                        \
      }                                                               \
    }                                                                 \
    break;

  switch (options_.window_step) {
    CASE_WINDOW_STEP(1)
    CASE_WINDOW_STEP(2)
    default: {
      std::cerr << "Error: Window step not supported" << std::endl;
      break;
    }
  }

#undef SWITCH_WINDOW_RADIUS
#undef CALL_RUN_FUNC
}

DepthMap PatchMatchCuda::GetDepthMap() const {
  return DepthMap(depth_map_->CopyToMat(), options_.depth_min,
                  options_.depth_max);
}

NormalMap PatchMatchCuda::GetNormalMap() const {
  return NormalMap(normal_map_->CopyToMat());
}

Mat<float> PatchMatchCuda::GetSelProbMap() const {
  return prev_sel_prob_map_->CopyToMat();
}

std::vector<int> PatchMatchCuda::GetConsistentImageIdxs() const {
  const Mat<uint8_t> mask = consistency_mask_->CopyToMat();
  std::vector<int> consistent_image_idxs;
  std::vector<int> pixel_consistent_image_idxs;
  pixel_consistent_image_idxs.reserve(mask.GetDepth());
  for (size_t r = 0; r < mask.GetHeight(); ++r) {
    for (size_t c = 0; c < mask.GetWidth(); ++c) {
      pixel_consistent_image_idxs.clear();
      for (size_t d = 0; d < mask.GetDepth(); ++d) {
        if (mask.Get(r, c, d)) {
          pixel_consistent_image_idxs.push_back(problem_.src_image_idxs[d]);
        }
      }
      if (pixel_consistent_image_idxs.size() > 0) {
        consistent_image_idxs.push_back(c);
        consistent_image_idxs.push_back(r);
        consistent_image_idxs.push_back(pixel_consistent_image_idxs.size());
        consistent_image_idxs.insert(consistent_image_idxs.end(),
                                     pixel_consistent_image_idxs.begin(),
                                     pixel_consistent_image_idxs.end());
      }
    }
  }
  return consistent_image_idxs;
}

template <int kWindowSize, int kWindowStep>
void PatchMatchCuda::RunWithWindowSizeAndStep() {
  // Wait for all initializations to finish.
  CUDA_SYNC_AND_CHECK();

  CudaTimer total_timer;
  CudaTimer init_timer;

  ComputeCudaConfig();
  ComputeInitialCost<kWindowSize, kWindowStep>
      <<<sweep_grid_size_, sweep_block_size_>>>(
          *cost_map_, *depth_map_, *normal_map_, *ref_image_->sum_image,
          *ref_image_->squared_sum_image, options_.sigma_spatial,
          options_.sigma_color);
  CUDA_SYNC_AND_CHECK();

  init_timer.Print("Initialization");

  const float total_num_steps = options_.num_iterations * 4;

  SweepOptions sweep_options;
  sweep_options.depth_min = options_.depth_min;
  sweep_options.depth_max = options_.depth_max;
  sweep_options.sigma_spatial = options_.sigma_spatial;
  sweep_options.sigma_color = options_.sigma_color;
  sweep_options.num_samples = options_.num_samples;
  sweep_options.ncc_sigma = options_.ncc_sigma;
  sweep_options.min_triangulation_angle =
      DEG2RAD(options_.min_triangulation_angle);
  sweep_options.incident_angle_sigma = options_.incident_angle_sigma;
  sweep_options.geom_consistency_regularizer =
      options_.geom_consistency_regularizer;
  sweep_options.geom_consistency_max_cost = options_.geom_consistency_max_cost;
  sweep_options.filter_min_ncc = options_.filter_min_ncc;
  sweep_options.filter_min_triangulation_angle =
      DEG2RAD(options_.filter_min_triangulation_angle);
  sweep_options.filter_min_num_consistent = options_.filter_min_num_consistent;
  sweep_options.filter_geom_consistency_max_cost =
      options_.filter_geom_consistency_max_cost;

  for (int iter = 0; iter < options_.num_iterations; ++iter) {
    CudaTimer iter_timer;

    for (int sweep = 0; sweep < 4; ++sweep) {
      CudaTimer sweep_timer;

      // Expenentially reduce amount of perturbation during the optimization.
      sweep_options.perturbation = 1.0f / std::pow(2.0f, iter + sweep / 4.0f);

      // Linearly increase the influence of previous selection probabilities.
      sweep_options.prev_sel_prob_weight =
          static_cast<float>(iter * 4 + sweep) / total_num_steps;

      const bool last_sweep = iter == options_.num_iterations - 1 && sweep == 3;

#define CALL_SWEEP_FUNC                                                  \
  SweepFromTopToBottom<kWindowSize, kWindowStep, kGeomConsistencyTerm,   \
                       kFilterPhotoConsistency, kFilterGeomConsistency>  \
      <<<sweep_grid_size_, sweep_block_size_>>>(                         \
          *global_workspace_, *rand_state_map_, *cost_map_, *depth_map_, \
          *normal_map_, *consistency_mask_, *sel_prob_map_,              \
          *prev_sel_prob_map_, *ref_image_->sum_image,                   \
          *ref_image_->squared_sum_image, sweep_options);

      if (last_sweep) {
        if (options_.filter) {
          consistency_mask_.reset(new GpuMat<uint8_t>(cost_map_->GetWidth(),
                                                      cost_map_->GetHeight(),
                                                      cost_map_->GetDepth()));
          consistency_mask_->FillWithScalar(0);
        }
        if (options_.geom_consistency) {
          const bool kGeomConsistencyTerm = true;
          if (options_.filter) {
            const bool kFilterPhotoConsistency = true;
            const bool kFilterGeomConsistency = true;
            CALL_SWEEP_FUNC
          } else {
            const bool kFilterPhotoConsistency = false;
            const bool kFilterGeomConsistency = false;
            CALL_SWEEP_FUNC
          }
        } else {
          const bool kGeomConsistencyTerm = false;
          if (options_.filter) {
            const bool kFilterPhotoConsistency = true;
            const bool kFilterGeomConsistency = false;
            CALL_SWEEP_FUNC
          } else {
            const bool kFilterPhotoConsistency = false;
            const bool kFilterGeomConsistency = false;
            CALL_SWEEP_FUNC
          }
        }
      } else {
        const bool kFilterPhotoConsistency = false;
        const bool kFilterGeomConsistency = false;
        if (options_.geom_consistency) {
          const bool kGeomConsistencyTerm = true;
          CALL_SWEEP_FUNC
        } else {
          const bool kGeomConsistencyTerm = false;
          CALL_SWEEP_FUNC
        }
      }

#undef CALL_SWEEP_FUNC

      CUDA_SYNC_AND_CHECK();

      Rotate();

      // Rotate selected image map.
      if (last_sweep && options_.filter) {
        std::unique_ptr<GpuMat<uint8_t>> rot_consistency_mask_(
            new GpuMat<uint8_t>(cost_map_->GetWidth(), cost_map_->GetHeight(),
                                cost_map_->GetDepth()));
        consistency_mask_->Rotate(rot_consistency_mask_.get());
        consistency_mask_.swap(rot_consistency_mask_);
      }

      sweep_timer.Print(" Sweep " + std::to_string(sweep + 1));
    }

    iter_timer.Print("Iteration " + std::to_string(iter + 1));
  }

  total_timer.Print("Total");
}

void PatchMatchCuda::ComputeCudaConfig() {
  sweep_block_size_.x = THREADS_PER_BLOCK;
  sweep_block_size_.y = 1;
  sweep_block_size_.z = 1;
  sweep_grid_size_.x = (depth_map_->GetWidth() - 1) / THREADS_PER_BLOCK + 1;
  sweep_grid_size_.y = 1;
  sweep_grid_size_.z = 1;

  elem_wise_block_size_.x = THREADS_PER_BLOCK;
  elem_wise_block_size_.y = THREADS_PER_BLOCK;
  elem_wise_block_size_.z = 1;
  elem_wise_grid_size_.x = (depth_map_->GetWidth() - 1) / THREADS_PER_BLOCK + 1;
  elem_wise_grid_size_.y =
      (depth_map_->GetHeight() - 1) / THREADS_PER_BLOCK + 1;
  elem_wise_grid_size_.z = 1;
}

void PatchMatchCuda::InitRefImage() {
  const Image& ref_image = problem_.images->at(problem_.ref_image_idx);

  ref_width_ = ref_image.GetWidth();
  ref_height_ = ref_image.GetHeight();

  // Upload to device.
  ref_image_.reset(new GpuMatRefImage(ref_width_, ref_height_));
  const std::vector<uint8_t> ref_image_array =
      ref_image.GetBitmap().ConvertToRowMajorArray();
  ref_image_->Filter(ref_image_array.data(), options_.window_radius,
                     options_.window_step, options_.sigma_spatial,
                     options_.sigma_color);

  ref_image_device_.reset(
      new CudaArrayWrapper<uint8_t>(ref_width_, ref_height_, 1));
  ref_image_device_->CopyFromGpuMat(*ref_image_->image);

  // Create texture.
  ref_image_texture.addressMode[0] = hipAddressModeBorder;
  ref_image_texture.addressMode[1] = hipAddressModeBorder;
  ref_image_texture.addressMode[2] = hipAddressModeBorder;
  ref_image_texture.filterMode = hipFilterModePoint;
  ref_image_texture.normalized = false;
  CUDA_SAFE_CALL(
      hipBindTextureToArray(ref_image_texture, ref_image_device_->GetPtr()));
}

void PatchMatchCuda::InitSourceImages() {
  // Determine maximum image size.
  size_t max_width = 0;
  size_t max_height = 0;
  for (const auto image_idx : problem_.src_image_idxs) {
    const Image& image = problem_.images->at(image_idx);
    if (image.GetWidth() > max_width) {
      max_width = image.GetWidth();
    }
    if (image.GetHeight() > max_height) {
      max_height = image.GetHeight();
    }
  }

  // Upload source images to device.
  {
    // Copy source images to contiguous memory block.
    const uint8_t kDefaultValue = 0;
    std::vector<uint8_t> src_images_host_data(
        static_cast<size_t>(max_width * max_height *
                            problem_.src_image_idxs.size()),
        kDefaultValue);
    for (size_t i = 0; i < problem_.src_image_idxs.size(); ++i) {
      const Image& image = problem_.images->at(problem_.src_image_idxs[i]);
      const Bitmap& bitmap = image.GetBitmap();
      uint8_t* dest = src_images_host_data.data() + max_width * max_height * i;
      for (size_t r = 0; r < image.GetHeight(); ++r) {
        memcpy(dest, bitmap.GetScanline(r), image.GetWidth() * sizeof(uint8_t));
        dest += max_width;
      }
    }

    // Upload to device.
    src_images_device_.reset(new CudaArrayWrapper<uint8_t>(
        max_width, max_height, problem_.src_image_idxs.size()));
    src_images_device_->CopyToDevice(src_images_host_data.data());

    // Create source images texture.
    src_images_texture.addressMode[0] = hipAddressModeBorder;
    src_images_texture.addressMode[1] = hipAddressModeBorder;
    src_images_texture.addressMode[2] = hipAddressModeBorder;
    src_images_texture.filterMode = hipFilterModeLinear;
    src_images_texture.normalized = false;
    CUDA_SAFE_CALL(hipBindTextureToArray(src_images_texture,
                                          src_images_device_->GetPtr()));
  }

  // Upload source depth maps to device.
  if (options_.geom_consistency) {
    const float kDefaultValue = 0.0f;
    std::vector<float> src_depth_maps_host_data(
        static_cast<size_t>(max_width * max_height *
                            problem_.src_image_idxs.size()),
        kDefaultValue);
    for (size_t i = 0; i < problem_.src_image_idxs.size(); ++i) {
      const DepthMap& depth_map =
          problem_.depth_maps->at(problem_.src_image_idxs[i]);
      float* dest =
          src_depth_maps_host_data.data() + max_width * max_height * i;
      for (size_t r = 0; r < depth_map.GetHeight(); ++r) {
        memcpy(dest, depth_map.GetPtr() + r * depth_map.GetWidth(),
               depth_map.GetWidth() * sizeof(float));
        dest += max_width;
      }
    }

    src_depth_maps_device_.reset(new CudaArrayWrapper<float>(
        max_width, max_height, problem_.src_image_idxs.size()));
    src_depth_maps_device_->CopyToDevice(src_depth_maps_host_data.data());

    // Create source depth maps texture.
    src_depth_maps_texture.addressMode[0] = hipAddressModeBorder;
    src_depth_maps_texture.addressMode[1] = hipAddressModeBorder;
    src_depth_maps_texture.addressMode[2] = hipAddressModeBorder;
    // TODO: Check if linear interpolation improves results or not.
    src_depth_maps_texture.filterMode = hipFilterModePoint;
    src_depth_maps_texture.normalized = false;
    CUDA_SAFE_CALL(hipBindTextureToArray(src_depth_maps_texture,
                                          src_depth_maps_device_->GetPtr()));
  }
}

void PatchMatchCuda::InitTransforms() {
  const Image& ref_image = problem_.images->at(problem_.ref_image_idx);

  //////////////////////////////////////////////////////////////////////////////
  // Generate rotated versions (counter-clockwise) of calibration matrix.
  //////////////////////////////////////////////////////////////////////////////

  for (size_t i = 0; i < 4; ++i) {
    ref_K_host_[i][0] = ref_image.GetK()[0];
    ref_K_host_[i][1] = ref_image.GetK()[2];
    ref_K_host_[i][2] = ref_image.GetK()[4];
    ref_K_host_[i][3] = ref_image.GetK()[5];
  }

  // Rotated by 90 degrees.
  std::swap(ref_K_host_[1][0], ref_K_host_[1][2]);
  std::swap(ref_K_host_[1][1], ref_K_host_[1][3]);
  ref_K_host_[1][3] = ref_width_ - 1 - ref_K_host_[1][3];

  // Rotated by 180 degrees.
  ref_K_host_[2][1] = ref_width_ - 1 - ref_K_host_[2][1];
  ref_K_host_[2][3] = ref_height_ - 1 - ref_K_host_[2][3];

  // Rotated by 270 degrees.
  std::swap(ref_K_host_[3][0], ref_K_host_[3][2]);
  std::swap(ref_K_host_[3][1], ref_K_host_[3][3]);
  ref_K_host_[3][1] = ref_height_ - 1 - ref_K_host_[3][1];

  // Extract 1/fx, -cx/fx, fy, -cy/fy.
  for (size_t i = 0; i < 4; ++i) {
    ref_inv_K_host_[i][0] = 1.0f / ref_K_host_[i][0];
    ref_inv_K_host_[i][1] = -ref_K_host_[i][1] / ref_K_host_[i][0];
    ref_inv_K_host_[i][2] = 1.0f / ref_K_host_[i][2];
    ref_inv_K_host_[i][3] = -ref_K_host_[i][3] / ref_K_host_[i][2];
  }

  // Bind 0 degrees version to constant global memory.
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_K), ref_K_host_[0], sizeof(float) * 4, 0,
                                    hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_inv_K), ref_inv_K_host_[0],
                                    sizeof(float) * 4, 0,
                                    hipMemcpyHostToDevice));

  //////////////////////////////////////////////////////////////////////////////
  // Generate rotated versions of camera poses.
  //////////////////////////////////////////////////////////////////////////////

  float rotated_R[9];
  memcpy(rotated_R, ref_image.GetR(), 9 * sizeof(float));

  float rotated_T[3];
  memcpy(rotated_T, ref_image.GetT(), 3 * sizeof(float));

  // Matrix for 90deg rotation around Z-axis in counter-clockwise direction.
  const float R_z90[9] = {0, 1, 0, -1, 0, 0, 0, 0, 1};

  for (size_t i = 0; i < 4; ++i) {
    const size_t kNumTformParams = 4 + 9 + 3 + 3 + 12 + 12;
    std::vector<float> poses_host_data(kNumTformParams *
                                       problem_.src_image_idxs.size());
    int offset = 0;
    for (const auto image_idx : problem_.src_image_idxs) {
      const Image& image = problem_.images->at(image_idx);

      const float K[4] = {image.GetK()[0], image.GetK()[2], image.GetK()[4],
                          image.GetK()[5]};
      memcpy(poses_host_data.data() + offset, K, 4 * sizeof(float));
      offset += 4;

      float rel_R[9];
      float rel_T[3];
      ComputeRelativePose(rotated_R, rotated_T, image.GetR(), image.GetT(),
                          rel_R, rel_T);
      memcpy(poses_host_data.data() + offset, rel_R, 9 * sizeof(float));
      offset += 9;
      memcpy(poses_host_data.data() + offset, rel_T, 3 * sizeof(float));
      offset += 3;

      float C[3];
      ComputeProjectionCenter(rel_R, rel_T, C);
      memcpy(poses_host_data.data() + offset, C, 3 * sizeof(float));
      offset += 3;

      float P[12];
      ComposeProjectionMatrix(image.GetK(), rel_R, rel_T, P);
      memcpy(poses_host_data.data() + offset, P, 12 * sizeof(float));
      offset += 12;

      float inv_P[12];
      ComposeInverseProjectionMatrix(image.GetK(), rel_R, rel_T, inv_P);
      memcpy(poses_host_data.data() + offset, inv_P, 12 * sizeof(float));
      offset += 12;
    }

    poses_device_[i].reset(new CudaArrayWrapper<float>(
        kNumTformParams, problem_.src_image_idxs.size(), 1));
    poses_device_[i]->CopyToDevice(poses_host_data.data());

    RotatePose(R_z90, rotated_R, rotated_T);
  }

  poses_texture.addressMode[0] = hipAddressModeBorder;
  poses_texture.addressMode[1] = hipAddressModeBorder;
  poses_texture.addressMode[2] = hipAddressModeBorder;
  poses_texture.filterMode = hipFilterModePoint;
  poses_texture.normalized = false;
  CUDA_SAFE_CALL(
      hipBindTextureToArray(poses_texture, poses_device_[0]->GetPtr()));
}

void PatchMatchCuda::InitWorkspaceMemory() {
  rand_state_map_.reset(new GpuMatPRNG(ref_width_, ref_height_));

  depth_map_.reset(new GpuMat<float>(ref_width_, ref_height_));
  if (options_.geom_consistency) {
    const DepthMap& init_depth_map =
        problem_.depth_maps->at(problem_.ref_image_idx);
    depth_map_->CopyToDevice(init_depth_map.GetPtr(),
                             init_depth_map.GetWidth() * sizeof(float));
  } else {
    depth_map_->FillWithRandomNumbers(options_.depth_min, options_.depth_max,
                                      *rand_state_map_);
  }

  normal_map_.reset(new GpuMat<float>(ref_width_, ref_height_, 3));

  // Note that it is not necessary to keep the selection probability map in
  // memory for all pixels. Theoretically, it is possible to incorporate
  // the temporary selection probabilities in the global_workspace_.
  // However, it is useful to keep the probabilities for the entire image
  // in memory, so that it can be exported.
  sel_prob_map_.reset(new GpuMat<float>(ref_width_, ref_height_,
                                        problem_.src_image_idxs.size()));
  prev_sel_prob_map_.reset(new GpuMat<float>(ref_width_, ref_height_,
                                             problem_.src_image_idxs.size()));
  prev_sel_prob_map_->FillWithScalar(0.5f);

  cost_map_.reset(new GpuMat<float>(ref_width_, ref_height_,
                                    problem_.src_image_idxs.size()));

  const int ref_max_dim = std::max(ref_width_, ref_height_);
  global_workspace_.reset(
      new GpuMat<float>(ref_max_dim, problem_.src_image_idxs.size(), 2));

  consistency_mask_.reset(new GpuMat<uint8_t>(0, 0, 0));

  ComputeCudaConfig();

  if (options_.geom_consistency) {
    const NormalMap& init_normal_map =
        problem_.normal_maps->at(problem_.ref_image_idx);
    normal_map_->CopyToDevice(init_normal_map.GetPtr(),
                              init_normal_map.GetWidth() * sizeof(float));
  } else {
    InitNormalMap<<<elem_wise_grid_size_, elem_wise_block_size_>>>(
        *normal_map_, *rand_state_map_);
  }
}

void PatchMatchCuda::Rotate() {
  rotation_in_half_pi_ = (rotation_in_half_pi_ + 1) % 4;

  size_t width;
  size_t height;
  if (rotation_in_half_pi_ % 2 == 0) {
    width = ref_width_;
    height = ref_height_;
  } else {
    width = ref_height_;
    height = ref_width_;
  }

  // Rotate random map.
  {
    std::unique_ptr<GpuMatPRNG> rotated_rand_state_map(
        new GpuMatPRNG(width, height));
    rand_state_map_->Rotate(rotated_rand_state_map.get());
    rand_state_map_.swap(rotated_rand_state_map);
  }

  // Rotate depth map.
  {
    std::unique_ptr<GpuMat<float>> rotated_depth_map(
        new GpuMat<float>(width, height));
    depth_map_->Rotate(rotated_depth_map.get());
    depth_map_.swap(rotated_depth_map);
  }

  // Rotate normal map.
  {
    RotateNormalMap<<<elem_wise_grid_size_, elem_wise_block_size_>>>(
        *normal_map_);
    std::unique_ptr<GpuMat<float>> rotated_normal_map(
        new GpuMat<float>(width, height, 3));
    normal_map_->Rotate(rotated_normal_map.get());
    normal_map_.swap(rotated_normal_map);
  }

  // Rotate reference image.
  {
    std::unique_ptr<GpuMatRefImage> rotated_ref_image(
        new GpuMatRefImage(width, height));
    ref_image_->image->Rotate(rotated_ref_image->image.get());
    ref_image_->sum_image->Rotate(rotated_ref_image->sum_image.get());
    ref_image_->squared_sum_image->Rotate(
        rotated_ref_image->squared_sum_image.get());
    ref_image_.swap(rotated_ref_image);
  }

  // Bind rotated reference image to texture.
  ref_image_device_.reset(new CudaArrayWrapper<uint8_t>(width, height, 1));
  ref_image_device_->CopyFromGpuMat(*ref_image_->image);
  CUDA_SAFE_CALL(hipUnbindTexture(ref_image_texture));
  CUDA_SAFE_CALL(
      hipBindTextureToArray(ref_image_texture, ref_image_device_->GetPtr()));

  // Rotate selection probability map.
  prev_sel_prob_map_.reset(
      new GpuMat<float>(width, height, problem_.src_image_idxs.size()));
  sel_prob_map_->Rotate(prev_sel_prob_map_.get());
  sel_prob_map_.reset(
      new GpuMat<float>(width, height, problem_.src_image_idxs.size()));

  // Rotate cost map.
  {
    std::unique_ptr<GpuMat<float>> rotated_cost_map(
        new GpuMat<float>(width, height, problem_.src_image_idxs.size()));
    cost_map_->Rotate(rotated_cost_map.get());
    cost_map_.swap(rotated_cost_map);
  }

  // Rotate transformations.
  CUDA_SAFE_CALL(hipUnbindTexture(poses_texture));
  CUDA_SAFE_CALL(hipBindTextureToArray(
      poses_texture, poses_device_[rotation_in_half_pi_]->GetPtr()));

  // Rotate calibration.
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ref_K), ref_K_host_[rotation_in_half_pi_],
                                    sizeof(float) * 4, 0,
                                    hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(
      hipMemcpyToSymbol(HIP_SYMBOL(ref_inv_K), ref_inv_K_host_[rotation_in_half_pi_],
                         sizeof(float) * 4, 0, hipMemcpyHostToDevice));

  // Recompute Cuda configuration for rotated reference image.
  ComputeCudaConfig();
}

}  // namespace mvs
}  // namespace colmap
