#include "hip/hip_runtime.h"
// Copyright (c) 2018, ETH Zurich and UNC Chapel Hill.
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//
//     * Neither the name of ETH Zurich and UNC Chapel Hill nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDERS OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// Author: Johannes L. Schoenberger (jsch-at-demuc-dot-de)

#include "mvs/gpu_mat_prng.h"

namespace colmap {
namespace mvs {
namespace {

__global__ void InitRandomStateKernel(GpuMat<hiprandState> output) {
  const size_t row = blockIdx.y * blockDim.y + threadIdx.y;
  const size_t col = blockIdx.x * blockDim.x + threadIdx.x;

  const size_t uniqueBlockIndex = blockIdx.y * gridDim.x + blockIdx.x;
  const size_t id = uniqueBlockIndex * blockDim.y * blockDim.x +
                    threadIdx.y * blockDim.x + threadIdx.x;

  // Each thread gets same seed, a different sequence number, no offset.
  if (col < output.GetWidth() && row < output.GetHeight()) {
    hiprand_init(id, 0, 0, &output.GetRef(row, col));
  }
}

}  // namespace

GpuMatPRNG::GpuMatPRNG(const int width, const int height)
    : GpuMat(width, height) {
  InitRandomStateKernel<<<gridSize_, blockSize_>>>(*this);
}

}  // namespace mvs
}  // namespace colmap
