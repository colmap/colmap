#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//  File:           ProgramCU.cu
//  Author:         Changchang Wu
//  Description :   implementation of ProgramCU and all CUDA kernels
//
//  Copyright (c) 2011  Changchang Wu (ccwu@cs.washington.edu)
//    and the University of Washington at Seattle
//
//  This library is free software; you can redistribute it and/or
//  modify it under the terms of the GNU General Public
//  License as published by the Free Software Foundation; either
//  Version 3 of the License, or (at your option) any later version.
//
//  This library is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
//  General Public License for more details.
//
////////////////////////////////////////////////////////////////////////////////

#include <stdio.h>
#include <float.h>
#include "CuTexImage.h"
#include "ProgramCU.h"

#define IMUL(X, Y) __mul24(X, Y)
#define FDIV(X, Y) __fdividef(X, Y)
#define FDIV2(X, Y) ((X) / (Y))
#define MAX_BLOCKLEN 65535
#define MAX_BLOCKLEN_ALIGN 65504
#define MAX_TEXSIZE (1 << 29)
#define TEX_TOOBIG4(sz) (sz >> 31)
#define REDUCTION_NBLOCK 32

namespace pba {

inline void CuTexImage::BindTexture(textureReference& texRef) {
  size_t sz = GetDataSize();
  if (sz > MAX_TEXSIZE)
    fprintf(stderr, "hipBindTexture: %lX > %d\n", sz, MAX_TEXSIZE);
  hipError_t e =
      hipBindTexture(NULL, &texRef, data(), &texRef.channelDesc, sz);
}

inline void CuTexImage::BindTexture(textureReference& texRef, int offset,
                                    size_t size) {
  hipError_t e = hipBindTexture(NULL, &texRef, (char*)_cuData + offset,
                                  &texRef.channelDesc, size);
  if (e) fprintf(stderr, "hipBindTexture: none-zero offset\n");
}

inline void CuTexImage::BindTexture2(textureReference& texRef1,
                                     textureReference& texRef2) {
  size_t sz = GetDataSize();
  if (sz <= MAX_TEXSIZE) {
    BindTexture(texRef1);
  } else {
    BindTexture(texRef1, 0, MAX_TEXSIZE);
    BindTexture(texRef2, MAX_TEXSIZE, sz - MAX_TEXSIZE);
  }
}

inline void CuTexImage::BindTexture4(textureReference& texRef1,
                                     textureReference& texRef2,
                                     textureReference& texRef3,
                                     textureReference& texRef4) {
  size_t sz = GetDataSize();
  if (sz <= MAX_TEXSIZE) {
    BindTexture(texRef1);
  } else {
    BindTexture(texRef1, 0, MAX_TEXSIZE);
    if (sz <= 2 * MAX_TEXSIZE) {
      BindTexture(texRef2, MAX_TEXSIZE, sz - MAX_TEXSIZE);
    } else {
      BindTexture(texRef2, MAX_TEXSIZE, MAX_TEXSIZE);
      if (sz <= 3 * MAX_TEXSIZE) {
        BindTexture(texRef3, MAX_TEXSIZE * 2, sz - MAX_TEXSIZE * 2);
      } else {
        BindTexture(texRef3, MAX_TEXSIZE * 2, MAX_TEXSIZE);
        BindTexture(texRef4, MAX_TEXSIZE * 3, sz - MAX_TEXSIZE * 3);
      }
    }
  }
}

inline int CuTexImage::BindTextureX(textureReference& texRef1,
                                    textureReference& texRef2,
                                    textureReference& texRef3,
                                    textureReference& texRef4, bool force4) {
  size_t szjc = GetDataSize();
  if (TEX_TOOBIG4(szjc)) {
    return 0;
  } else if (force4) {
    BindTexture4(texRef1, texRef2, texRef3, texRef4);
    return 4;
  } else if (szjc > 2 * MAX_TEXSIZE) {
    return 0;
  } else if (szjc > MAX_TEXSIZE) {
    BindTexture2(texRef1, texRef2);
    return 2;
  } else {
    BindTexture(texRef1);
    return 1;
  }
}

void ProgramCU::FinishWorkCUDA() { hipDeviceSynchronize(); }

int ProgramCU::CheckErrorCUDA(const char* location) {
  hipError_t e = hipGetLastError();
  if (e) {
    if (location) fprintf(stderr, "%s:\t", location);
    fprintf(stderr, "%s(%d)\n", hipGetErrorString(e), e);
    throw location;
  } else {
    // fprintf(stderr, "%s:\n",  location);
    return 0;
  }
}

inline void ProgramCU::GetBlockConfiguration(unsigned int nblock,
                                             unsigned int& bw,
                                             unsigned int& bh) {
  if (nblock <= MAX_BLOCKLEN) {
    bw = nblock;
    bh = 1;
  } else {
    bh = (nblock + MAX_BLOCKLEN_ALIGN - 1) / MAX_BLOCKLEN_ALIGN;
    bw = (nblock + bh - 1) / bh;
    bw = ((bw + 31) / 32) * 32;
    bh = (nblock + bw - 1) / bw;
  }
}

void ProgramCU::ClearPreviousError() { hipGetLastError(); }

void ProgramCU::ResetCurrentDevice() {
  int device = 0;
  hipGetDevice(&device);
  hipDeviceReset();
  if (device > 0) hipSetDevice(device);
}

size_t ProgramCU::GetCudaMemoryCap() {
  int device;
  if (hipGetDevice(&device) != hipSuccess) return 0;
  hipDeviceProp_t prop;
  if (hipGetDeviceProperties(&prop, device) == hipSuccess) {
    if (prop.major == 9999 && prop.minor == 9999) return 0;
    return prop.totalGlobalMem;
  } else
    return 0;
}
int ProgramCU::SetCudaDevice(int device) {
  int count = 0, device_used;
  if (hipGetDeviceCount(&count) || count <= 0) {
    ProgramCU::CheckErrorCUDA("CheckCudaDevice");
    return 0;
  } else if (count == 1) {
    hipDeviceProp_t deviceProp;
    if (hipGetDeviceProperties(&deviceProp, 0) != hipSuccess) {
      fprintf(stderr, "CheckCudaDevice: no device supporting CUDA.\n");
      return 0;
    }
    if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
      fprintf(stderr, "CheckCudaDevice: no device supporting CUDA.\n");
      return 0;
    }
  }

  if (device > 0 && device < count) {
    hipSetDevice(device);
    CheckErrorCUDA("hipSetDevice\n");
  }
  hipGetDevice(&device_used);
  if (device != device_used)
    fprintf(stderr,
            "ERROR:   Cannot set device to %d\n"
            "WARNING: Use  device-%d instead (out of %d)\n",
            device, device_used, count);
  return 1;
}

#define WARP_REDUCTION_32(value)                                       \
  __syncthreads();                                                     \
  if (threadIdx.x < 16) value[threadIdx.x] += value[threadIdx.x + 16]; \
  if (threadIdx.x < 8) value[threadIdx.x] += value[threadIdx.x + 8];   \
  if (threadIdx.x < 4) value[threadIdx.x] += value[threadIdx.x + 4];   \
  if (threadIdx.x < 2) value[threadIdx.x] += value[threadIdx.x + 2];

#define WARP_REDUCTION_64(value)                                       \
  __syncthreads();                                                     \
  if (threadIdx.x < 32) value[threadIdx.x] += value[threadIdx.x + 32]; \
  WARP_REDUCTION_32(value)

#define WARP_REDUCTION_128(value)                                      \
  __syncthreads();                                                     \
  if (threadIdx.x < 64) value[threadIdx.x] += value[threadIdx.x + 64]; \
  WARP_REDUCTION_64(value)

#define WARP_REDUCTION_256(value)                                        \
  __syncthreads();                                                       \
  if (threadIdx.x < 128) value[threadIdx.x] += value[threadIdx.x + 128]; \
  WARP_REDUCTION_128(value)

__global__ void vector_max_kernel(const float* x, int len, int blen,
                                  float* result) {
  __shared__ float value[256];
  int bstart = blen * blockIdx.x;
  int start = bstart + threadIdx.x;
  int end = min(len, bstart + blen);

  float v = 0;
  for (int i = start; i < end; i += blockDim.x) v = max(v, fabs(x[i]));
  value[threadIdx.x] = v;
  // reduce to the first two values
  __syncthreads();
  if (threadIdx.x < 128)
    value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 128]);
  __syncthreads();
  if (threadIdx.x < 64)
    value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 64]);
  __syncthreads();
  if (threadIdx.x < 32)
    value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 32]);
  if (threadIdx.x < 16)
    value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 16]);
  if (threadIdx.x < 8)
    value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 8]);
  if (threadIdx.x < 4)
    value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 4]);
  if (threadIdx.x < 2)
    value[threadIdx.x] = max(value[threadIdx.x], value[threadIdx.x + 2]);
  // write back
  if (threadIdx.x == 0) result[blockIdx.x] = max(value[0], value[1]);
}

float ProgramCU::ComputeVectorMax(CuTexImage& vector, CuTexImage& buf) {
  const unsigned int nblock = 32;
  const unsigned int bsize = 256;
  int len = vector.GetLength();
  int blen = ((len + nblock - 1) / nblock + bsize - 1) / bsize * bsize;

  ////////////////////////////////
  dim3 grid(nblock), block(bsize);

  /////////////////////////////////
  buf.InitTexture(nblock, 1);
  vector_max_kernel<<<grid, block>>>(vector.data(), len, blen, buf.data());
  ProgramCU::CheckErrorCUDA("ComputeVectorMax");

  float data[nblock], result = 0;
  buf.CopyToHost(data);
  for (unsigned int i = 0; i < nblock; ++i) result = max(result, data[i]);
  return result;
}

__global__ void vector_norm_kernel(const float* x, int len, int blen,
                                   float* result) {
  __shared__ float value[256];
  int bstart = blen * blockIdx.x;
  int start = bstart + threadIdx.x;
  int end = min(len, bstart + blen);

  float v = 0;
  for (int i = start; i < end; i += blockDim.x) {
    float temp = x[i];
    v += (temp * temp);
  }
  value[threadIdx.x] = v;
  // reduce to the first two values
  WARP_REDUCTION_256(value);

  // write back
  if (threadIdx.x == 0) result[blockIdx.x] = (value[0] + value[1]);
}

double ProgramCU::ComputeVectorNorm(CuTexImage& vector, CuTexImage& buf) {
  const unsigned int nblock = REDUCTION_NBLOCK;
  unsigned int bsize = 256;
  int len = vector.GetLength();
  int blen = ((len + nblock - 1) / nblock + bsize - 1) / bsize * bsize;

  ////////////////////////////////
  dim3 grid(nblock), block(bsize);

  /////////////////////////////////
  buf.InitTexture(nblock, 1);
  vector_norm_kernel<<<grid, block>>>(vector.data(), len, blen, buf.data());
  ProgramCU::CheckErrorCUDA("ComputeVectorNorm");

  float data[nblock];
  buf.CopyToHost(data);
  double result = 0;
  for (unsigned int i = 0; i < nblock; ++i) result += data[i];
  return result;
}

__global__ void vector_sum_kernel(const float* x, int len, int blen,
                                  float* result) {
  __shared__ float value[256];
  int bstart = blen * blockIdx.x;
  int start = bstart + threadIdx.x;
  int end = min(len, bstart + blen);
  float v = 0;
  for (int i = start; i < end; i += blockDim.x) v += x[i];

  value[threadIdx.x] = v;
  // reduce to the first two values
  WARP_REDUCTION_256(value);

  // write back
  if (threadIdx.x == 0) result[blockIdx.x] = (value[0] + value[1]);
}

float ProgramCU::ComputeVectorSum(CuTexImage& vector, CuTexImage& buf,
                                  int skip) {
  const unsigned int nblock = REDUCTION_NBLOCK;
  unsigned int bsize = 256;
  int len = vector.GetLength() - skip;
  int blen = ((len + nblock - 1) / nblock + bsize - 1) / bsize * bsize;

  ////////////////////////////////
  dim3 grid(nblock), block(bsize);

  /////////////////////////////////
  buf.InitTexture(nblock, 1);
  vector_sum_kernel<<<grid, block>>>((vector.data()) + skip, len, blen,
                                     buf.data());
  ProgramCU::CheckErrorCUDA("ComputeVectorSum");

  float data[nblock];
  buf.CopyToHost(data);
  double result = 0;
  for (unsigned int i = 0; i < nblock; ++i) result += data[i];
  return (float)result;
}

__global__ void vector_dotproduct_kernel(const float* a, const float* b,
                                         int len, int blen, float* result) {
  __shared__ float value[256];
  int bstart = blen * blockIdx.x;
  int start = bstart + threadIdx.x;
  int end = min(len, bstart + blen);

  float v = 0;
  for (int i = start; i < end; i += blockDim.x) v += (a[i] * b[i]);
  value[threadIdx.x] = v;

  // reduce to the first two values
  WARP_REDUCTION_256(value);

  // write back
  if (threadIdx.x == 0) result[blockIdx.x] = (value[0] + value[1]);
}

double ProgramCU::ComputeVectorDot(CuTexImage& vector1, CuTexImage& vector2,
                                   CuTexImage& buf) {
  const unsigned int nblock = REDUCTION_NBLOCK;
  unsigned int bsize = 256;
  int len = vector1.GetLength();
  int blen = ((len + nblock - 1) / nblock + bsize - 1) / bsize * bsize;

  ////////////////////////////////
  dim3 grid(nblock), block(bsize);

  /////////////////////////////////
  buf.InitTexture(nblock, 1);
  vector_dotproduct_kernel<<<grid, block>>>(vector1.data(), vector2.data(), len,
                                            blen, buf.data());
  ProgramCU::CheckErrorCUDA("ComputeVectorDot");

  float data[nblock];
  buf.CopyToHost(data);

  double result = 0;
  for (unsigned int i = 0; i < nblock; ++i) result += data[i];
  return result;
}

__global__ void vector_weighted_norm_kernel(const float* vec, const float* w,
                                            int len, int blen, float* result) {
  __shared__ float value[256];
  int bstart = blen * blockIdx.x;
  int start = bstart + threadIdx.x;
  int end = min(len, bstart + blen);

  float v = 0;
  for (int i = start; i < end; i += blockDim.x) v += (vec[i] * w[i] * vec[i]);
  value[threadIdx.x] = v;

  // reduce to the first two values
  WARP_REDUCTION_256(value);

  // write back
  if (threadIdx.x == 0) result[blockIdx.x] = (value[0] + value[1]);
}

double ProgramCU::ComputeVectorNormW(CuTexImage& vector, CuTexImage& weight,
                                     CuTexImage& buf) {
  if (weight.IsValid()) {
    const unsigned int nblock = REDUCTION_NBLOCK;
    unsigned int bsize = 256;
    int len = vector.GetLength();
    int blen = ((len + nblock - 1) / nblock + bsize - 1) / bsize * bsize;

    ////////////////////////////////
    dim3 grid(nblock), block(bsize);

    /////////////////////////////////
    buf.InitTexture(nblock, 1);

    vector_weighted_norm_kernel<<<grid, block>>>(vector.data(), weight.data(),
                                                 len, blen, buf.data());

    ProgramCU::CheckErrorCUDA("ComputeVectorNormW");

    float data[nblock];
    buf.CopyToHost(data);

    double result = 0;
    for (unsigned int i = 0; i < nblock; ++i) result += data[i];
    return result;
  } else {
    return ComputeVectorNorm(vector, buf);
  }
}
// given vector x, y, and a weight a
// return a * x + y
__global__ void saxpy_kernel(const float a, const float* x, const float* y,
                             float* result, unsigned int len) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < len) result[idx] = a * x[idx] + y[idx];
}

__global__ void saxpy_kernel_large(const float a, const float* x,
                                   const float* y, float* result,
                                   unsigned int len, unsigned int rowsz) {
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
  if (idx < len) result[idx] = a * x[idx] + y[idx];
}

void ProgramCU::ComputeSAXPY(float a, CuTexImage& texX, CuTexImage& texY,
                             CuTexImage& result) {
  unsigned int len = result.GetLength();
  unsigned int bsize = 128;
  unsigned int nblock = (len + bsize - 1) / bsize;
  if (nblock > MAX_BLOCKLEN) {
    unsigned int bw, bh;
    GetBlockConfiguration(nblock, bw, bh);
    dim3 grid(bw, bh), block(bsize);
    saxpy_kernel_large<<<grid, block>>>(a, texX.data(), texY.data(),
                                        result.data(), len, bw * bsize);
  } else {
    dim3 grid(nblock), block(bsize);
    saxpy_kernel<<<grid, block>>>(a, texX.data(), texY.data(), result.data(),
                                  len);
  }
  ProgramCU::CheckErrorCUDA("ComputeSAXPY");
}

__global__ void sxypz_kernel_large(float a, const float* x, const float* y,
                                   const float* z, float* result,
                                   unsigned int len, unsigned int rowsz) {
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
  if (idx < len) result[idx] = a * x[idx] * y[idx] + z[idx];
}

void ProgramCU::ComputeSXYPZ(float a, CuTexImage& texX, CuTexImage& texY,
                             CuTexImage& texZ, CuTexImage& result) {
  if (texX.IsValid()) {
    unsigned int len = texX.GetLength();
    unsigned int bsize = 128;
    unsigned int nblock = (len + bsize - 1) / bsize;
    unsigned int bw, bh;
    GetBlockConfiguration(nblock, bw, bh);
    dim3 grid(bw, bh), block(bsize);
    sxypz_kernel_large<<<grid, block>>>(a, texX.data(), texY.data(),
                                        texZ.data(), result.data(), len,
                                        bw * bsize);
  } else {
    ComputeSAXPY(a, texY, texZ, result);
  }
}

__global__ void vxy_kernel(const float* x, float* y, float* result,
                           unsigned int len) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < len) result[idx] = x[idx] * y[idx];
}

__global__ void vxy_kernel_large(const float* x, float* y, float* result,
                                 unsigned int len, unsigned int rowsz) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x + rowsz * blockIdx.y;
  if (idx < len) result[idx] = x[idx] * y[idx];
}

void ProgramCU::ComputeVXY(CuTexImage& texX, CuTexImage& texY,
                           CuTexImage& result, unsigned int part,
                           unsigned int skip) {
  unsigned int len = part ? part : texX.GetLength();
  unsigned int bsize = 128;
  unsigned int nblock = (len + bsize - 1) / bsize;
  if (nblock > MAX_BLOCKLEN) {
    unsigned int bw, bh;
    GetBlockConfiguration(nblock, bw, bh);
    dim3 grid(bw, bh), block(bsize);
    vxy_kernel_large<<<grid, block>>>(texX.data() + skip, texY.data() + skip,
                                      result.data() + skip, len, bsize * bw);
  } else {
    dim3 grid(nblock), block(bsize);
    vxy_kernel<<<grid, block>>>(texX.data() + skip, texY.data() + skip,
                                result.data() + skip, len);
  }
  ProgramCU::CheckErrorCUDA("ComputeVXY");
}

__global__ void sqrt_kernel_large(float* x, unsigned int len,
                                  unsigned int rowsz) {
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
  if (idx < len) x[idx] = sqrt(x[idx]);
}

void ProgramCU::ComputeSQRT(CuTexImage& tex) {
  unsigned int len = tex.GetLength();
  unsigned int bsize = 128;
  unsigned int nblock = (len + bsize - 1) / bsize;
  unsigned int bw, bh;
  GetBlockConfiguration(nblock, bw, bh);
  dim3 grid(bw, bh), block(bsize);
  sqrt_kernel_large<<<grid, block>>>(tex.data(), len, bw * bsize);
  ProgramCU::CheckErrorCUDA("ComputeSQRT");
}

__global__ void rsqrt_kernel_large(float* x, unsigned int len,
                                   unsigned int rowsz) {
  unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
  if (idx < len) x[idx] = x[idx] > 0 ? rsqrt(x[idx]) : 0;
}

void ProgramCU::ComputeRSQRT(CuTexImage& tex) {
  unsigned int len = tex.GetLength();
  unsigned int bsize = 128;
  unsigned int nblock = (len + bsize - 1) / bsize;
  unsigned int bw, bh;
  GetBlockConfiguration(nblock, bw, bh);
  dim3 grid(bw, bh), block(bsize);
  rsqrt_kernel_large<<<grid, block>>>(tex.data(), len, bw * bsize);

  ProgramCU::CheckErrorCUDA("ComputeRSQRT");
}

__global__ void sax_kernel(const float a, const float* x, float* result,
                           unsigned int len) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < len) result[idx] = a * x[idx];
}

__global__ void sax_kernel_large(const float a, const float* x, float* result,
                                 unsigned int len, unsigned int rowsz) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * rowsz;
  if (idx < len) result[idx] = a * x[idx];
}

void ProgramCU::ComputeSAX(float a, CuTexImage& texX, CuTexImage& result) {
  unsigned int len = texX.GetLength();
  unsigned int bsize = 128;
  unsigned int nblock = (len + bsize - 1) / bsize;

  if (nblock > MAX_BLOCKLEN) {
    unsigned int bw, bh;
    GetBlockConfiguration(nblock, bw, bh);
    dim3 grid(bw, bh), block(bsize);
    sax_kernel_large<<<grid, block>>>(a, texX.data(), result.data(), len,
                                      bw * bsize);
  } else {
    dim3 grid(nblock), block(bsize);
    sax_kernel<<<grid, block>>>(a, texX.data(), result.data(), len);
  }
  ProgramCU::CheckErrorCUDA("ComputeSAX");
}

#define JACOBIAN_FRT_KWIDTH 64

texture<float4, 1, hipReadModeElementType> tex_jacobian_cam;
texture<float4, 1, hipReadModeElementType> tex_jacobian_pts;
texture<int2, 1, hipReadModeElementType> tex_jacobian_idx;
texture<float2, 1, hipReadModeElementType> tex_jacobian_meas;
texture<float4, 1, hipReadModeElementType> tex_jacobian_sj;
texture<int, 1, hipReadModeElementType> tex_jacobian_shuffle;

#ifndef PBA_DISABLE_CONST_CAMERA
#define JACOBIAN_SET_JC_BEGIN if (r3.w == 0.0f) {
#define JFRT_SET_JC_END                       \
  }                                           \
  else {                                      \
    jc[jc_pos] = make_float4(0, 0, 0, 0);     \
    jc[jc_pos + 1] = make_float4(0, 0, 0, 0); \
    jc[jc_pos + 2] = make_float4(0, 0, 0, 0); \
    jc[jc_pos + 3] = make_float4(0, 0, 0, 0); \
  }
#define JACOBIAN_SET_JC_END \
  }                         \
  else {                    \
    jxc[0] = 0;             \
    jxc[1] = 0;             \
    jxc[2] = 0;             \
    jxc[3] = 0;             \
    jxc[4] = 0;             \
    jxc[5] = 0;             \
    jxc[6] = 0;             \
    jxc[7] = 0;             \
    jyc[0] = 0;             \
    jyc[1] = 0;             \
    jyc[2] = 0;             \
    jyc[3] = 0;             \
    jyc[4] = 0;             \
    jyc[5] = 0;             \
    jyc[6] = 0;             \
    jyc[7] = 0;             \
  }
#else
#define JACOBIAN_SET_JC_BEGIN
#define JFRT_SET_JC_END
#define JACOBIAN_SET_JC_END
#endif

// projection model ei = K(RX + T)  - (1 + r * m^2) * m
template <bool md, bool pd, bool scaling, bool shuffle>
__global__ void jacobian_frt_kernel(float4* jc, float4* jp, int nproj, int ptx,
                                    int rowsz, float jic) {
  ////////////////////////////////
  int tidx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * rowsz;

  if (tidx >= nproj) return;
  int2 proj = tex1Dfetch(tex_jacobian_idx, tidx);
  int camera_pos = proj.x << 1;

  __shared__ float rr_data[JACOBIAN_FRT_KWIDTH * 9];
  float* r = rr_data + IMUL(9, threadIdx.x);
  float4 ft = tex1Dfetch(tex_jacobian_cam, camera_pos);
  float4 r1 = tex1Dfetch(tex_jacobian_cam, camera_pos + 1);
  r[0] = r1.x;
  r[1] = r1.y;
  r[2] = r1.z;
  r[3] = r1.w;
  float4 r2 = tex1Dfetch(tex_jacobian_cam, camera_pos + 2);
  r[4] = r2.x;
  r[5] = r2.y;
  r[6] = r2.z;
  r[7] = r2.w;
  float4 r3 = tex1Dfetch(tex_jacobian_cam, camera_pos + 3);
  r[8] = r3.x;

  float4 temp = tex1Dfetch(tex_jacobian_pts, proj.y);
  float m[3];
  m[0] = temp.x;
  m[1] = temp.y;
  m[2] = temp.z;

  float x0 = r[0] * m[0] + r[1] * m[1] + r[2] * m[2];
  float y0 = r[3] * m[0] + r[4] * m[1] + r[5] * m[2];
  float z0 = r[6] * m[0] + r[7] * m[1] + r[8] * m[2];
  float f_p2 = FDIV(ft.x, z0 + ft.w);
  float p0_p2 = FDIV(x0 + ft.y, z0 + ft.w);
  float p1_p2 = FDIV(y0 + ft.z, z0 + ft.w);

  // dp/dx = [f/p2  0      -f*p0/p2/p2]
  //        [0     f/p2   -f*p1/p2/p2]
  // dx/dw = [ 0  z -y]
  //        [-z  0  x]
  //        [ y -x  0]
  // R(dw) (x y z)' = (0 -z y)' dw0 + (z 0  -x)'dw1 + (-y x 0)'dw2
  int jc_pos;
  if (shuffle) {
    jc_pos = tex1Dfetch(tex_jacobian_shuffle, tidx) << 2;
  } else {
    jc_pos = tidx << 2;
  }

  if (pd) {
    float rr1 = r3.y * p0_p2 * p0_p2;
    float rr2 = r3.y * p1_p2 * p1_p2;
    float f_p2_x = f_p2 * (1.0 + 3.0 * rr1 + rr2);
    float f_p2_y = f_p2 * (1.0 + 3.0 * rr2 + rr1);
    if (scaling == false) {
      if (jc) {
        JACOBIAN_SET_JC_BEGIN
        // float jic = (r3.w != 1.0f && r3.w != 2.0f) ? 1.0f : 0.0f;
        // float jec = (r3.w != 1.0f && r3.w != 3.0f) ? 1.0f : 0.0f;
        float jfc = jic * (1 + rr1 + rr2);
        float ft_x_pn = jic * ft.x * (p0_p2 * p0_p2 + p1_p2 * p1_p2);
        jc[jc_pos] = make_float4(p0_p2 * jfc, f_p2_x, 0, -f_p2_x * p0_p2);
        jc[jc_pos + 1] =
            make_float4(-f_p2_x * p0_p2 * y0, f_p2_x * (z0 + x0 * p0_p2),
                        -f_p2_x * y0, ft_x_pn * p0_p2);
        jc[jc_pos + 2] = make_float4(p1_p2 * jfc, 0, f_p2_y, -f_p2 * p1_p2);
        jc[jc_pos + 3] =
            make_float4(-f_p2_y * (z0 + y0 * p1_p2), f_p2_y * x0 * p1_p2,
                        f_p2_y * x0, ft_x_pn * p1_p2);
        JFRT_SET_JC_END
      }
      ////////////////////
      jp[(tidx << 1)] = make_float4(f_p2_x * (r[0] - r[6] * p0_p2),
                                    f_p2_x * (r[1] - r[7] * p0_p2),
                                    f_p2_x * (r[2] - r[8] * p0_p2), 0);
      jp[(tidx << 1) + 1] = make_float4(f_p2_y * (r[3] - r[6] * p1_p2),
                                        f_p2_y * (r[4] - r[7] * p1_p2),
                                        f_p2_y * (r[5] - r[8] * p1_p2), 0);
    } else {
      ////////////////////
      if (jc) {
        JACOBIAN_SET_JC_BEGIN
        float jfc = jic * (1 + rr1 + rr2);
        float ft_x_pn = jic * ft.x * (p0_p2 * p0_p2 + p1_p2 * p1_p2);
        float4 sc1 = tex1Dfetch(tex_jacobian_sj, proj.x);
        jc[jc_pos] = make_float4(p0_p2 * jfc * sc1.x, f_p2_x * sc1.y, 0,
                                 -f_p2_x * p0_p2 * sc1.w);
        jc[jc_pos + 2] = make_float4(p1_p2 * jfc * sc1.x, 0, f_p2_y * sc1.z,
                                     -f_p2_y * p1_p2 * sc1.w);

        float4 sc2 = tex1Dfetch(tex_jacobian_sj, proj.x + 1);
        jc[jc_pos + 1] = make_float4(
            -sc2.x * f_p2_x * p0_p2 * y0, sc2.y * f_p2_x * (z0 + x0 * p0_p2),
            -sc2.z * f_p2_x * y0, ft_x_pn * p0_p2 * sc2.w);
        jc[jc_pos + 3] = make_float4(
            -sc2.x * f_p2_y * (z0 + y0 * p1_p2), sc2.y * f_p2_y * x0 * p1_p2,
            sc2.z * f_p2_y * x0, ft_x_pn * p1_p2 * sc2.w);
        JFRT_SET_JC_END
      }

      float4 sc3 = tex1Dfetch(tex_jacobian_sj, proj.y + ptx);
      jp[(tidx << 1)] = make_float4(sc3.x * f_p2_x * (r[0] - r[6] * p0_p2),
                                    sc3.y * f_p2_x * (r[1] - r[7] * p0_p2),
                                    sc3.z * f_p2_x * (r[2] - r[8] * p0_p2), 0);
      jp[(tidx << 1) + 1] =
          make_float4(sc3.x * f_p2_y * (r[3] - r[6] * p1_p2),
                      sc3.y * f_p2_y * (r[4] - r[7] * p1_p2),
                      sc3.z * f_p2_y * (r[5] - r[8] * p1_p2), 0);
    }
  } else if (md) {
    if (scaling == false) {
      if (jc) {
        JACOBIAN_SET_JC_BEGIN
        float2 ms = tex1Dfetch(tex_jacobian_meas, tidx);
        float msn = (ms.x * ms.x + ms.y * ms.y) * jic;
        jc[jc_pos] = make_float4(p0_p2 * jic, f_p2, 0, -f_p2 * p0_p2);
        jc[jc_pos + 1] =
            make_float4(-f_p2 * p0_p2 * y0, f_p2 * (z0 + x0 * p0_p2),
                        -f_p2 * y0, -ms.x * msn);
        jc[jc_pos + 2] = make_float4(p1_p2 * jic, 0, f_p2, -f_p2 * p1_p2);
        jc[jc_pos + 3] = make_float4(-f_p2 * (z0 + y0 * p1_p2),
                                     f_p2 * x0 * p1_p2, f_p2 * x0, -ms.y * msn);
        JFRT_SET_JC_END
      }
      ////////////////////
      jp[(tidx << 1)] = make_float4(f_p2 * (r[0] - r[6] * p0_p2),
                                    f_p2 * (r[1] - r[7] * p0_p2),
                                    f_p2 * (r[2] - r[8] * p0_p2), 0);
      jp[(tidx << 1) + 1] = make_float4(f_p2 * (r[3] - r[6] * p1_p2),
                                        f_p2 * (r[4] - r[7] * p1_p2),
                                        f_p2 * (r[5] - r[8] * p1_p2), 0);
    } else {
      if (jc) {
        JACOBIAN_SET_JC_BEGIN
        float4 sc1 = tex1Dfetch(tex_jacobian_sj, proj.x);
        jc[jc_pos] = make_float4(p0_p2 * jic * sc1.x, f_p2 * sc1.y, 0,
                                 -f_p2 * p0_p2 * sc1.w);
        jc[jc_pos + 2] = make_float4(p1_p2 * jic * sc1.x, 0, f_p2 * sc1.z,
                                     -f_p2 * p1_p2 * sc1.w);

        float4 sc2 = tex1Dfetch(tex_jacobian_sj, proj.x + 1);
        float2 ms = tex1Dfetch(tex_jacobian_meas, tidx);
        float msn = (ms.x * ms.x + ms.y * ms.y) * jic;
        jc[jc_pos + 1] = make_float4(-sc2.x * f_p2 * p0_p2 * y0,
                                     sc2.y * f_p2 * (z0 + x0 * p0_p2),
                                     -sc2.z * f_p2 * y0, -msn * ms.x * sc2.w);
        jc[jc_pos + 3] = make_float4(-sc2.x * f_p2 * (z0 + y0 * p1_p2),
                                     sc2.y * f_p2 * x0 * p1_p2,
                                     sc2.z * f_p2 * x0, -msn * ms.y * sc2.w);
        JFRT_SET_JC_END
      }
      float4 sc3 = tex1Dfetch(tex_jacobian_sj, proj.y + ptx);
      jp[(tidx << 1)] = make_float4(sc3.x * f_p2 * (r[0] - r[6] * p0_p2),
                                    sc3.y * f_p2 * (r[1] - r[7] * p0_p2),
                                    sc3.z * f_p2 * (r[2] - r[8] * p0_p2), 0);
      jp[(tidx << 1) + 1] =
          make_float4(sc3.x * f_p2 * (r[3] - r[6] * p1_p2),
                      sc3.y * f_p2 * (r[4] - r[7] * p1_p2),
                      sc3.z * f_p2 * (r[5] - r[8] * p1_p2), 0);
    }

  } else {
    if (scaling == false) {
      if (jc) {
        JACOBIAN_SET_JC_BEGIN
        jc[jc_pos] = make_float4(p0_p2 * jic, f_p2, 0, -f_p2 * p0_p2);
        jc[jc_pos + 1] = make_float4(-f_p2 * p0_p2 * y0,
                                     f_p2 * (z0 + x0 * p0_p2), -f_p2 * y0, 0);
        jc[jc_pos + 2] = make_float4(p1_p2 * jic, 0, f_p2, -f_p2 * p1_p2);
        jc[jc_pos + 3] = make_float4(-f_p2 * (z0 + y0 * p1_p2),
                                     f_p2 * x0 * p1_p2, f_p2 * x0, 0);
        JFRT_SET_JC_END
      }
      ////////////////////
      jp[(tidx << 1)] = make_float4(f_p2 * (r[0] - r[6] * p0_p2),
                                    f_p2 * (r[1] - r[7] * p0_p2),
                                    f_p2 * (r[2] - r[8] * p0_p2), 0);
      jp[(tidx << 1) + 1] = make_float4(f_p2 * (r[3] - r[6] * p1_p2),
                                        f_p2 * (r[4] - r[7] * p1_p2),
                                        f_p2 * (r[5] - r[8] * p1_p2), 0);
    } else {
      if (jc) {
        JACOBIAN_SET_JC_BEGIN
        float4 sc1 = tex1Dfetch(tex_jacobian_sj, proj.x);
        jc[jc_pos] = make_float4(p0_p2 * jic * sc1.x, f_p2 * sc1.y, 0,
                                 -f_p2 * p0_p2 * sc1.w);
        jc[jc_pos + 2] = make_float4(p1_p2 * jic * sc1.x, 0, f_p2 * sc1.z,
                                     -f_p2 * p1_p2 * sc1.w);
        float4 sc2 = tex1Dfetch(tex_jacobian_sj, proj.x + 1);
        jc[jc_pos + 1] = make_float4(-sc2.x * f_p2 * p0_p2 * y0,
                                     sc2.y * f_p2 * (z0 + x0 * p0_p2),
                                     -sc2.z * f_p2 * y0, 0);
        jc[jc_pos + 3] =
            make_float4(-sc2.x * f_p2 * (z0 + y0 * p1_p2),
                        sc2.y * f_p2 * x0 * p1_p2, sc2.z * f_p2 * x0, 0);
        JFRT_SET_JC_END
      }

      float4 sc3 = tex1Dfetch(tex_jacobian_sj, proj.y + ptx);
      jp[(tidx << 1)] = make_float4(sc3.x * f_p2 * (r[0] - r[6] * p0_p2),
                                    sc3.y * f_p2 * (r[1] - r[7] * p0_p2),
                                    sc3.z * f_p2 * (r[2] - r[8] * p0_p2), 0);
      jp[(tidx << 1) + 1] =
          make_float4(sc3.x * f_p2 * (r[3] - r[6] * p1_p2),
                      sc3.y * f_p2 * (r[4] - r[7] * p1_p2),
                      sc3.z * f_p2 * (r[5] - r[8] * p1_p2), 0);
    }
  }
}

/////////////////////////////////
void ProgramCU::ComputeJacobian(CuTexImage& camera, CuTexImage& point,
                                CuTexImage& jc, CuTexImage& jp,
                                CuTexImage& proj_map, CuTexImage& sj,
                                CuTexImage& meas, CuTexImage& cmlist,
                                bool intrinsic_fixed, int radial_distortion,
                                bool shuffle) {
  float jfc = intrinsic_fixed ? 0.0f : 1.0f;
  unsigned int len = proj_map.GetImgWidth();
  unsigned int bsize = JACOBIAN_FRT_KWIDTH;
  unsigned int nblock = (len + bsize - 1) / bsize;
  unsigned int bw, bh;
  GetBlockConfiguration(nblock, bw, bh);
  dim3 grid(bw, bh), block(bsize);

  camera.BindTexture(tex_jacobian_cam);
  point.BindTexture(tex_jacobian_pts);
  proj_map.BindTexture(tex_jacobian_idx);

  if (!jc.IsValid()) shuffle = false;
  if (shuffle) cmlist.BindTexture(tex_jacobian_shuffle);
  if (sj.IsValid()) sj.BindTexture(tex_jacobian_sj);

  if (radial_distortion == -1) {
    meas.BindTexture(tex_jacobian_meas);
    if (sj.IsValid()) {
      if (shuffle)
        jacobian_frt_kernel<true, false, true, true><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
      else
        jacobian_frt_kernel<true, false, true, false><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
    } else {
      if (shuffle)
        jacobian_frt_kernel<true, false, false, true><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
      else
        jacobian_frt_kernel<true, false, false, false><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
    }
  } else if (radial_distortion) {
    if (sj.IsValid()) {
      if (shuffle)
        jacobian_frt_kernel<false, true, true, true><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
      else
        jacobian_frt_kernel<false, true, true, false><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
    } else {
      if (shuffle)
        jacobian_frt_kernel<false, true, false, true><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
      else
        jacobian_frt_kernel<false, true, false, false><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
    }
  } else {
    if (sj.IsValid()) {
      if (shuffle)
        jacobian_frt_kernel<false, false, true, true><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
      else
        jacobian_frt_kernel<false, false, true, false><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
    } else {
      if (shuffle)
        jacobian_frt_kernel<false, false, false, true><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
      else
        jacobian_frt_kernel<false, false, false, false><<<grid, block>>>(
            (float4*)jc.data(), (float4*)jp.data(), len,
            camera.GetImgWidth() * 2, bw * bsize, jfc);
    }
  }

  ProgramCU::CheckErrorCUDA("ComputeJacobian");
}

texture<float4, 1, hipReadModeElementType> tex_compact_cam;
__global__ void uncompress_frt_kernel(int ncam, float4* ucam) {
  int tidx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
  if (tidx >= ncam) return;
  int fetch_index = tidx << 1;
  int write_index = IMUL(tidx, 4);
  float4 temp1 = tex1Dfetch(tex_compact_cam, fetch_index);
  ucam[write_index] = temp1;

  float4 temp2 = tex1Dfetch(tex_compact_cam, fetch_index + 1);
  float rx = temp2.x;
  float ry = temp2.y;
  float rz = temp2.z;
  float rx_rx = rx * rx;
  float ry_ry = ry * ry;
  float rz_rz = rz * rz;
  float aa = sqrt(rx_rx + ry_ry + rz_rz);
  float caa, saa;
  sincosf(aa, &saa, &caa);
  float ct = aa == 0.0 ? 0.5 : FDIV2(1.0 - caa, aa * aa);
  float st = aa == 0.0 ? 1 : FDIV2(saa, aa);
  float rz_st = rz * st;
  float rx_st = rx * st;
  float ry_st = ry * st;
  float ry_ry_ct = ry_ry * ct;
  float rx_rx_ct = rx_rx * ct;
  float rz_rz_ct = rz_rz * ct;
  float rx_ry_ct = rx * ry * ct;
  float rz_rx_ct = rz * rx * ct;
  float ry_rz_ct = ry * rz * ct;

  ////////////////////////////////////////////////////////////
  ucam[write_index + 1] =
      make_float4((1.0 - (ry_ry_ct + rz_rz_ct)), (rx_ry_ct - rz_st),
                  (rz_rx_ct + ry_st), (rx_ry_ct + rz_st));

  ucam[write_index + 2] =
      make_float4((1.0 - (rz_rz_ct + rx_rx_ct)), (ry_rz_ct - rx_st),
                  (rz_rx_ct - ry_st), (ry_rz_ct + rx_st));

  ucam[write_index + 3] =
      make_float4((1.0 - (rx_rx_ct + ry_ry_ct)), temp2.w, 0, 0);
}

void ProgramCU::UncompressCamera(int ncam, CuTexImage& camera,
                                 CuTexImage& result) {
  unsigned int len = ncam;
  unsigned int bsize = 64;
  unsigned int nblock = (len + bsize - 1) / bsize;
  dim3 grid(nblock);
  dim3 block(bsize);
  camera.BindTexture(tex_compact_cam);
  uncompress_frt_kernel<<<grid, block>>>(len, (float4*)result.data());
  CheckErrorCUDA("UncompressCamera");
}

texture<float4, 1, hipReadModeElementType> tex_uncompressed_cam;

__global__ void compress_frt_kernel(int ncam, float4* zcam) {
  int tidx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
  if (tidx >= ncam) return;
  int fetch_index = tidx << 2;
  int write_index = tidx << 1;
  float4 temp1 = tex1Dfetch(tex_compact_cam, fetch_index);
  zcam[write_index] = temp1;

  float4 r1 = tex1Dfetch(tex_compact_cam, fetch_index + 1);
  float4 r2 = tex1Dfetch(tex_compact_cam, fetch_index + 2);
  float4 r3 = tex1Dfetch(tex_compact_cam, fetch_index + 3);

  float a = (r1.x + r2.x + r3.x - 1.0) / 2.0;
  if (a >= 1.0) {
    zcam[write_index + 1] = make_float4(0, 0, 0, 0);
  } else {
    float aa = acos(a), b = 0.5 * aa * rsqrt(1 - a * a);
    zcam[write_index + 1] = make_float4(b * (r2.w - r2.y), b * (r1.z - r2.z),
                                        b * (r1.w - r1.y), r3.y);
  }
}

void ProgramCU::CompressCamera(int ncam, CuTexImage& camera0,
                               CuTexImage& result) {
  unsigned int len = ncam;
  unsigned int bsize = 64;
  unsigned int nblock = (len + bsize - 1) / bsize;
  dim3 grid(nblock), block(bsize);
  camera0.BindTexture(tex_uncompressed_cam);
  compress_frt_kernel<<<grid, block>>>(ncam, (float4*)result.data());
  CheckErrorCUDA("CompressCamera");
}

__device__ inline void uncompress_rodrigues_rotation(float rx, float ry,
                                                     float rz, float* r) {
  float rx_rx = rx * rx;
  float ry_ry = ry * ry;
  float rz_rz = rz * rz;
  float aa = sqrt(rx_rx + ry_ry + rz_rz);
  float caa, saa;
  sincosf(aa, &saa, &caa);
  float ct = aa == 0.0 ? 0.5 : FDIV2(1.0 - caa, aa * aa);
  float st = aa == 0.0 ? 1 : FDIV2(saa, aa);
  float rz_st = rz * st;
  float rx_st = rx * st;
  float ry_st = ry * st;
  float ry_ry_ct = ry_ry * ct;
  float rx_rx_ct = rx_rx * ct;
  float rz_rz_ct = rz_rz * ct;
  float rx_ry_ct = rx * ry * ct;
  float rz_rx_ct = rz * rx * ct;
  float ry_rz_ct = ry * rz * ct;
  r[0] = (1.0 - (ry_ry_ct + rz_rz_ct));
  r[1] = (rx_ry_ct - rz_st);
  r[2] = (rz_rx_ct + ry_st);
  r[3] = (rx_ry_ct + rz_st);
  r[4] = (1.0 - (rz_rz_ct + rx_rx_ct));
  r[5] = (ry_rz_ct - rx_st);
  r[6] = (rz_rx_ct - ry_st);
  r[7] = (ry_rz_ct + rx_st);
  r[8] = (1.0 - (rx_rx_ct + ry_ry_ct));
}

texture<float4, 1, hipReadModeElementType> tex_update_cam;
texture<float4, 1, hipReadModeElementType> tex_update_cam_delta;

__global__ void update_camera_kernel(int ncam, float4* newcam) {
  int tidx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
  if (tidx >= ncam) return;
  int index0 = tidx << 2;
  int index1 = tidx << 1;
  {
    float4 c1 = tex1Dfetch(tex_update_cam, index0);
    float4 d1 = tex1Dfetch(tex_update_cam_delta, index1);
    float4 c2 = make_float4(max(c1.x + d1.x, 1e-10f), c1.y + d1.y, c1.z + d1.z,
                            c1.w + d1.w);
    newcam[index0] = c2;
  }
  {
    float r[9], dr[9];  //, nr[9];
    float4 r1 = tex1Dfetch(tex_update_cam, index0 + 1);
    r[0] = r1.x;
    r[1] = r1.y;
    r[2] = r1.z;
    r[3] = r1.w;
    float4 r2 = tex1Dfetch(tex_update_cam, index0 + 2);
    r[4] = r2.x;
    r[5] = r2.y;
    r[6] = r2.z;
    r[7] = r2.w;
    float4 r3 = tex1Dfetch(tex_update_cam, index0 + 3);
    r[8] = r3.x;

    float4 dd = tex1Dfetch(tex_update_cam_delta, index1 + 1);
    uncompress_rodrigues_rotation(dd.x, dd.y, dd.z, dr);

    ///////////////////////////////////////////////
    newcam[index0 + 1] =
        make_float4(dr[0] * r[0] + dr[1] * r[3] + dr[2] * r[6],
                    dr[0] * r[1] + dr[1] * r[4] + dr[2] * r[7],
                    dr[0] * r[2] + dr[1] * r[5] + dr[2] * r[8],
                    dr[3] * r[0] + dr[4] * r[3] + dr[5] * r[6]);
    newcam[index0 + 2] =
        make_float4(dr[3] * r[1] + dr[4] * r[4] + dr[5] * r[7],
                    dr[3] * r[2] + dr[4] * r[5] + dr[5] * r[8],
                    dr[6] * r[0] + dr[7] * r[3] + dr[8] * r[6],
                    dr[6] * r[1] + dr[7] * r[4] + dr[8] * r[7]);
    newcam[index0 + 3] = make_float4(dr[6] * r[2] + dr[7] * r[5] + dr[8] * r[8],
                                     r3.y + dd.w, r3.z, r3.w);
  }
}

void ProgramCU::UpdateCameraPoint(int ncam, CuTexImage& camera,
                                  CuTexImage& point, CuTexImage& delta,
                                  CuTexImage& new_camera, CuTexImage& new_point,
                                  int mode) {
  if (mode != 2) {
    unsigned int len = ncam;
    unsigned int bsize = 64;
    unsigned int nblock = (len + bsize - 1) / bsize;
    dim3 grid(nblock), block(bsize);
    camera.BindTexture(tex_update_cam);
    delta.BindTexture(tex_update_cam_delta);
    update_camera_kernel<<<grid, block>>>(len, (float4*)new_camera.data());
    CheckErrorCUDA("UpdateCamera");
  }

  // update the points
  if (mode != 1) {
    CuTexImage dp;
    dp.SetTexture(delta.data() + 8 * ncam, point.GetLength());
    ComputeSAXPY(1.0f, dp, point, new_point);
    CheckErrorCUDA("UpdatePoint");
  }
}

#define PROJECTION_FRT_KWIDTH 64

texture<float4, 1, hipReadModeElementType> tex_projection_cam;
texture<int2, 1, hipReadModeElementType> tex_projection_idx;
texture<float4, 1, hipReadModeElementType> tex_projection_pts;
texture<float2, 1, hipReadModeElementType> tex_projection_mea;

// run 32/64/128 projections in a block
template <bool md, bool pd>
__global__ void projection_frt_kernel(int nproj, int rowsz, float2* pj) {
  ////////////////////////////////
  int tidx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
  if (tidx >= nproj) return;
  float f, m[3], t[3];  // r[9],
  __shared__ float rr_data[PROJECTION_FRT_KWIDTH * 9];
  float* r = rr_data + IMUL(9, threadIdx.x);
  int2 proj = tex1Dfetch(tex_projection_idx, tidx);
  int cpos = proj.x << 1;
  float4 ft = tex1Dfetch(tex_projection_cam, cpos);
  f = ft.x;
  t[0] = ft.y;
  t[1] = ft.z;
  t[2] = ft.w;
  float4 r1 = tex1Dfetch(tex_projection_cam, cpos + 1);
  r[0] = r1.x;
  r[1] = r1.y;
  r[2] = r1.z;
  r[3] = r1.w;
  float4 r2 = tex1Dfetch(tex_projection_cam, cpos + 2);
  r[4] = r2.x;
  r[5] = r2.y;
  r[6] = r2.z;
  r[7] = r2.w;
  float4 r3 = tex1Dfetch(tex_projection_cam, cpos + 3);
  r[8] = r3.x;

  float4 temp = tex1Dfetch(tex_projection_pts, proj.y);
  m[0] = temp.x;
  m[1] = temp.y;
  m[2] = temp.z;

  float p0 = r[0] * m[0] + r[1] * m[1] + r[2] * m[2] + t[0];
  float p1 = r[3] * m[0] + r[4] * m[1] + r[5] * m[2] + t[1];
  float p2 = r[6] * m[0] + r[7] * m[1] + r[8] * m[2] + t[2];

  if (pd) {
    float rr = 1.0 + r3.y * (p0 * p0 + p1 * p1) / (p2 * p2);
    float f_p2 = FDIV2(f * rr, p2);
    float2 ms = tex1Dfetch(tex_projection_mea, tidx);
    pj[tidx] = make_float2(ms.x - p0 * f_p2, ms.y - p1 * f_p2);
  } else if (md) {
    float f_p2 = FDIV2(f, p2);
    float2 ms = tex1Dfetch(tex_projection_mea, tidx);
    float rd = 1.0 + r3.y * (ms.x * ms.x + ms.y * ms.y);
    pj[tidx] = make_float2(ms.x * rd - p0 * f_p2, ms.y * rd - p1 * f_p2);
  } else {
    float f_p2 = FDIV2(f, p2);
    float2 ms = tex1Dfetch(tex_projection_mea, tidx);
    pj[tidx] = make_float2(ms.x - p0 * f_p2, ms.y - p1 * f_p2);
  }
}

void ProgramCU::ComputeProjection(CuTexImage& camera, CuTexImage& point,
                                  CuTexImage& meas, CuTexImage& proj_map,
                                  CuTexImage& proj, int radial) {
  unsigned int len = proj_map.GetImgWidth();
  unsigned int bsize = PROJECTION_FRT_KWIDTH;
  unsigned int nblock = (len + bsize - 1) / bsize;
  camera.BindTexture(tex_projection_cam);
  point.BindTexture(tex_projection_pts);
  proj_map.BindTexture(tex_projection_idx);
  unsigned int bw, bh;
  GetBlockConfiguration(nblock, bw, bh);
  dim3 grid(bw, bh), block(bsize);
  meas.BindTexture(tex_projection_mea);
  if (radial == -1)
    projection_frt_kernel<true, false><<<grid, block>>>(len, bw * bsize,
                                                        (float2*)proj.data());
  else if (radial)
    projection_frt_kernel<false, true><<<grid, block>>>(len, bw * bsize,
                                                        (float2*)proj.data());
  else
    projection_frt_kernel<false, false><<<grid, block>>>(len, bw * bsize,
                                                         (float2*)proj.data());
  CheckErrorCUDA("ComputeProjection");
}

template <bool md, bool pd>
__global__ void projectionx_frt_kernel(int nproj, int rowsz, float2* pj) {
  ////////////////////////////////
  int tidx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
  if (tidx >= nproj) return;
  float f, m[3], t[3];  // r[9],
  __shared__ float rr_data[PROJECTION_FRT_KWIDTH * 9];
  float* r = rr_data + IMUL(9, threadIdx.x);
  int2 proj = tex1Dfetch(tex_projection_idx, tidx);
  int cpos = proj.x << 1;
  float4 ft = tex1Dfetch(tex_projection_cam, cpos);
  f = ft.x;
  t[0] = ft.y;
  t[1] = ft.z;
  t[2] = ft.w;
  float4 r1 = tex1Dfetch(tex_projection_cam, cpos + 1);
  r[0] = r1.x;
  r[1] = r1.y;
  r[2] = r1.z;
  r[3] = r1.w;
  float4 r2 = tex1Dfetch(tex_projection_cam, cpos + 2);
  r[4] = r2.x;
  r[5] = r2.y;
  r[6] = r2.z;
  r[7] = r2.w;
  float4 r3 = tex1Dfetch(tex_projection_cam, cpos + 3);
  r[8] = r3.x;

  float4 temp = tex1Dfetch(tex_projection_pts, proj.y);
  m[0] = temp.x;
  m[1] = temp.y;
  m[2] = temp.z;

  float p0 = r[0] * m[0] + r[1] * m[1] + r[2] * m[2] + t[0];
  float p1 = r[3] * m[0] + r[4] * m[1] + r[5] * m[2] + t[1];
  float p2 = r[6] * m[0] + r[7] * m[1] + r[8] * m[2] + t[2];
  if (pd) {
    float rr = 1.0 + r3.y * (p0 * p0 + p1 * p1) / (p2 * p2);
    float f_p2 = FDIV2(f, p2);
    float2 ms = tex1Dfetch(tex_projection_mea, tidx);
    pj[tidx] = make_float2(ms.x / rr - p0 * f_p2, ms.y / rr - p1 * f_p2);
  } else if (md) {
    float f_p2 = FDIV2(f, p2);
    float2 ms = tex1Dfetch(tex_projection_mea, tidx);
    float rd = 1.0 + r3.y * (ms.x * ms.x + ms.y * ms.y);
    pj[tidx] = make_float2(ms.x - p0 * f_p2 / rd, ms.y - p1 * f_p2 / rd);
  } else {
    float f_p2 = FDIV2(f, p2);
    float2 ms = tex1Dfetch(tex_projection_mea, tidx);
    pj[tidx] = make_float2(ms.x - p0 * f_p2, ms.y - p1 * f_p2);
  }
}

void ProgramCU::ComputeProjectionX(CuTexImage& camera, CuTexImage& point,
                                   CuTexImage& meas, CuTexImage& proj_map,
                                   CuTexImage& proj, int radial) {
  unsigned int len = proj_map.GetImgWidth();
  unsigned int bsize = PROJECTION_FRT_KWIDTH;
  unsigned int nblock = (len + bsize - 1) / bsize;
  camera.BindTexture(tex_projection_cam);
  point.BindTexture(tex_projection_pts);
  proj_map.BindTexture(tex_projection_idx);
  unsigned int bw, bh;
  GetBlockConfiguration(nblock, bw, bh);
  dim3 grid(bw, bh), block(bsize);
  meas.BindTexture(tex_projection_mea);
  if (radial == -1)
    projectionx_frt_kernel<true, false><<<grid, block>>>(len, bw * bsize,
                                                         (float2*)proj.data());
  else if (radial)
    projectionx_frt_kernel<false, true><<<grid, block>>>(len, bw * bsize,
                                                         (float2*)proj.data());
  else
    projectionx_frt_kernel<false, false><<<grid, block>>>(len, bw * bsize,
                                                          (float2*)proj.data());
  CheckErrorCUDA("ComputeProjection");
}

texture<float2, 1, hipReadModeElementType> tex_jte_pe;
texture<float, 1, hipReadModeElementType> tex_jte_pex;
texture<float4, 1, hipReadModeElementType> tex_jte_jc;
texture<float4, 1, hipReadModeElementType> tex_jte_jc2;
texture<int, 1, hipReadModeElementType> tex_jte_cmp;
texture<int, 1, hipReadModeElementType> tex_jte_cmt;
texture<float4, 1, hipReadModeElementType> tex_jte_jc3;
texture<float4, 1, hipReadModeElementType> tex_jte_jc4;

__global__ void jte_cam_kernel(int num, float* jc, float* jte) {
  __shared__ float value[128];

  // 8thread per camera
  int col = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
  if (col >= num) return;

  int cam = col >> 4;  // 8 thread per camera

  // read data range for this camera, 8 thread will do the same thing
  int idx1 = tex1Dfetch(tex_jte_cmp, cam) << 4;  // first camera
  int idx2 = tex1Dfetch(tex_jte_cmp, cam + 1) << 4;  // last camera + 1

  ///////////////////////////////
  int offset = threadIdx.x & 0xf;  // which parameter of this camera
  int part = offset >= 8 ? 1 : 0;
  /////////////////////////////

  float result = 0;
  // loop to read the index of the projection.
  // so to get the location to read the jacobian
  for (int i = idx1 + offset; i < idx2; i += 16) {
    float temp = jc[i];
    // every 8 thread will read the same position.
    int index = tex1Dfetch(tex_jte_cmt, i >> 4);
    float v = tex1Dfetch(tex_jte_pex, (index << 1) + part);
    //////////////////////
    result += temp * v;
  }
  value[threadIdx.x] = result;
  // write back
  if (offset < 8) jte[(cam << 3) + offset] = (result + value[threadIdx.x + 8]);
}

template <int KH, int TEXN>
__global__ void jte_cam_vec_kernel(int num, float* jte) {
  __shared__ float value[KH * 128];
  int cam = blockIdx.x * KH + threadIdx.y;
  if (cam >= num) return;

  // read data range for this camera
  // 8 thread will do the same thing
  int idx1 = tex1Dfetch(tex_jte_cmp, cam) << 2;  // first camera
  int idx2 = tex1Dfetch(tex_jte_cmp, cam + 1) << 2;  // last camera + 1
  int part = (threadIdx.x & 0x02) ? 1 : 0;

  float rx = 0, ry = 0, rz = 0, rw = 0;
  // loop to read the index of the projection.
  // so to get the location to read the jacobian
  for (int i = idx1 + threadIdx.x; i < idx2; i += 32) {
    float4 temp;
    if (TEXN == 1) {
      temp = tex1Dfetch(tex_jte_jc, i);
    }
    if (TEXN == 2) {
      int texid = i >> 25;
      if (texid == 0)
        temp = tex1Dfetch(tex_jte_jc, i);
      else
        temp = tex1Dfetch(tex_jte_jc2, (i & 0x1ffffff));
    }
    if (TEXN == 4) {
      int index = tex1Dfetch(tex_jte_cmt, i >> 2);
      int iii = (index << 2) + (i & 0x3);
      int texid = iii >> 25;
      /////////////////////////////////
      if (texid == 0)
        temp = tex1Dfetch(tex_jte_jc, iii);
      else if (texid == 1)
        temp = tex1Dfetch(tex_jte_jc2, (iii & 0x1ffffff));
      else if (texid == 2)
        temp = tex1Dfetch(tex_jte_jc3, (iii & 0x1ffffff));
      else
        temp = tex1Dfetch(tex_jte_jc4, (iii & 0x1ffffff));
    }
    int index = tex1Dfetch(tex_jte_cmt, i >> 2);
    float vv = tex1Dfetch(tex_jte_pex, (index << 1) + part);
    rx += temp.x * vv;
    ry += temp.y * vv;
    rz += temp.z * vv;
    rw += temp.w * vv;
  }
  ////////////////////////////////////
  int widx = (threadIdx.y << 7) + (threadIdx.x << 2);
  ///////////////////////////////////
  // write back
  value[widx] = rx;
  value[widx + 1] = ry;
  value[widx + 2] = rz;
  value[widx + 3] = rw;
  ////////////////////////////////////
  int ridx = (threadIdx.y << 7) + threadIdx.x;
  value[ridx] = ((value[ridx] + value[ridx + 32]) +
                 (value[ridx + 64] + value[ridx + 96]));
  if (threadIdx.x < 16) value[ridx] += value[ridx + 16];
  if (threadIdx.x < 8)
    jte[(cam << 3) + threadIdx.x] = value[ridx] + value[ridx + 8];
}

template <int KH, bool JT>
__global__ void jte_cam_vec32_kernel(int num, float* jc, float* jte) {
  __shared__ float value[KH * 32];
  int cam = blockIdx.x * KH + threadIdx.y;
  if (cam >= num) return;
  float sum = 0;
  int rowpos = (threadIdx.y << 5);
  int index = threadIdx.x + rowpos;
  int xypart = (threadIdx.x & 0x08) ? 1 : 0;
  int part2 = threadIdx.x & 0xf;
  // read data range for this camera
  // 8 thread will do the same thing
  int idx1 = tex1Dfetch(tex_jte_cmp, cam) << 4;  // first camera
  int idx2 = tex1Dfetch(tex_jte_cmp, cam + 1) << 4;  // last camera + 1

  // loop to read the index of the projection.
  // so to get the location to read the jacobian
  for (int i = idx1 + threadIdx.x; i < idx2; i += 32) {
    int index = tex1Dfetch(tex_jte_cmt, i >> 4);
    float temp;
    if (JT)
      temp = jc[i];
    else
      temp = jc[(index << 4) + part2];

    float v = tex1Dfetch(tex_jte_pex, (index << 1) + xypart);
    sum += temp * v;
  }
  value[index] = sum;

  if (threadIdx.x < 16) value[index] += value[index + 16];
  if (threadIdx.x < 8)
    jte[(cam << 3) + threadIdx.x] = value[index] + value[index + 8];
}

/////////////////////////////////////////////////////////////
texture<float4, 1, hipReadModeElementType> tex_jte_jp;
texture<int, 1, hipReadModeElementType> tex_jte_pmp;
texture<float4, 1, hipReadModeElementType> tex_jte_jp2;

__global__ void jte_point_kernel(int num, float4* jte) {
  ////////////////////////////
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= num) return;

  int idx1 = tex1Dfetch(tex_jte_pmp, index);  // first camera
  int idx2 = tex1Dfetch(tex_jte_pmp, index + 1);  // last camera + 1
  float4 result = make_float4(0, 0, 0, 0);
  for (int i = idx1; i < idx2; ++i) {
    // error vector
    float2 ev = tex1Dfetch(tex_jte_pe, i);

    float4 j1 = tex1Dfetch(tex_jte_jp, i << 1);
    result.x += j1.x * ev.x;
    result.y += j1.y * ev.x;
    result.z += j1.z * ev.x;

    float4 j2 = tex1Dfetch(tex_jte_jp, 1 + (i << 1));
    result.x += j2.x * ev.y;
    result.y += j2.y * ev.y;
    result.z += j2.z * ev.y;
  }
  jte[index] = result;
}

////////////////////
// faster but not always more accurate
//#define JTE_POINT_VEC2

template <int KH, int TEXN>
__global__ void jte_point_vec_kernel(int num, int rowsz, float* jte) {
  ////////////////////////////
  __shared__ float value[KH * 128];
  int index = blockIdx.x * KH + threadIdx.y + blockIdx.y * rowsz;
  if (index >= num) return;
#ifdef JTE_POINT_VEC2
  int idx1 = tex1Dfetch(tex_jte_pmp, index);  // first
  int idx2 = tex1Dfetch(tex_jte_pmp, index + 1);  // last  + 1
#else
  int idx1 = tex1Dfetch(tex_jte_pmp, index) << 1;  // first
  int idx2 = tex1Dfetch(tex_jte_pmp, index + 1) << 1;  // last  + 1
#endif
  float rx = 0, ry = 0, rz = 0;
  for (int i = idx1 + threadIdx.x; i < idx2; i += 32) {
    if (TEXN == 2 && i >> 25) {
#ifdef JTE_POINT_VEC2

      float2 vv = tex1Dfetch(tex_jte_pe, i);
      float4 jp1 = tex1Dfetch(tex_jte_jp, ((i & 0x1ffffff) << 1));
      float4 jp2 = tex1Dfetch(tex_jte_jp, ((i & 0x1ffffff) << 1) + 1);
      rx += (jp1.x * vv.x + jp2.x * vv.y);
      ry += (jp1.y * vv.x + jp2.y * vv.y);
      rz += (jp1.z * vv.x + jp2.z * vv.y);
#else
      float vv = tex1Dfetch(tex_jte_pex, i);
      float4 jpi = tex1Dfetch(tex_jte_jp2, i & 0x1ffffff);
      rx += jpi.x * vv;
      ry += jpi.y * vv;
      rz += jpi.z * vv;
#endif
    } else {
#ifdef JTE_POINT_VEC2
      float2 vv = tex1Dfetch(tex_jte_pe, i);
      float4 jp1 = tex1Dfetch(tex_jte_jp, (i << 1));
      float4 jp2 = tex1Dfetch(tex_jte_jp, (i << 1) + 1);
      rx += (jp1.x * vv.x + jp2.x * vv.y);
      ry += (jp1.y * vv.x + jp2.y * vv.y);
      rz += (jp1.z * vv.x + jp2.z * vv.y);
#else
      float vv = tex1Dfetch(tex_jte_pex, i);
      float4 jpi = tex1Dfetch(tex_jte_jp, i);
      rx += jpi.x * vv;
      ry += jpi.y * vv;
      rz += jpi.z * vv;
#endif
    }
  }

  int rowp = threadIdx.y << 7;
  int loc = (threadIdx.x << 2) + rowp;
  value[loc] = rx;
  value[loc + 1] = ry;
  value[loc + 2] = rz;
  value[loc + 3] = 0;

  int ridx = threadIdx.x + rowp;
  value[ridx] = ((value[ridx] + value[ridx + 32]) +
                 (value[ridx + 64] + value[ridx + 96]));
  if (threadIdx.x < 16) value[ridx] += value[ridx + 16];
  if (threadIdx.x < 8) value[ridx] += value[ridx + 8];
  if (threadIdx.x < 4)
    jte[(index << 2) + threadIdx.x] = value[ridx] + value[ridx + 4];
}

#define JTE_CAMERA_VEC
#define JTE_POINT_VEC

void ProgramCU::ComputeJtE(CuTexImage& pe, CuTexImage& jc, CuTexImage& cmap,
                           CuTexImage& cmlist, CuTexImage& jp, CuTexImage& pmap,
                           CuTexImage& jte, bool jc_transpose, int mode) {
  //////////////////////////////////////////////////////////
  int ncam = int(cmap.GetImgWidth() - 1);  // how many cameras
  size_t szjc = jc.GetDataSize();

  //////////////////////////////
  cmap.BindTexture(tex_jte_cmp);
  cmlist.BindTexture(tex_jte_cmt);
#ifdef JTE_CAMERA_VEC2
  pe.BindTexture(tex_jte_pex);
  const unsigned int bheight = 2;
  dim3 block1(32, bheight), grid1((ncam + bheight - 1) / bheight);
  if (mode == 2) {
  } else if (jc_transpose)
    jte_cam_vec32_kernel<bheight, true><<<grid1, block1>>>(ncam, jc.data(),
                                                           jte.data());
  else
    jte_cam_vec32_kernel<bheight, false><<<grid1, block1>>>(ncam, jc.data(),
                                                            jte.data());

#elif defined(JTE_CAMERA_VEC)
  pe.BindTexture(tex_jte_pex);
  const unsigned int bheight = 2;
  unsigned int len1 = ncam * 32;
  unsigned int bsize1 = 32 * bheight;
  unsigned int nblock1 = (len1 + bsize1 - 1) / bsize1;
  dim3 grid1(nblock1);
  dim3 block1(32, bheight);
  if (mode == 2) {
    // skip camera
  } else if (szjc > 2 * MAX_TEXSIZE || !jc_transpose) {
    if (jc_transpose)
      jte_cam_vec32_kernel<bheight, true><<<grid1, block1>>>(ncam, jc.data(),
                                                             jte.data());
    else
      jte_cam_vec32_kernel<bheight, false><<<grid1, block1>>>(ncam, jc.data(),
                                                              jte.data());
  } else if (szjc > MAX_TEXSIZE) {
    jc.BindTexture2(tex_jte_jc, tex_jte_jc2);
    jte_cam_vec_kernel<bheight, 2><<<grid1, block1>>>(ncam, jte.data());
  } else {
    jc.BindTexture(tex_jte_jc);
    jte_cam_vec_kernel<bheight, 1><<<grid1, block1>>>(ncam, jte.data());
  }
#else
  pe.BindTexture(tex_jte_pex);
  unsigned int len1 = ncam * 16;
  unsigned int bsize1 = len1 > 32 * 128 ? 128 : (len1 > 32 * 64 ? 64 : 32);
  unsigned int nblock1 = (len1 + bsize1 - 1) / bsize1;
  dim3 grid1(nblock1), block1(bsize1);
  jte_cam_kernel<<<grid1, block1>>>(len1, jc.data(), jte.data());
#endif
  CheckErrorCUDA("ComputeJtE<Camera>");

  ////////////////////////////////////////////
  pmap.BindTexture(tex_jte_pmp);
  unsigned int npoint = (pmap.GetImgWidth() - 1);
#ifndef JTE_POINT_VEC
  size_t len2 = npoint;
  unsigned int bsize2 = 64;
  unsigned int nblock2 = (len2 + bsize2 - 1) / bsize2;
  dim3 grid2(nblock2), block2(bsize2);
  pe.BindTexture(tex_jte_pe);
  jp.BindTexture(tex_jte_jp);
  jte_point_kernel<<<grid2, block2>>>(len2, ((float4*)jte.data()) + 2 * ncam);
#else

#ifdef JTE_POINT_VEC2
  pe.BindTexture(tex_jte_pe);
#else
  pe.BindTexture(tex_jte_pex);
#endif
  const unsigned int bheight2 = 2;
  unsigned int bsize2 = 32;
  unsigned int nblock2 = (unsigned int)((npoint + bheight2 - 1) / bheight2);
  unsigned int offsetv = 8 * ncam;
  unsigned int bw, bh;
  GetBlockConfiguration(nblock2, bw, bh);
  dim3 grid2(bw, bh), block2(bsize2, bheight2);
  if (mode == 1) {
    // skip point
  } else if (jp.GetDataSize() > MAX_TEXSIZE) {
    jp.BindTexture2(tex_jte_jp, tex_jte_jp2);
    jte_point_vec_kernel<bheight2, 2><<<grid2, block2>>>(
        npoint, bw * bheight2, ((float*)jte.data()) + offsetv);
  } else {
    jp.BindTexture(tex_jte_jp);
    jte_point_vec_kernel<bheight2, 1><<<grid2, block2>>>(
        npoint, bw * bheight2, ((float*)jte.data()) + offsetv);
  }
#endif
  CheckErrorCUDA("ComputeJtE<Point>");
}

texture<int, 1, hipReadModeElementType> tex_jtjd_cmp;
texture<int, 1, hipReadModeElementType> tex_jtjd_cmlist;

template <int VN, int KH, bool JT>
__global__ void jtjd_cam_vec32_kernel(int num, int add_existing_dq, float* jc,
                                      float* jtjd, float* jtjdi) {
  __shared__ float value[KH * 32];

  // 8thread per camera
  int cam = blockIdx.x * KH + threadIdx.y;
  int part = threadIdx.x & 0x7;  // which parameter of this camera
  int part2 = threadIdx.x & 0xf;
  int campos = threadIdx.y << 5;
  int index = threadIdx.x + campos;
  float sum = 0;
  if (cam < num && part < VN) {
    // read data range for this camera
    // 8 thread will do the same thing
    int idx1 = tex1Dfetch(tex_jtjd_cmp, cam) << 4;  // first camera
    int idx2 = tex1Dfetch(tex_jtjd_cmp, cam + 1) << 4;  // last camera + 1

    // loop to read the index of the projection.
    // so to get the location to read the jacobian
    for (int i = idx1 + threadIdx.x; i < idx2; i += 32) {
      if (JT) {
        float temp = jc[i];
        sum += temp * temp;
      } else {
        int ii = tex1Dfetch(tex_jtjd_cmlist, i >> 4) << 4;
        float temp = jc[ii + part2];
        sum += temp * temp;
      }
    }
  }
  __syncthreads();

  if (cam >= num) return;
  // save all the results?
  value[index] = sum;
  if (threadIdx.x < 16) value[index] += value[index + 16];
  if (threadIdx.x < 8)

    // write back
    if (threadIdx.x < 8) {
      float temp = value[index] + value[index + 8];
      int wpos = threadIdx.x + (cam << 3);
      if (add_existing_dq) temp += jtjd[wpos];
      jtjd[wpos] = temp;
      jtjdi[wpos] = temp == 0 ? 0 : 1 / (temp);
    }
}

texture<float4, 1, hipReadModeElementType> tex_jtjd_jp;
texture<int, 1, hipReadModeElementType> tex_jtjd_pmp;
texture<float4, 1, hipReadModeElementType> tex_jtjd_jp2;

#define JTJD_POINT_KWIDTH 64

template <int TEXN>
__global__ void jtjd_point_kernel(int num, int rowsz, float4* jtjd,
                                  float4* jtjdi) {
  ////////////////////////////
  int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * rowsz;
  if (index >= num) return;

  int idx1 = tex1Dfetch(tex_jtjd_pmp, index);  // first camera
  int idx2 = tex1Dfetch(tex_jtjd_pmp, index + 1);  // last camera + 1
  float rx = 0, ry = 0, rz = 0;
  for (int i = idx1; i < idx2; ++i) {
    if (TEXN == 2 && i > 0xffffff) {
      float4 j1 = tex1Dfetch(tex_jtjd_jp2, (i & 0xffffff) << 1);
      rx += j1.x * j1.x;
      ry += j1.y * j1.y;
      rz += j1.z * j1.z;

      float4 j2 = tex1Dfetch(tex_jtjd_jp2, 1 + ((i & 0xffffff) << 1));
      rx += j2.x * j2.x;
      ry += j2.y * j2.y;
      rz += j2.z * j2.z;
    } else {
      float4 j1 = tex1Dfetch(tex_jtjd_jp, i << 1);
      rx += j1.x * j1.x;
      ry += j1.y * j1.y;
      rz += j1.z * j1.z;

      float4 j2 = tex1Dfetch(tex_jtjd_jp, 1 + (i << 1));
      rx += j2.x * j2.x;
      ry += j2.y * j2.y;
      rz += j2.z * j2.z;
    }
  }

  if (jtjd) jtjd[index] = make_float4(rx, ry, rz, 0.0f);
  jtjdi[index] = make_float4(1.0f / rx, 1.0f / ry, 1.0f / rz, 0.0f);
}

void ProgramCU::ComputeDiagonal(CuTexImage& jc, CuTexImage& cmap,
                                CuTexImage& jp, CuTexImage& pmap,
                                CuTexImage& cmlist, CuTexImage& jtjd,
                                CuTexImage& jtjdi, bool jc_transpose,
                                int radial, bool add_existing_diagc) {
  //////////////////////////////////////////////////////////
  size_t szjc = jc.GetDataSize();
  unsigned int ncam = (cmap.GetImgWidth() - 1);  // how many cameras

  const unsigned int bheight = 2;
  dim3 block1x(32, bheight), grid1x((ncam + bheight - 1) / bheight);
  cmap.BindTexture(tex_jtjd_cmp);
  if (jc_transpose) {
    if (radial)
      jtjd_cam_vec32_kernel<8, bheight, true><<<grid1x, block1x>>>(
          ncam, add_existing_diagc, jc.data(), jtjd.data(), jtjdi.data());
    else
      jtjd_cam_vec32_kernel<7, bheight, true><<<grid1x, block1x>>>(
          ncam, add_existing_diagc, jc.data(), jtjd.data(), jtjdi.data());
  } else {
    cmlist.BindTexture(tex_jtjd_cmlist);
    if (radial)
      jtjd_cam_vec32_kernel<8, bheight, false><<<grid1x, block1x>>>(
          ncam, add_existing_diagc, jc.data(), jtjd.data(), jtjdi.data());
    else
      jtjd_cam_vec32_kernel<7, bheight, false><<<grid1x, block1x>>>(
          ncam, add_existing_diagc, jc.data(), jtjd.data(), jtjdi.data());
  }
  CheckErrorCUDA("ComputeDiagonal<Camera>");

  ////////////////////////////////////////////
  unsigned int npoint = (pmap.GetImgWidth() - 1);
  unsigned int len2 = npoint;
  unsigned int bsize2 = JTJD_POINT_KWIDTH;
  unsigned int nblock2 = (len2 + bsize2 - 1) / bsize2;
  unsigned int bw, bh;
  GetBlockConfiguration(nblock2, bw, bh);
  dim3 grid2(bw, bh), block2(bsize2);
  pmap.BindTexture(tex_jtjd_pmp);

  if (jp.GetDataSize() > MAX_TEXSIZE) {
    jp.BindTexture2(tex_jtjd_jp, tex_jtjd_jp2);
    jtjd_point_kernel<2><<<grid2, block2>>>(len2, (bw * bsize2),
                                            ((float4*)jtjd.data()) + 2 * ncam,
                                            ((float4*)jtjdi.data()) + 2 * ncam);
  } else {
    jp.BindTexture(tex_jtjd_jp);
    jtjd_point_kernel<1><<<grid2, block2>>>(len2, (bw * bsize2),
                                            ((float4*)jtjd.data()) + 2 * ncam,
                                            ((float4*)jtjdi.data()) + 2 * ncam);
  }
  CheckErrorCUDA("ComputeDiagonal<Point>");
}

// for each
template <bool SJ>
__global__ void jtjd_cam_q_kernel(int num, int rowsz, float* qw, float4* diag) {
  int bindex = IMUL(blockIdx.x, blockDim.x) + rowsz * blockIdx.y;
  int index = bindex + threadIdx.x;
  if (index >= num) return;
  int tid = index & 0x1;
  float w = qw[index], ws = w * w * 2.0f;
  if (SJ) {
    float4 sj = tex1Dfetch(tex_jacobian_sj, index);
    float4 dj = tid == 0 ? make_float4(sj.x * sj.x * ws, 0, 0, 0)
                         : make_float4(0, 0, 0, sj.w * sj.w * ws);
    diag[index] = dj;
  } else {
    float4 dj = tid == 0 ? make_float4(ws, 0, 0, 0) : make_float4(0, 0, 0, ws);
    diag[index] = dj;
  }
}

void ProgramCU::ComputeDiagonalQ(CuTexImage& qlistw, CuTexImage& sj,
                                 CuTexImage& diag) {
  unsigned int bsize = 32;
  unsigned int len = qlistw.GetImgWidth() * 2;
  unsigned int nblock = (len + bsize - 1) / bsize;
  unsigned int bw, bh;
  GetBlockConfiguration(nblock, bw, bh);
  dim3 grid(bw, bh), block(bsize);
  if (sj.IsValid()) {
    sj.BindTexture(tex_jacobian_sj);
    jtjd_cam_q_kernel<true><<<grid, block>>>(len, (bw * bsize), qlistw.data(),
                                             (float4*)diag.data());
  } else {
    jtjd_cam_q_kernel<false><<<grid, block>>>(len, (bw * bsize), qlistw.data(),
                                              (float4*)diag.data());
  }
  CheckErrorCUDA("ComputeDiagonalQ");
}

template <int VN, int KH, bool JT>
__global__ void jtjd_cam_block_vec32_kernel(int num, float lambda1,
                                            float lambda2, float* jc,
                                            float* diag, float* blocks,
                                            bool add_existing_diagc) {
  __shared__ float value[KH * 32 * VN];

  // 8thread per camera
  int cam = blockIdx.x * KH + threadIdx.y;
  int part = threadIdx.x & 0x7;  // which parameter of this camera
  int part2 = threadIdx.x & 0xf;
  int index = threadIdx.x + (threadIdx.y << 5);
  float row[8] = {0, 0, 0, 0, 0, 0, 0, 0};
  if (cam < num) {
    int rowpos = index - part;
    // read data range for this camera
    // 8 thread will do the same thing
    int idx1 = tex1Dfetch(tex_jtjd_cmp, cam) << 4;  // first camera
    int idx2 = tex1Dfetch(tex_jtjd_cmp, cam + 1) << 4;  // last camera + 1

    // loop to read the index of the projection.
    // so to get the location to read the jacobian
    for (int i = idx1 + threadIdx.x; i < idx2; i += 32) {
      if (JT) {
        float temp = jc[i];
        value[index] = temp;
        for (int j = 0; j < VN; ++j) row[j] += (temp * value[rowpos + j]);
      } else {
        int ii = tex1Dfetch(tex_jtjd_cmlist, i >> 4) << 4;
        float temp = jc[ii + part2];
        value[index] = temp;
        for (int j = 0; j < VN; ++j) row[j] += (temp * value[rowpos + j]);
      }
    }
  }
  __syncthreads();

  if (cam >= num) return;
  // save all the results?
  for (int i = 0; i < VN; ++i) value[index * VN + i] = row[i];
  int campos = threadIdx.y * (32 * VN);
  for (int i = threadIdx.x; i < (VN * 16); i += 32)
    value[campos + i] += value[campos + i + (16 * VN)];
  for (int i = threadIdx.x; i < (VN * 8); i += 32)
    value[campos + i] += value[campos + i + (8 * VN)];

  if (VN == 7) {
    bool zero = (part >= VN);

    // write back
    if (threadIdx.x < 8) {
      float* dp = value + campos + threadIdx.x * (VN + 1);
      float temp = zero ? 0 : dp[0];
      int didx = threadIdx.x + (cam << 3);
      if (add_existing_diagc) temp += diag[didx];
      diag[didx] = temp;
      dp[0] = lambda1 + lambda2 * temp;
    }
    int wpos = cam * (8 * VN) + threadIdx.x;
    int rpos = campos + threadIdx.x - (threadIdx.x >> 3);
    blocks[wpos] = zero ? 0 : value[rpos];
    if (threadIdx.x < (VN * 8 - 32))
      blocks[wpos + 32] = zero ? 0 : value[rpos + 28];
  } else {
    // write back
    if (threadIdx.x < 8) {
      float* dp = value + campos + threadIdx.x * (VN + 1);
      float temp = dp[0];
      int didx = threadIdx.x + (cam << 3);
      if (add_existing_diagc) temp += diag[didx];
      diag[didx] = temp;
      dp[0] = lambda1 + lambda2 * temp;  // max(, 1e-6) ;
    }
    int wpos = cam * (8 * VN) + threadIdx.x;
    int rpos = campos + threadIdx.x;
    blocks[wpos] = value[rpos];
    blocks[wpos + 32] = value[rpos + 32];
  }
}

#define JTJD_POINT_BLOCK_KWIDTH 64

template <int TEXN>
__global__ void jtjd_point_block_kernel(int num, int rowsz, float lambda1,
                                        float lambda2, float4* diag,
                                        float4* blocks) {
  ////////////////////////////
  int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * rowsz;
  if (index >= num) return;

  int idx1 = tex1Dfetch(tex_jtjd_pmp, index);  // first camera
  int idx2 = tex1Dfetch(tex_jtjd_pmp, index + 1);  // last camera + 1

  float M00 = 0, M01 = 0, M02 = 0, M11 = 0, M12 = 0, M22 = 0;
  for (int i = idx1; i < idx2; ++i) {
    if (TEXN == 2 && i > 0xffffff) {
      float4 j1 = tex1Dfetch(tex_jtjd_jp2, (i & 0xffffff) << 1);
      M00 += j1.x * j1.x;
      M01 += j1.x * j1.y;
      M02 += j1.x * j1.z;
      M11 += j1.y * j1.y;
      M12 += j1.y * j1.z;
      M22 += j1.z * j1.z;

      float4 j2 = tex1Dfetch(tex_jtjd_jp2, 1 + ((i & 0xffffff) << 1));
      M00 += j2.x * j2.x;
      M01 += j2.x * j2.y;
      M02 += j2.x * j2.z;
      M11 += j2.y * j2.y;
      M12 += j2.y * j2.z;
      M22 += j2.z * j2.z;
    } else {
      float4 j1 = tex1Dfetch(tex_jtjd_jp, i << 1);
      M00 += j1.x * j1.x;
      M01 += j1.x * j1.y;
      M02 += j1.x * j1.z;
      M11 += j1.y * j1.y;
      M12 += j1.y * j1.z;
      M22 += j1.z * j1.z;

      float4 j2 = tex1Dfetch(tex_jtjd_jp, 1 + (i << 1));
      M00 += j2.x * j2.x;
      M01 += j2.x * j2.y;
      M02 += j2.x * j2.z;
      M11 += j2.y * j2.y;
      M12 += j2.y * j2.z;
      M22 += j2.z * j2.z;
    }
  }

  diag[index] = make_float4(M00, M11, M22, 0);

  M00 = lambda2 * M00 + lambda1;
  M11 = lambda2 * M11 + lambda1;
  M22 = lambda2 * M22 + lambda1;

  // invert the 3x3 matrix.
  float det = (M00 * M11 - M01 * M01) * M22 + 2.0 * M01 * M12 * M02 -
              M02 * M02 * M11 - M12 * M12 * M00;
  if (det >= FLT_MAX || det <= FLT_MIN * 2.0f) {
    int write_pos = index * 3;
    blocks[write_pos] = make_float4(0, 0, 0, 0);
    blocks[write_pos + 1] = make_float4(0, 0, 0, 0);
    blocks[write_pos + 2] = make_float4(0, 0, 0, 0);
  } else {
    float m00 = (M11 * M22 - M12 * M12) / det;
    float m01 = -(M01 * M22 - M12 * M02) / det;
    float m02 = (M01 * M12 - M02 * M11) / det;
    int write_pos = index * 3;
    blocks[write_pos] = make_float4(m00, m01, m02, 0);

    float m11 = (M00 * M22 - M02 * M02) / det;
    float m12 = -(M00 * M12 - M01 * M02) / det;
    blocks[write_pos + 1] = make_float4(m01, m11, m12, 0);

    float m22 = (M00 * M11 - M01 * M01) / det;
    blocks[write_pos + 2] = make_float4(m02, m12, m22, 0);
  }
}

#define JTJD_BLOCK_CAM_INVERT_KWIDTH 64
template <int VN>
__global__ void jtjd_cam_block_invert_kernel(int num, float4* blocks) {
  // N /  8 cameras...each have 64 floats,,,, N * 8 float
  // each will read 8 float......
  __shared__ float value[JTJD_BLOCK_CAM_INVERT_KWIDTH * VN];
  __shared__ bool invalid[JTJD_BLOCK_CAM_INVERT_KWIDTH / 8];
  //////////////////////////////////////////////

  int bindex = IMUL(blockIdx.x, blockDim.x);
  int index = bindex + threadIdx.x;
  int block_read_pos = IMUL(bindex, VN);
  for (int i = 0; i < JTJD_BLOCK_CAM_INVERT_KWIDTH * VN;
       i += JTJD_BLOCK_CAM_INVERT_KWIDTH)
    value[threadIdx.x + i] = ((float*)blocks)[block_read_pos + threadIdx.x + i];
  __syncthreads();
  const int cam_id = threadIdx.x >> 3;
  const int cam_pos = IMUL(cam_id, VN * 8);
  const int col = threadIdx.x & 0x7, rowj_pos = col << 3;
  ;  //

  float* a = value + cam_pos;
  for (int i = 0; i < VN; ++i) {
    int rowi_pos = i << 3, dpos = i + rowi_pos;
    if (col == i && a[dpos] > 0) a[dpos] = rsqrt(a[dpos]);
    __syncthreads();
    float diag = a[dpos];
    if (diag == 0 || col >= VN) continue;
    if (col < i) {
      a[rowi_pos + col] = 0;
    } else if (col > i) {
      float aij = a[rowi_pos + col] * diag;
      a[rowi_pos + col] = aij;
      for (int k = col; k < VN; ++k) a[rowj_pos + k] -= a[rowi_pos + k] * aij;
    }
  }

  if (index >= num) return;

  if (col == 0) invalid[cam_id] = false;
  if (col < VN) {
    for (int i = 1; i < VN; ++i) {
      int rowi_pos = i << 3, dpos = i + rowi_pos;
      if (a[dpos] == 0) continue;
      if (col < i) {
        float sum = 0;
        for (int k = col; k < i; ++k)
          sum += (a[(k << 3) + i] * a[rowj_pos + k]);
        a[rowj_pos + i] = -sum * a[dpos];
      }
    }
    float ai[8], amax = 0;
    for (int i = 0; i < VN * 8; i += 8) {
      float sum = 0;
      for (int k = 0; k < VN; k++) sum += a[rowj_pos + k] * a[i + k];
      ai[i >> 3] = sum;
      amax = max(amax, sum);
    }

    if (isinf(amax)) invalid[cam_id] = true;
    int write_pos = IMUL((index >> 3), (VN * 2)) + (col << 1);
    if (invalid[cam_id])  // a better way would be using a threshold
    {
      blocks[write_pos] = make_float4(0, 0, 0, 0);
      blocks[write_pos + 1] = make_float4(0, 0, 0, 0);
    } else {
      blocks[write_pos] = make_float4(ai[0], ai[1], ai[2], ai[3]);
      blocks[write_pos + 1] =
          make_float4(ai[4], ai[5], ai[6], VN < 8 ? 0 : ai[7]);
    }
  }
}

void ProgramCU::ComputeDiagonalBlock(float lambda, bool dampd, CuTexImage& jc,
                                     CuTexImage& cmap, CuTexImage& jp,
                                     CuTexImage& pmap, CuTexImage& cmlist,
                                     CuTexImage& diag, CuTexImage& blocks,
                                     int radial_distortion, bool jc_transpose,
                                     bool add_existing_diagc, int mode) {
  size_t szjc = jc.GetDataSize();
  unsigned int ncam = (cmap.GetImgWidth() - 1);  // how many cameras
  float lambda1 = dampd ? 0.0f : lambda;
  float lambda2 = dampd ? (1.0f + lambda) : 1.0f;
  const unsigned int bheight = 2;
  dim3 block1x(32, bheight), grid1x((ncam + bheight - 1) / bheight);
  cmap.BindTexture(tex_jtjd_cmp);

  if (mode == 2) {
    // point only mode?
  } else if (radial_distortion) {
    if (jc_transpose) {
      jtjd_cam_block_vec32_kernel<8, bheight, true><<<grid1x, block1x>>>(
          ncam, lambda1, lambda2, jc.data(), diag.data(), blocks.data(),
          add_existing_diagc);
    } else {
      cmlist.BindTexture(tex_jtjd_cmlist);
      jtjd_cam_block_vec32_kernel<8, bheight, false><<<grid1x, block1x>>>(
          ncam, lambda1, lambda2, jc.data(), diag.data(), blocks.data(),
          add_existing_diagc);
    }
  } else {
    if (jc_transpose) {
      jtjd_cam_block_vec32_kernel<7, bheight, true><<<grid1x, block1x>>>(
          ncam, lambda1, lambda2, jc.data(), diag.data(), blocks.data(),
          add_existing_diagc);
    } else {
      cmlist.BindTexture(tex_jtjd_cmlist);
      jtjd_cam_block_vec32_kernel<7, bheight, false><<<grid1x, block1x>>>(
          ncam, lambda1, lambda2, jc.data(), diag.data(), blocks.data(),
          add_existing_diagc);
    }
  }
  CheckErrorCUDA("ComputeDiagonalBlock<Camera>");

  ////////////////////////////////////////////
  unsigned int npoint = (pmap.GetImgWidth() - 1);
  unsigned int len2 = npoint;
  unsigned int bsize2 = JTJD_POINT_BLOCK_KWIDTH;
  unsigned int nblock2 = (len2 + bsize2 - 1) / bsize2;
  unsigned int bw, bh;
  unsigned int offsetd = 2 * ncam;
  unsigned int offsetb = (radial_distortion ? 16 : 14) * ncam;
  GetBlockConfiguration(nblock2, bw, bh);
  dim3 grid2(bw, bh), block2(bsize2);
  pmap.BindTexture(tex_jtjd_pmp);
  if (mode == 1) {
    // camera only mode?
  } else if (jp.GetDataSize() > MAX_TEXSIZE) {
    jp.BindTexture2(tex_jtjd_jp, tex_jtjd_jp2);
    jtjd_point_block_kernel<2><<<grid2, block2>>>(
        len2, (bw * bsize2), lambda1, lambda2, ((float4*)diag.data()) + offsetd,
        ((float4*)blocks.data()) + offsetb);
  } else {
    jp.BindTexture(tex_jtjd_jp);
    jtjd_point_block_kernel<1><<<grid2, block2>>>(
        len2, (bw * bsize2), lambda1, lambda2, ((float4*)diag.data()) + offsetd,
        ((float4*)blocks.data()) + offsetb);
  }
  CheckErrorCUDA("ComputeDiagonalBlock<Point>");

  if (mode != 2) {
    unsigned int len3 = ncam * 8;
    unsigned int bsize3 = JTJD_BLOCK_CAM_INVERT_KWIDTH;
    unsigned int nblock3 = (len3 + bsize3 - 1) / bsize3;
    dim3 grid3(nblock3), block3(bsize3);
    if (radial_distortion)
      jtjd_cam_block_invert_kernel<8><<<grid3, block3>>>(
          len3, (float4*)blocks.data());
    else
      jtjd_cam_block_invert_kernel<7><<<grid3, block3>>>(
          len3, (float4*)blocks.data());
    CheckErrorCUDA("ComputeDiagonalBlockInverse<Camera>");
  }
}

template <int WIDTH, int BBIT, int VSZ>
__global__ void multiply_block_conditioner_kernel(int num, int rowsz,
                                                  float* blocks, float* x,
                                                  float* result) {
  __shared__ float mat[WIDTH * VSZ];
  __shared__ float val[WIDTH];
  const int BSZ = 1 << BBIT;
  const int BMASK = BSZ - 1;
  int bindex = IMUL(blockIdx.x, blockDim.x) + rowsz * blockIdx.y;
  int index = bindex + threadIdx.x;
  int block_read_pos = bindex * VSZ;
  val[threadIdx.x] = x[index];
  for (int i = 0; i < VSZ * WIDTH; i += WIDTH)
    mat[i + threadIdx.x] = blocks[i + block_read_pos + threadIdx.x];
  __syncthreads();
  if (index >= num) return;
  float* ac = mat + (threadIdx.x >> BBIT) * (BSZ * VSZ) + (threadIdx.x & BMASK);
  float* xc = val + (threadIdx.x & (~BMASK));
  float sum = 0;
  for (int i = 0; i < VSZ; ++i) sum += ac[i << BBIT] * xc[i];
  result[index] = sum;  // isinf(sum) ? 0 : sum ; //
}

void ProgramCU::MultiplyBlockConditioner(int ncam, int npoint,
                                         CuTexImage& blocks, CuTexImage& vector,
                                         CuTexImage& result, int radial,
                                         int mode) {
  const unsigned int bsize1 = 64;
  unsigned int bw, bh;

  if (mode != 2) {
    unsigned int len1 = ncam * 8;
    unsigned int nblock1 = (len1 + bsize1 - 1) / bsize1;
    GetBlockConfiguration(nblock1, bw, bh);
    dim3 grid1(bw, bh), block1(bsize1);
    if (radial)
      multiply_block_conditioner_kernel<bsize1, 3, 8><<<grid1, block1>>>(
          len1, (bw * bsize1), blocks.data(), vector.data(), result.data());
    else
      multiply_block_conditioner_kernel<bsize1, 3, 7><<<grid1, block1>>>(
          len1, (bw * bsize1), blocks.data(), vector.data(), result.data());
    CheckErrorCUDA("MultiplyBlockConditioner<Camera>");
  }

  if (mode != 1) {
    const unsigned int bsize2 = 128;
    unsigned int len2 = npoint * 4;
    unsigned int nblock2 = (len2 + bsize2 - 1) / bsize2;
    unsigned int cbsz = radial ? 64 : 56;
    unsigned int offsetb = ncam * cbsz;
    unsigned int offsetd = ncam * 8;
    GetBlockConfiguration(nblock2, bw, bh);
    dim3 grid2(bw, bh), block2(bsize2);
    multiply_block_conditioner_kernel<bsize2, 2, 3><<<grid2, block2>>>(
        len2, (bw * bsize2), blocks.data() + offsetb, vector.data() + offsetd,
        result.data() + offsetd);
    CheckErrorCUDA("MultiplyBlockConditioner<Point>");
  }
}

texture<float4, 1, hipReadModeElementType> tex_shuffle_jc;
texture<int, 1, hipReadModeElementType> tex_shuffle_map;
texture<float4, 1, hipReadModeElementType> tex_shuffle_jc2;
template <int TEXN>
__global__ void shuffle_camera_jacobian_kernel(int num, int bwidth,
                                               float4* jc) {
  int index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  if (index >= num) return;
  int fetch_idx = tex1Dfetch(tex_shuffle_map, index >> 2);
  if (TEXN == 2) {
    int texidx = fetch_idx >> 23,
        fidx = ((fetch_idx & 0x7fffff) << 2) + (index & 0x3);
    if (texidx == 0)
      jc[index] = tex1Dfetch(tex_shuffle_jc, fidx);
    else if (texidx == 1)
      jc[index] = tex1Dfetch(tex_shuffle_jc2, fidx);
  }
  if (TEXN == 1) {
    jc[index] = tex1Dfetch(tex_shuffle_jc, (fetch_idx << 2) + (index & 0x3));
  }
}

bool ProgramCU::ShuffleCameraJacobian(CuTexImage& jc, CuTexImage& map,
                                      CuTexImage& result) {
  if (!result.IsValid()) return false;
  size_t szjc = jc.GetDataSize();
  unsigned int len = map.GetImgWidth() * 4;
  unsigned int bsize = 128;
  unsigned int nblock = (len + bsize - 1) / bsize;

  map.BindTexture(tex_shuffle_map);

  if (szjc > 2 * MAX_TEXSIZE) {
    fprintf(stderr, "datasize way too big %lX, %lX+...\n", szjc,
            (szjc) / MAX_TEXSIZE);
    return false;
  } else if (szjc > MAX_TEXSIZE) {
    unsigned int bw, bh;
    GetBlockConfiguration(nblock, bw, bh);
    dim3 grid(bw, bh), block(bsize);
    jc.BindTexture2(tex_shuffle_jc, tex_shuffle_jc2);
    shuffle_camera_jacobian_kernel<2><<<grid, block>>>(len, (bw * bsize),
                                                       (float4*)result.data());
  } else {
    jc.BindTexture(tex_shuffle_jc);
    unsigned int bw, bh;
    GetBlockConfiguration(nblock, bw, bh);
    dim3 grid(bw, bh), block(bsize);
    shuffle_camera_jacobian_kernel<1><<<grid, block>>>(len, (bw * bsize),
                                                       (float4*)result.data());
  }
  CheckErrorCUDA("ShuffleCameraJacobian");
  return true;
}

texture<float4, 1, hipReadModeElementType> tex_mjx_jc;
texture<float4, 1, hipReadModeElementType> tex_mjx_jc2;
texture<float4, 1, hipReadModeElementType> tex_mjx_jc3;
texture<float4, 1, hipReadModeElementType> tex_mjx_jc4;
texture<float4, 1, hipReadModeElementType> tex_mjx_jp;
texture<float4, 1, hipReadModeElementType> tex_mjx_jp2;
texture<int2, 1, hipReadModeElementType> tex_mjx_idx;
texture<float4, 1, hipReadModeElementType> tex_mjx_x;

template <int TEXN>
__global__ void multiply_jx_kernel(int num, int bwidth, int offset,
                                   float* result) {
  int index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  if (index >= num) return;

  if (TEXN == 4 && (index >> 24) == 3) {
    ////////////////////////////////////////////
    int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
    float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
    float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);
    float4 xp = tex1Dfetch(tex_mjx_x, proj.y + offset);

    ////////////////////////////////////////////
    float4 jp, jc1, jc2;
    jp = tex1Dfetch(tex_mjx_jp2, index & 0x1ffffff);
    jc1 = tex1Dfetch(tex_mjx_jc4, (index & 0xffffff) << 1);
    jc2 = tex1Dfetch(tex_mjx_jc4, ((index & 0xffffff) << 1) + 1);

    /////////////////////////////////////
    result[index] = jc1.x * xc1.x + jc1.y * xc1.y + jc1.z * xc1.z +
                    jc1.w * xc1.w + jc2.x * xc2.x + jc2.y * xc2.y +
                    jc2.z * xc2.z + jc2.w * xc2.w + jp.x * xp.x + jp.y * xp.y +
                    jp.z * xp.z;
  } else if (TEXN > 2 && (index >> 24) == 2) {
    ////////////////////////////////////////////
    int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
    float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
    float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);
    float4 xp = tex1Dfetch(tex_mjx_x, proj.y + offset);

    ////////////////////////////////////////////
    float4 jp, jc1, jc2;
    jp = tex1Dfetch(tex_mjx_jp2, index & 0x1ffffff);
    jc1 = tex1Dfetch(tex_mjx_jc3, (index & 0xffffff) << 1);
    jc2 = tex1Dfetch(tex_mjx_jc3, ((index & 0xffffff) << 1) + 1);

    /////////////////////////////////////
    result[index] = jc1.x * xc1.x + jc1.y * xc1.y + jc1.z * xc1.z +
                    jc1.w * xc1.w + jc2.x * xc2.x + jc2.y * xc2.y +
                    jc2.z * xc2.z + jc2.w * xc2.w + jp.x * xp.x + jp.y * xp.y +
                    jp.z * xp.z;
  } else if (TEXN > 1 && (index > 0xffffff)) {
    ////////////////////////////////////////////
    int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
    float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
    float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);
    float4 xp = tex1Dfetch(tex_mjx_x, proj.y + offset);

    ////////////////////////////////////////////
    float4 jp, jc1, jc2;
    jp = tex1Dfetch(tex_mjx_jp, index & 0x1ffffff);
    jc1 = tex1Dfetch(tex_mjx_jc2, (index & 0xffffff) << 1);
    jc2 = tex1Dfetch(tex_mjx_jc2, ((index & 0xffffff) << 1) + 1);

    /////////////////////////////////////
    result[index] = jc1.x * xc1.x + jc1.y * xc1.y + jc1.z * xc1.z +
                    jc1.w * xc1.w + jc2.x * xc2.x + jc2.y * xc2.y +
                    jc2.z * xc2.z + jc2.w * xc2.w + jp.x * xp.x + jp.y * xp.y +
                    jp.z * xp.z;
  } else {
    ////////////////////////////////////////////
    int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
    float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
    float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);
    float4 xp = tex1Dfetch(tex_mjx_x, proj.y + offset);

    ////////////////////////////////////////////
    float4 jp, jc1, jc2;
    jp = tex1Dfetch(tex_mjx_jp, index);
    jc1 = tex1Dfetch(tex_mjx_jc, index << 1);
    jc2 = tex1Dfetch(tex_mjx_jc, (index << 1) + 1);

    /////////////////////////////////////
    result[index] = jc1.x * xc1.x + jc1.y * xc1.y + jc1.z * xc1.z +
                    jc1.w * xc1.w + jc2.x * xc2.x + jc2.y * xc2.y +
                    jc2.z * xc2.z + jc2.w * xc2.w + jp.x * xp.x + jp.y * xp.y +
                    jp.z * xp.z;
  }
}

template <int TEXN>
__global__ void multiply_jcx_kernel(int num, int bwidth, float* result) {
  int index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  if (index >= num) return;

  if (TEXN == 4 && (index >> 24) == 3) {
    ////////////////////////////////////////////
    int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
    float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
    float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);

    ////////////////////////////////////////////
    float4 jc1, jc2;
    jc1 = tex1Dfetch(tex_mjx_jc4, (index & 0xffffff) << 1);
    jc2 = tex1Dfetch(tex_mjx_jc4, ((index & 0xffffff) << 1) + 1);

    /////////////////////////////////////
    result[index] = jc1.x * xc1.x + jc1.y * xc1.y + jc1.z * xc1.z +
                    jc1.w * xc1.w + jc2.x * xc2.x + jc2.y * xc2.y +
                    jc2.z * xc2.z + jc2.w * xc2.w;
  } else if (TEXN > 2 && (index >> 24) == 2) {
    ////////////////////////////////////////////
    int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
    float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
    float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);

    ////////////////////////////////////////////
    float4 jc1, jc2;
    jc1 = tex1Dfetch(tex_mjx_jc3, (index & 0xffffff) << 1);
    jc2 = tex1Dfetch(tex_mjx_jc3, ((index & 0xffffff) << 1) + 1);

    /////////////////////////////////////
    result[index] = jc1.x * xc1.x + jc1.y * xc1.y + jc1.z * xc1.z +
                    jc1.w * xc1.w + jc2.x * xc2.x + jc2.y * xc2.y +
                    jc2.z * xc2.z + jc2.w * xc2.w;
  } else if (TEXN > 1 && (index > 0xffffff)) {
    ////////////////////////////////////////////
    int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
    float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
    float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);

    ////////////////////////////////////////////
    float4 jc1, jc2;
    jc1 = tex1Dfetch(tex_mjx_jc2, (index & 0xffffff) << 1);
    jc2 = tex1Dfetch(tex_mjx_jc2, ((index & 0xffffff) << 1) + 1);

    /////////////////////////////////////
    result[index] = jc1.x * xc1.x + jc1.y * xc1.y + jc1.z * xc1.z +
                    jc1.w * xc1.w + jc2.x * xc2.x + jc2.y * xc2.y +
                    jc2.z * xc2.z + jc2.w * xc2.w;
  } else {
    ////////////////////////////////////////////
    int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
    float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
    float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);

    ////////////////////////////////////////////
    float4 jc1, jc2;
    jc1 = tex1Dfetch(tex_mjx_jc, index << 1);
    jc2 = tex1Dfetch(tex_mjx_jc, (index << 1) + 1);

    /////////////////////////////////////
    result[index] = jc1.x * xc1.x + jc1.y * xc1.y + jc1.z * xc1.z +
                    jc1.w * xc1.w + jc2.x * xc2.x + jc2.y * xc2.y +
                    jc2.z * xc2.z + jc2.w * xc2.w;
  }
}

template <int TEXN>
__global__ void multiply_jpx_kernel(int num, int bwidth, int offset,
                                    float* result) {
  int index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  if (index >= num) return;

  if (TEXN == 2 && index > 0x1ffffff) {
    ////////////////////////////////////////////
    int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
    float4 xp = tex1Dfetch(tex_mjx_x, proj.y + offset);
    ////////////////////////////////////////////
    float4 jp = tex1Dfetch(tex_mjx_jp2, index & 0x1ffffff);
    /////////////////////////////////////
    result[index] = jp.x * xp.x + jp.y * xp.y + jp.z * xp.z;
  } else {
    ////////////////////////////////////////////
    int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
    float4 xp = tex1Dfetch(tex_mjx_x, proj.y + offset);

    ////////////////////////////////////////////
    float4 jp = tex1Dfetch(tex_mjx_jp, index);
    /////////////////////////////////////
    result[index] = jp.x * xp.x + jp.y * xp.y + jp.z * xp.z;
  }
}

template <int KW>
__global__ void multiply_jx_notex2_kernel(int num, int bwidth, int offset,
                                          float* jcx, float* jpx,
                                          float* result) {
  int bindex = blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  int index = threadIdx.x + bindex;

  ////////////////////////////////////////////
  int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
  float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
  float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);
  float4 xp = tex1Dfetch(tex_mjx_x, proj.y + offset);
  ////////////////////////////////////////////
  __shared__ float jps[KW * 4];
  __shared__ float jcs[KW * 8];

  for (int i = threadIdx.x; i < 4 * KW; i += KW)
    jps[i] = jpx[(bindex << 2) + i];
  for (int i = threadIdx.x; i < 8 * KW; i += KW)
    jcs[i] = jcx[(bindex << 3) + i];

  __syncthreads();
  if (index >= num) return;

  /////////////////////////////////////
  float *jp = jps + threadIdx.x * 4, *jc = jcs + threadIdx.x * 8;
  result[index] = jc[0] * xc1.x + jc[1] * xc1.y + jc[2] * xc1.z +
                  jc[3] * xc1.w + jc[4] * xc2.x + jc[5] * xc2.y +
                  jc[6] * xc2.z + jc[7] * xc2.w + jp[0] * xp.x + jp[1] * xp.y +
                  jp[2] * xp.z;
}

template <int KW>
__global__ void multiply_jpx_notex2_kernel(int num, int bwidth, int offset,
                                           float* jpx, float* result) {
  int bindex = blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  int index = threadIdx.x + bindex;

  ////////////////////////////////////////////
  int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
  float4 xp = tex1Dfetch(tex_mjx_x, proj.y + offset);
  ////////////////////////////////////////////
  __shared__ float jps[KW * 4];

  for (int i = threadIdx.x; i < 4 * KW; i += KW)
    jps[i] = jpx[(bindex << 2) + i];

  __syncthreads();
  if (index >= num) return;

  /////////////////////////////////////
  float* jp = jps + threadIdx.x * 4;
  result[index] = jp[0] * xp.x + jp[1] * xp.y + jp[2] * xp.z;
}

template <int KW>
__global__ void multiply_jcx_notex2_kernel(int num, int bwidth, float* jcx,
                                           float* result) {
  int bindex = blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  int index = threadIdx.x + bindex;

  ////////////////////////////////////////////
  int2 proj = tex1Dfetch(tex_mjx_idx, index >> 1);
  float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
  float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);
  ////////////////////////////////////////////

  __shared__ float jcs[KW * 8];
  for (int i = threadIdx.x; i < 8 * KW; i += KW)
    jcs[i] = jcx[(bindex << 3) + i];

  __syncthreads();
  if (index >= num) return;

  /////////////////////////////////////
  float* jc = jcs + threadIdx.x * 8;
  result[index] = jc[0] * xc1.x + jc[1] * xc1.y + jc[2] * xc1.z +
                  jc[3] * xc1.w + jc[4] * xc2.x + jc[5] * xc2.y +
                  jc[6] * xc2.z + jc[7] * xc2.w;
}

void ProgramCU::ComputeJX(int point_offset, CuTexImage& x, CuTexImage& jc,
                          CuTexImage& jp, CuTexImage& jmap, CuTexImage& result,
                          int mode) {
  // given a vector of parameters....
  // multiply the Jacobian Matrix with it [jc jp] * p
  // for each measurment, read back the jacobian
  // multiply and summ up th corresponding

  unsigned int nproj = jmap.GetImgWidth();
  unsigned int len = nproj * 2;
  unsigned int bsize = 64;
  unsigned int nblock = (len + bsize - 1) / bsize;
  unsigned int bw, bh;
  jmap.BindTexture(tex_mjx_idx);
  x.BindTexture(tex_mjx_x);

  if (mode == 0) {
    size_t szjc = jc.GetDataSize();
    if (TEX_TOOBIG4(szjc)) {
      GetBlockConfiguration(nblock, bw, bh);
      dim3 grid(bw, bh), block(bsize);
      multiply_jx_notex2_kernel<64><<<grid, block>>>(
          len, (bw * bsize), point_offset, jc.data(), jp.data(), result.data());
    } else if (szjc > 2 * MAX_TEXSIZE) {
      jp.BindTexture2(tex_mjx_jp, tex_mjx_jp2);
      jc.BindTexture4(tex_mjx_jc, tex_mjx_jc2, tex_mjx_jc3, tex_mjx_jc4);
      GetBlockConfiguration(nblock, bw, bh);
      dim3 grid(bw, bh), block(bsize);
      multiply_jx_kernel<4><<<grid, block>>>(len, (bw * bsize), point_offset,
                                             result.data());
    } else if (szjc > MAX_TEXSIZE) {
      jp.BindTexture(tex_mjx_jp);
      jc.BindTexture2(tex_mjx_jc, tex_mjx_jc2);
      GetBlockConfiguration(nblock, bw, bh);
      dim3 grid(bw, bh), block(bsize);
      multiply_jx_kernel<2><<<grid, block>>>(len, (bw * bsize), point_offset,
                                             result.data());
    } else {
      jp.BindTexture(tex_mjx_jp);
      jc.BindTexture(tex_mjx_jc);
      GetBlockConfiguration(nblock, bw, bh);
      dim3 grid(bh, bw), block(bsize);
      multiply_jx_kernel<1><<<grid, block>>>(len, (bh * bsize), point_offset,
                                             result.data());
    }
    CheckErrorCUDA("ComputeJX");
  } else if (mode == 1) {
    size_t szjc = jc.GetDataSize();
    if (TEX_TOOBIG4(szjc)) {
      GetBlockConfiguration(nblock, bw, bh);
      dim3 grid(bw, bh), block(bsize);
      multiply_jcx_notex2_kernel<64><<<grid, block>>>(len, (bw * bsize),
                                                      jc.data(), result.data());
    } else if (szjc > 2 * MAX_TEXSIZE) {
      jc.BindTexture4(tex_mjx_jc, tex_mjx_jc2, tex_mjx_jc3, tex_mjx_jc4);
      GetBlockConfiguration(nblock, bw, bh);
      dim3 grid(bw, bh), block(bsize);
      multiply_jcx_kernel<4><<<grid, block>>>(len, (bw * bsize), result.data());
    } else if (szjc > MAX_TEXSIZE) {
      jc.BindTexture2(tex_mjx_jc, tex_mjx_jc2);
      GetBlockConfiguration(nblock, bw, bh);
      dim3 grid(bw, bh), block(bsize);
      multiply_jcx_kernel<2><<<grid, block>>>(len, (bw * bsize), result.data());
    } else {
      jc.BindTexture(tex_mjx_jc);
      GetBlockConfiguration(nblock, bw, bh);
      dim3 grid(bh, bw), block(bsize);
      multiply_jcx_kernel<1><<<grid, block>>>(len, (bh * bsize), result.data());
    }
    CheckErrorCUDA("ComputeJCX");
  } else if (mode == 2) {
    size_t szjp = jp.GetDataSize();
    if (szjp > MAX_TEXSIZE) {
      jp.BindTexture(tex_mjx_jp);
      GetBlockConfiguration(nblock, bw, bh);
      dim3 grid(bw, bh), block(bsize);
      multiply_jpx_kernel<2><<<grid, block>>>(len, (bw * bsize), point_offset,
                                              result.data());
    } else {
      jp.BindTexture(tex_mjx_jp);
      GetBlockConfiguration(nblock, bw, bh);
      dim3 grid(bh, bw), block(bsize);
      multiply_jpx_kernel<1><<<grid, block>>>(len, (bh * bsize), point_offset,
                                              result.data());
    }
    CheckErrorCUDA("ComputeJPX");
  }
}

template <bool md, bool pd>
__device__ void jacobian_internal(int camera_pos, int pt_pos, int tidx,
                                  float* r, float jic, float* jxc, float* jyc,
                                  float* jxp, float* jyp) {
  float m[3];
  float4 ft = tex1Dfetch(tex_jacobian_cam, camera_pos);
  float4 r1 = tex1Dfetch(tex_jacobian_cam, camera_pos + 1);
  r[0] = r1.x;
  r[1] = r1.y;
  r[2] = r1.z;
  r[3] = r1.w;
  float4 r2 = tex1Dfetch(tex_jacobian_cam, camera_pos + 2);
  r[4] = r2.x;
  r[5] = r2.y;
  r[6] = r2.z;
  r[7] = r2.w;
  float4 r3 = tex1Dfetch(tex_jacobian_cam, camera_pos + 3);
  r[8] = r3.x;

  float4 temp = tex1Dfetch(tex_jacobian_pts, pt_pos);
  m[0] = temp.x;
  m[1] = temp.y;
  m[2] = temp.z;

  float x0 = r[0] * m[0] + r[1] * m[1] + r[2] * m[2];
  float y0 = r[3] * m[0] + r[4] * m[1] + r[5] * m[2];
  float z0 = r[6] * m[0] + r[7] * m[1] + r[8] * m[2];
  float f_p2 = FDIV(ft.x, z0 + ft.w);
  float p0_p2 = FDIV(x0 + ft.y, z0 + ft.w);
  float p1_p2 = FDIV(y0 + ft.z, z0 + ft.w);

  if (pd) {
    float rr1 = r3.y * p0_p2 * p0_p2;
    float rr2 = r3.y * p1_p2 * p1_p2;
    float f_p2_x = f_p2 * (1.0 + 3.0 * rr1 + rr2);
    float f_p2_y = f_p2 * (1.0 + 3.0 * rr2 + rr1);

    JACOBIAN_SET_JC_BEGIN
    float jfc = jic * (1 + rr1 + rr2);
    float ft_x_pn = jic * ft.x * (p0_p2 * p0_p2 + p1_p2 * p1_p2);
    /////////////////////////////////////////////////////
    jxc[0] = p0_p2 * jfc;
    jxc[1] = f_p2_x;
    jxc[2] = 0;
    jxc[3] = -f_p2_x * p0_p2;
    jxc[4] = -f_p2_x * p0_p2 * y0;
    jxc[5] = f_p2_x * (z0 + x0 * p0_p2);
    jxc[6] = -f_p2_x * y0;
    jxc[7] = ft_x_pn * p0_p2;

    jyc[0] = p1_p2 * jfc;
    jyc[1] = 0;
    jyc[2] = f_p2_y;
    jyc[3] = -f_p2_y * p1_p2;
    jyc[4] = -f_p2_y * (z0 + y0 * p1_p2);
    jyc[5] = f_p2_y * x0 * p1_p2;
    jyc[6] = f_p2_y * x0;
    jyc[7] = ft_x_pn * p1_p2;
    JACOBIAN_SET_JC_END
    ///////////////////////////////////
    jxp[0] = f_p2_x * (r[0] - r[6] * p0_p2);
    jxp[1] = f_p2_x * (r[1] - r[7] * p0_p2);
    jxp[2] = f_p2_x * (r[2] - r[8] * p0_p2);
    jyp[0] = f_p2_y * (r[3] - r[6] * p1_p2);
    jyp[1] = f_p2_y * (r[4] - r[7] * p1_p2);
    jyp[2] = f_p2_y * (r[5] - r[8] * p1_p2);
  } else {
    JACOBIAN_SET_JC_BEGIN
    jxc[0] = p0_p2 * jic;
    jxc[1] = f_p2;
    jxc[2] = 0;
    jxc[3] = -f_p2 * p0_p2;
    jxc[4] = -f_p2 * p0_p2 * y0;
    jxc[5] = f_p2 * (z0 + x0 * p0_p2);
    jxc[6] = -f_p2 * y0;

    jyc[0] = p1_p2 * jic;
    jyc[1] = 0;
    jyc[2] = f_p2;
    jyc[3] = -f_p2 * p1_p2;
    jyc[4] = -f_p2 * (z0 + y0 * p1_p2);
    jyc[5] = f_p2 * x0 * p1_p2;
    jyc[6] = f_p2 * x0;

    if (md) {
      float2 ms = tex1Dfetch(tex_jacobian_meas, tidx);
      float msn = (ms.x * ms.x + ms.y * ms.y) * jic;
      jxc[7] = -ms.x * msn;
      jyc[7] = -ms.y * msn;
    } else {
      jxc[7] = 0;
      jyc[7] = 0;
    }
    JACOBIAN_SET_JC_END
    ///////////////////////////////////
    jxp[0] = f_p2 * (r[0] - r[6] * p0_p2);
    jxp[1] = f_p2 * (r[1] - r[7] * p0_p2);
    jxp[2] = f_p2 * (r[2] - r[8] * p0_p2);
    jyp[0] = f_p2 * (r[3] - r[6] * p1_p2);
    jyp[1] = f_p2 * (r[4] - r[7] * p1_p2);
    jyp[2] = f_p2 * (r[5] - r[8] * p1_p2);
  }
}

template <bool md, bool pd>
__device__ void jacobian_camera_internal(int camera_pos, int pt_pos, int tidx,
                                         float* r, float jic, float* jxc,
                                         float* jyc) {
  float m[3];
  float4 ft = tex1Dfetch(tex_jacobian_cam, camera_pos);
  float4 r1 = tex1Dfetch(tex_jacobian_cam, camera_pos + 1);
  r[0] = r1.x;
  r[1] = r1.y;
  r[2] = r1.z;
  r[3] = r1.w;
  float4 r2 = tex1Dfetch(tex_jacobian_cam, camera_pos + 2);
  r[4] = r2.x;
  r[5] = r2.y;
  r[6] = r2.z;
  r[7] = r2.w;
  float4 r3 = tex1Dfetch(tex_jacobian_cam, camera_pos + 3);
  r[8] = r3.x;

  float4 temp = tex1Dfetch(tex_jacobian_pts, pt_pos);
  m[0] = temp.x;
  m[1] = temp.y;
  m[2] = temp.z;

  float x0 = r[0] * m[0] + r[1] * m[1] + r[2] * m[2];
  float y0 = r[3] * m[0] + r[4] * m[1] + r[5] * m[2];
  float z0 = r[6] * m[0] + r[7] * m[1] + r[8] * m[2];
  float f_p2 = FDIV(ft.x, z0 + ft.w);
  float p0_p2 = FDIV(x0 + ft.y, z0 + ft.w);
  float p1_p2 = FDIV(y0 + ft.z, z0 + ft.w);
#ifndef PBA_DISABLE_CONST_CAMERA
  if (r3.w != 0.0f) {
    jxc[0] = 0;
    jxc[1] = 0;
    jxc[2] = 0;
    jxc[3] = 0;
    jxc[4] = 0;
    jxc[5] = 0;
    jxc[6] = 0;
    jxc[7] = 0;
    jyc[0] = 0;
    jyc[1] = 0;
    jyc[2] = 0;
    jyc[3] = 0;
    jyc[4] = 0;
    jyc[5] = 0;
    jyc[6] = 0;
    jyc[7] = 0;
  } else
#endif
      if (pd) {
    float rr1 = r3.y * p0_p2 * p0_p2;
    float rr2 = r3.y * p1_p2 * p1_p2;
    float f_p2_x = f_p2 * (1.0 + 3.0 * rr1 + rr2);
    float f_p2_y = f_p2 * (1.0 + 3.0 * rr2 + rr1);
    float jfc = jic * (1 + rr1 + rr2);
    float ft_x_pn = jic * ft.x * (p0_p2 * p0_p2 + p1_p2 * p1_p2);
    /////////////////////////////////////////////////////
    jxc[0] = p0_p2 * jfc;
    jxc[1] = f_p2_x;
    jxc[2] = 0;
    jxc[3] = -f_p2_x * p0_p2;
    jxc[4] = -f_p2_x * p0_p2 * y0;
    jxc[5] = f_p2_x * (z0 + x0 * p0_p2);
    jxc[6] = -f_p2_x * y0;
    jxc[7] = ft_x_pn * p0_p2;

    jyc[0] = p1_p2 * jfc;
    jyc[1] = 0;
    jyc[2] = f_p2_y;
    jyc[3] = -f_p2_y * p1_p2;
    jyc[4] = -f_p2_y * (z0 + y0 * p1_p2);
    jyc[5] = f_p2_y * x0 * p1_p2;
    jyc[6] = f_p2_y * x0;
    jyc[7] = ft_x_pn * p1_p2;
  } else {
    jxc[0] = p0_p2 * jic;
    jxc[1] = f_p2;
    jxc[2] = 0;
    jxc[3] = -f_p2 * p0_p2;
    jxc[4] = -f_p2 * p0_p2 * y0;
    jxc[5] = f_p2 * (z0 + x0 * p0_p2);
    jxc[6] = -f_p2 * y0;

    jyc[0] = p1_p2 * jic;
    jyc[1] = 0;
    jyc[2] = f_p2;
    jyc[3] = -f_p2 * p1_p2;
    jyc[4] = -f_p2 * (z0 + y0 * p1_p2);
    jyc[5] = f_p2 * x0 * p1_p2;
    jyc[6] = f_p2 * x0;

    if (md) {
      float2 ms = tex1Dfetch(tex_jacobian_meas, tidx);
      float msn = (ms.x * ms.x + ms.y * ms.y) * jic;
      jxc[7] = -ms.x * msn;
      jyc[7] = -ms.y * msn;
    } else {
      jxc[7] = 0;
      jyc[7] = 0;
    }
  }
}

template <bool pd>
__device__ void jacobian_point_internal(int camera_pos, int pt_pos, int tidx,
                                        float* r, float* jxp, float* jyp) {
  float m[3];
  float4 ft = tex1Dfetch(tex_jacobian_cam, camera_pos);
  float4 r1 = tex1Dfetch(tex_jacobian_cam, camera_pos + 1);
  r[0] = r1.x;
  r[1] = r1.y;
  r[2] = r1.z;
  r[3] = r1.w;
  float4 r2 = tex1Dfetch(tex_jacobian_cam, camera_pos + 2);
  r[4] = r2.x;
  r[5] = r2.y;
  r[6] = r2.z;
  r[7] = r2.w;
  float4 r3 = tex1Dfetch(tex_jacobian_cam, camera_pos + 3);
  r[8] = r3.x;

  float4 temp = tex1Dfetch(tex_jacobian_pts, pt_pos);
  m[0] = temp.x;
  m[1] = temp.y;
  m[2] = temp.z;

  float x0 = r[0] * m[0] + r[1] * m[1] + r[2] * m[2];
  float y0 = r[3] * m[0] + r[4] * m[1] + r[5] * m[2];
  float z0 = r[6] * m[0] + r[7] * m[1] + r[8] * m[2];
  float f_p2 = FDIV(ft.x, z0 + ft.w);
  float p0_p2 = FDIV(x0 + ft.y, z0 + ft.w);
  float p1_p2 = FDIV(y0 + ft.z, z0 + ft.w);

  if (pd) {
    float rr1 = r3.y * p0_p2 * p0_p2;
    float rr2 = r3.y * p1_p2 * p1_p2;
    float f_p2_x = f_p2 * (1.0 + 3.0 * rr1 + rr2);
    float f_p2_y = f_p2 * (1.0 + 3.0 * rr2 + rr1);
    ///////////////////////////////////
    jxp[0] = f_p2_x * (r[0] - r[6] * p0_p2);
    jxp[1] = f_p2_x * (r[1] - r[7] * p0_p2);
    jxp[2] = f_p2_x * (r[2] - r[8] * p0_p2);
    jyp[0] = f_p2_y * (r[3] - r[6] * p1_p2);
    jyp[1] = f_p2_y * (r[4] - r[7] * p1_p2);
    jyp[2] = f_p2_y * (r[5] - r[8] * p1_p2);
  } else {
    ///////////////////////////////////
    jxp[0] = f_p2 * (r[0] - r[6] * p0_p2);
    jxp[1] = f_p2 * (r[1] - r[7] * p0_p2);
    jxp[2] = f_p2 * (r[2] - r[8] * p0_p2);
    jyp[0] = f_p2 * (r[3] - r[6] * p1_p2);
    jyp[1] = f_p2 * (r[4] - r[7] * p1_p2);
    jyp[2] = f_p2 * (r[5] - r[8] * p1_p2);
  }
}

template <bool md, bool pd>
__global__ void multiply_jx_noj_kernel(int num, int bwidth, int offset,
                                       float jic, float2* result) {
  int index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  if (index >= num) return;

  __shared__ float data[9 * 64];
  ////////////////////////////////////////////
  int2 proj = tex1Dfetch(tex_mjx_idx, index);
  float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
  float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);
  float4 xp = tex1Dfetch(tex_mjx_x, proj.y + offset);

  ////////////////////////////////////////////
  float jxc[8], jyc[8], jxp[3], jyp[3];
  jacobian_internal<md, pd>(proj.x << 1, proj.y, index, data + 9 * threadIdx.x,
                            jic, jxc, jyc, jxp, jyp);

  /////////////////////////////////////
  result[index] = make_float2(
      jxc[0] * xc1.x + jxc[1] * xc1.y + jxc[2] * xc1.z + jxc[3] * xc1.w +
          jxc[4] * xc2.x + jxc[5] * xc2.y + jxc[6] * xc2.z + jxc[7] * xc2.w +
          jxp[0] * xp.x + jxp[1] * xp.y + jxp[2] * xp.z,
      jyc[0] * xc1.x + jyc[1] * xc1.y + jyc[2] * xc1.z + jyc[3] * xc1.w +
          jyc[4] * xc2.x + jyc[5] * xc2.y + jyc[6] * xc2.z + jyc[7] * xc2.w +
          jyp[0] * xp.x + jyp[1] * xp.y + jyp[2] * xp.z);
}

template <bool md, bool pd>
__global__ void multiply_jcx_noj_kernel(int num, int bwidth, float jic,
                                        float2* result) {
  int index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  if (index >= num) return;

  __shared__ float data[9 * 64];
  ////////////////////////////////////////////
  int2 proj = tex1Dfetch(tex_mjx_idx, index);
  float4 xc1 = tex1Dfetch(tex_mjx_x, proj.x);
  float4 xc2 = tex1Dfetch(tex_mjx_x, proj.x + 1);

  ////////////////////////////////////////////
  float jxc[8], jyc[8];
  jacobian_camera_internal<md, pd>(proj.x << 1, proj.y, index,
                                   data + 9 * threadIdx.x, jic, jxc, jyc);

  /////////////////////////////////////
  result[index] = make_float2(
      jxc[0] * xc1.x + jxc[1] * xc1.y + jxc[2] * xc1.z + jxc[3] * xc1.w +
          jxc[4] * xc2.x + jxc[5] * xc2.y + jxc[6] * xc2.z + jxc[7] * xc2.w,
      jyc[0] * xc1.x + jyc[1] * xc1.y + jyc[2] * xc1.z + jyc[3] * xc1.w +
          jyc[4] * xc2.x + jyc[5] * xc2.y + jyc[6] * xc2.z + jyc[7] * xc2.w);
}

template <bool pd>
__global__ void multiply_jpx_noj_kernel(int num, int bwidth, int offset,
                                        float2* result) {
  int index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  if (index >= num) return;

  __shared__ float data[9 * 64];
  ////////////////////////////////////////////
  int2 proj = tex1Dfetch(tex_mjx_idx, index);
  float4 xp = tex1Dfetch(tex_mjx_x, proj.y + offset);

  ////////////////////////////////////////////
  float jxp[3], jyp[3];
  jacobian_point_internal<pd>(proj.x << 1, proj.y, index,
                              data + 9 * threadIdx.x, jxp, jyp);

  /////////////////////////////////////
  result[index] = make_float2(jxp[0] * xp.x + jxp[1] * xp.y + jxp[2] * xp.z,
                              jyp[0] * xp.x + jyp[1] * xp.y + jyp[2] * xp.z);
}

void ProgramCU::ComputeJX_(CuTexImage& x, CuTexImage& jx, CuTexImage& camera,
                           CuTexImage& point, CuTexImage& meas,
                           CuTexImage& pjmap, bool intrinsic_fixed,
                           int radial_distortion, int mode) {
  unsigned int nproj = pjmap.GetImgWidth();
  unsigned int len = nproj;
  unsigned int bsize = 64;
  unsigned int nblock = (len + bsize - 1) / bsize;
  unsigned int bw, bh;
  int point_offset = camera.GetImgWidth() * 2;
  float jfc = intrinsic_fixed ? 0 : 1.0f;

  /////////////////////////////
  pjmap.BindTexture(tex_mjx_idx);
  x.BindTexture(tex_mjx_x);
  camera.BindTexture(tex_jacobian_cam);
  point.BindTexture(tex_jacobian_pts);

  ///////////////////////////////////
  GetBlockConfiguration(nblock, bw, bh);
  dim3 grid(bw, bh), block(bsize);

  if (mode == 0) {
    if (radial_distortion == -1) {
      meas.BindTexture(tex_jacobian_meas);
      multiply_jx_noj_kernel<true, false><<<grid, block>>>(
          len, (bw * bsize), point_offset, jfc, (float2*)jx.data());
    } else if (radial_distortion) {
      multiply_jx_noj_kernel<false, true><<<grid, block>>>(
          len, (bw * bsize), point_offset, jfc, (float2*)jx.data());
    } else {
      multiply_jx_noj_kernel<false, false><<<grid, block>>>(
          len, (bw * bsize), point_offset, jfc, (float2*)jx.data());
    }

    CheckErrorCUDA("ComputeJX_");
  } else if (mode == 1) {
    if (radial_distortion == -1) {
      meas.BindTexture(tex_jacobian_meas);
      multiply_jcx_noj_kernel<true, false><<<grid, block>>>(
          len, (bw * bsize), jfc, (float2*)jx.data());
    } else if (radial_distortion) {
      multiply_jcx_noj_kernel<false, true><<<grid, block>>>(
          len, (bw * bsize), jfc, (float2*)jx.data());
    } else {
      multiply_jcx_noj_kernel<false, false><<<grid, block>>>(
          len, (bw * bsize), jfc, (float2*)jx.data());
    }

    CheckErrorCUDA("ComputeJCX_");
  } else if (mode == 2) {
    if (radial_distortion == 1) {
      multiply_jpx_noj_kernel<true><<<grid, block>>>(
          len, (bw * bsize), point_offset, (float2*)jx.data());
    } else {
      multiply_jpx_noj_kernel<false><<<grid, block>>>(
          len, (bw * bsize), point_offset, (float2*)jx.data());
    }

    CheckErrorCUDA("ComputeJX_");
  }
}

template <bool md, bool pd, int KH>
__global__ void jte_cam_vec_noj_kernel(int num, int rowsz, float jic,
                                       float* jte) {
  __shared__ float value[KH * 32 * 9];  // 8 * KH * 32
  int cam = blockIdx.x * KH + threadIdx.y + blockIdx.y * rowsz;
  if (cam >= num) return;

  // read data range for this camera
  // 8 thread will do the same thing
  int idx1 = tex1Dfetch(tex_jte_cmp, cam);  // first camera
  int idx2 = tex1Dfetch(tex_jte_cmp, cam + 1);  // last camera + 1

  float* valuec = value + 32 * 9 * threadIdx.y;
  float* rp = valuec + threadIdx.x * 9;
  float rr[8], jxc[8], jyc[8];
  for (int i = 0; i < 8; ++i) rr[i] = 0;

  // loop to read the index of the projection.
  // so to get the location to read the jacobian
  for (int i = idx1 + threadIdx.x; i < idx2; i += 32) {
    int index = tex1Dfetch(tex_jte_cmt, i);
    int2 proj = tex1Dfetch(tex_jacobian_idx, index);
    jacobian_camera_internal<md, pd>(cam << 2, proj.y, index, rp, jic, jxc,
                                     jyc);
    float2 vv = tex1Dfetch(tex_jte_pe, index);
    //
    for (int j = 0; j < 8; ++j) rr[j] += (jxc[j] * vv.x + jyc[j] * vv.y);
  }

  float* valuei = valuec + 8 * threadIdx.x;
  for (int i = 0; i < 8; ++i) valuei[i] = rr[i];
  valuec[threadIdx.x] = (valuec[threadIdx.x] + valuec[threadIdx.x + 32] +
                         valuec[threadIdx.x + 64] + valuec[threadIdx.x + 96] +
                         valuec[threadIdx.x + 128] + valuec[threadIdx.x + 160] +
                         valuec[threadIdx.x + 192] + valuec[threadIdx.x + 224]);
  if (threadIdx.x < 16) valuec[threadIdx.x] += valuec[threadIdx.x + 16];
  if (threadIdx.x < 8)
    valuec[threadIdx.x] = valuec[threadIdx.x] + valuec[threadIdx.x + 8];

  ////////////////////////////////////
  if (threadIdx.x < 8) jte[(cam << 3) + threadIdx.x] = valuec[threadIdx.x];
}

template <bool pd, int KH>
__global__ void jte_point_vec_noj_kernel(int num, int rowsz, float* jte) {
  ////////////////////////////
  __shared__ float value[KH * (9 * 32)];
  int index = blockIdx.x * KH + threadIdx.y + blockIdx.y * rowsz;
  if (index >= num) return;

  int idx1 = tex1Dfetch(tex_jte_pmp, index);  // first
  int idx2 = tex1Dfetch(tex_jte_pmp, index + 1);  // last + 1
  float rx = 0, ry = 0, rz = 0, jxp[3], jyp[3];
  int rowp = threadIdx.y * 9 * 32;
  float* rp = value + threadIdx.x * 9 + rowp;
  for (int i = idx1 + threadIdx.x; i < idx2; i += 32) {
    float2 ev = tex1Dfetch(tex_jte_pe, i);
    int2 proj = tex1Dfetch(tex_jacobian_idx, i);
    jacobian_point_internal<pd>(proj.x << 1, proj.y, i, rp, jxp, jyp);
    rx += (jxp[0] * ev.x + jyp[0] * ev.y);
    ry += (jxp[1] * ev.x + jyp[1] * ev.y);
    rz += (jxp[2] * ev.x + jyp[2] * ev.y);
  }

  int loc = (threadIdx.x << 2) + rowp;
  value[loc] = rx;
  value[loc + 1] = ry;
  value[loc + 2] = rz;
  value[loc + 3] = 0;

  int ridx = threadIdx.x + rowp;
  value[ridx] = ((value[ridx] + value[ridx + 32]) +
                 (value[ridx + 64] + value[ridx + 96]));
  if (threadIdx.x < 16) value[ridx] += value[ridx + 16];
  if (threadIdx.x < 8) value[ridx] += value[ridx + 8];
  if (threadIdx.x < 4)
    jte[(index << 2) + threadIdx.x] = value[ridx] + value[ridx + 4];
}

void ProgramCU::ComputeJtE_(CuTexImage& e, CuTexImage& jte, CuTexImage& camera,
                            CuTexImage& point, CuTexImage& meas,
                            CuTexImage& cmap, CuTexImage& cmlist,
                            CuTexImage& pmap, CuTexImage& pjmap, CuTexImage& jp,
                            bool intrinsic_fixed, int radial_distortion,
                            int mode) {
  pjmap.BindTexture(tex_jacobian_idx);
  camera.BindTexture(tex_jacobian_cam);
  point.BindTexture(tex_jacobian_pts);
  if (radial_distortion) meas.BindTexture(tex_jacobian_meas);

  cmap.BindTexture(tex_jte_cmp);
  cmlist.BindTexture(tex_jte_cmt);
  e.BindTexture(tex_jte_pe);

  //
  unsigned int bw, bh;
  float jfc = intrinsic_fixed ? 0 : 1.0f;
  int ncam = camera.GetImgWidth();
  const int bheight1 = 2, bsize = 32;
  int nblock1 = (ncam + bheight1 - 1) / bheight1;
  GetBlockConfiguration(nblock1, bw, bh);
  dim3 grid(bw, bh), block(bsize, bheight1);
  if (mode == 2) {
  } else if (radial_distortion == -1)
    jte_cam_vec_noj_kernel<true, false, bheight1><<<grid, block>>>(
        ncam, bw * bheight1, jfc, jte.data());
  else if (radial_distortion)
    jte_cam_vec_noj_kernel<false, true, bheight1><<<grid, block>>>(
        ncam, bw * bheight1, jfc, jte.data());
  else
    jte_cam_vec_noj_kernel<false, false, bheight1><<<grid, block>>>(
        ncam, bw * bheight1, jfc, jte.data());
  CheckErrorCUDA("ComputeJtE_<Camera>");

  int npt = point.GetImgWidth();
  unsigned int offsetv = 8 * ncam;
  const int bheight2 = 2, bsize2 = 32;
  int nblock2 = (npt + bheight2 - 1) / bheight2;
  GetBlockConfiguration(nblock2, bw, bh);
  dim3 grid2(bw, bh), block2(bsize2, bheight2);
  if (mode == 1) {
  } else if (jp.IsValid()) {
    pmap.BindTexture(tex_jte_pmp);
    e.BindTexture(tex_jte_pex);
    jp.BindTexture2(tex_jte_jp, tex_jte_jp2);
    if (jp.GetDataSize() > MAX_TEXSIZE)
      jte_point_vec_kernel<bheight2, 2><<<grid2, block2>>>(
          npt, bw * bheight2, jte.data() + offsetv);
    else
      jte_point_vec_kernel<bheight2, 1><<<grid2, block2>>>(
          npt, bw * bheight2, jte.data() + offsetv);
  } else {
    pmap.BindTexture(tex_jte_pmp);
    if (radial_distortion && radial_distortion != -1)
      jte_point_vec_noj_kernel<true, bheight2><<<grid2, block2>>>(
          npt, bw * bheight2, jte.data() + offsetv);
    else
      jte_point_vec_noj_kernel<false, bheight2><<<grid2, block2>>>(
          npt, bw * bheight2, jte.data() + offsetv);
  }
  CheckErrorCUDA("ComputeJtE_<Point>");
}

template <int KH, bool md, bool pd, bool scaling>
__global__ void jtjd_cam_block_noj_kernel(int num, int rowsz, float lambda1,
                                          float lambda2, float jic, float* diag,
                                          float* blocks,
                                          bool add_existing_diagc) {
  const int VN = (md || pd) ? 8 : 7;
  __shared__ float buffer_all[32 * 9 * KH];
  __shared__ float value_all[64 * KH];

  // 8thread per camera
  int bcam = blockIdx.x * KH + blockIdx.y * rowsz;

  int cam = bcam + threadIdx.y;
  if (cam >= num) return;

  float* buffer = buffer_all + threadIdx.y * (32 * 9);
  float* value = value_all + threadIdx.y * 64;

  float jxc[8], jyc[8];
  float* rp = buffer + threadIdx.x * 9;
  float row0[VN], row1[VN - 1], row2[VN - 2], row3[VN - 3];
  float row4[VN - 4], row5[VN - 5], row6[VN - 6], row7[1] = {0};
  // read data range for this camera
  // 8 thread will do the same thing
  int idx1 = tex1Dfetch(tex_jtjd_cmp, cam);  // first camera
  int idx2 = tex1Dfetch(tex_jtjd_cmp, cam + 1);  // last camera + 1

#define REPEAT7(FUNC) \
  FUNC(0);            \
  FUNC(1);            \
  FUNC(2);            \
  FUNC(3);            \
  FUNC(4);            \
  FUNC(5);            \
  FUNC(6);
#define SETZERO(k) \
  for (int j = 0; j < VN - k; ++j) row##k[j] = 0;
  REPEAT7(SETZERO);

  float4 sjv[2];
  if (scaling && (pd || md)) {
    sjv[0] = tex1Dfetch(tex_jacobian_sj, (cam << 1));
    sjv[1] = tex1Dfetch(tex_jacobian_sj, (cam << 1) + 1);
  }

  // loop to read the index of the projection.
  // so to get the location to read the jacobian
  for (int i = idx1 + threadIdx.x; i < idx2; i += 32) {
    /////////////////////////////////////////
    int index = tex1Dfetch(tex_jtjd_cmlist, i);
    int2 proj = tex1Dfetch(tex_jacobian_idx, index);

    ///////////////////////////////////////////////
    jacobian_camera_internal<md, pd>(cam << 2, proj.y, index, rp, jic, jxc,
                                     jyc);

    if (scaling && (pd || md)) {
      float* sj = (float*)sjv;  // 32 threads...64 values
      for (int j = 0; j < VN; ++j) {
        jxc[j] *= sj[j];
        jyc[j] *= sj[j];
      }
    }

////////////////////////////////////////////////
#define ADDROW(k)              \
  for (int j = k; j < VN; ++j) \
  row##k[j - k] += (jxc[k] * jxc[j] + jyc[k] * jyc[j])

    ///////////////
    REPEAT7(ADDROW);
    if (VN == 8) {
      ADDROW(7);
    }
  }

////////////////////////////////////
// make the matrix..//add up the 32 * 8 matrix
#define JTJDSUM8_V1()                                          \
  buffer[threadIdx.x] =                                        \
      (buffer[threadIdx.x] + buffer[threadIdx.x + 32] +        \
       buffer[threadIdx.x + 64] + buffer[threadIdx.x + 96] +   \
       buffer[threadIdx.x + 128] + buffer[threadIdx.x + 160] + \
       buffer[threadIdx.x + 192] + buffer[threadIdx.x + 224]);

#define JTJDSUM8_V2()                                             \
  buffer[threadIdx.x] =                                           \
      (((buffer[threadIdx.x] + buffer[threadIdx.x + 128]) +       \
        (buffer[threadIdx.x + 64] + buffer[threadIdx.x + 192])) + \
       ((buffer[threadIdx.x + 32] + buffer[threadIdx.x + 160]) +  \
        (buffer[threadIdx.x + 96] + buffer[threadIdx.x + 224])));

#define STORE_ROWS(k)                                                        \
  for (int i = 0; i < (VN - k); ++i) bufi[i] = row##k[i];                    \
  JTJDSUM8_V2();                                                             \
  if (threadIdx.x < 16 - k) buffer[threadIdx.x] += buffer[threadIdx.x + 16]; \
  if (threadIdx.x < 8 - k)                                                   \
    value[threadIdx.x + k * 9] = buffer[threadIdx.x] + buffer[threadIdx.x + 8];

  float* bufi = buffer + threadIdx.x * 8;
  REPEAT7(STORE_ROWS);
  if (VN == 8) {
    STORE_ROWS(7);
  }

  /////////////////////////////////////////////////////////////////////////////////////////////

  ////////////////////////////////    (8 * i + j) -> (8 * j + i)
  //#define COPYSYM(i) if(threadIdx.x < VN - i - 1) value[threadIdx.x * 8 +  i *
  //9 + 8] = value[threadIdx.x +  i * 9 + 1];
  if (threadIdx.x < VN - 1) value[threadIdx.x * 8 + 8] = value[threadIdx.x + 1];
  if (threadIdx.x < VN - 2)
    value[threadIdx.x * 8 + 17] = value[threadIdx.x + 10];
  if (threadIdx.x < VN - 3)
    value[threadIdx.x * 8 + 26] = value[threadIdx.x + 19];
  if (threadIdx.x < VN - 4)
    value[threadIdx.x * 8 + 35] = value[threadIdx.x + 28];
  if (threadIdx.x < VN - 5)
    value[threadIdx.x * 8 + 44] = value[threadIdx.x + 37];
  if (threadIdx.x < VN - 6)
    value[threadIdx.x * 8 + 53] = value[threadIdx.x + 46];
  if (VN == 8 && threadIdx.x < VN - 7)
    value[threadIdx.x * 8 + 62] = value[threadIdx.x + 55];

  if (scaling && !pd && !md) {
    float4 sjv[2];
    float* sj = (float*)sjv;  // 32 threads...64 values
    sjv[0] = tex1Dfetch(tex_jacobian_sj, (cam << 1));
    sjv[1] = tex1Dfetch(tex_jacobian_sj, (cam << 1) + 1);
    float sji = sj[threadIdx.x & 0x07];
    value[threadIdx.x] *= (sji * sj[threadIdx.x / 8]);
    value[threadIdx.x + 32] *= (sji * sj[4 + threadIdx.x / 8]);
  }

  bool zero = ((threadIdx.x & 0x7) == VN);

  ///////////write back
  if (threadIdx.x < 8) {
    float* dp = value + threadIdx.x * 9;
    float temp = zero ? 0 : dp[0];
    int didx = threadIdx.x + (cam << 3);
    if (add_existing_diagc) temp += diag[didx];
    diag[didx] = temp;
    dp[0] = lambda1 + lambda2 * temp;
  }
  int wpos = cam * (8 * VN) + threadIdx.x;
  blocks[wpos] = zero ? 0 : value[threadIdx.x];
  if (threadIdx.x < VN * 8 - 32)
    blocks[wpos + 32] = zero ? 0 : value[threadIdx.x + 32];
}

template <int KW, bool pd, bool scaling>
__global__ void jtjd_point_block_noj_kernel(int num, int rowsz, float lambda1,
                                            float lambda2, float4* diag,
                                            float4* blocks, int ptx) {
  ////////////////////////////
  int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * rowsz;
  if (index >= num) return;

  __shared__ float value[KW * 9];
  int idx1 = tex1Dfetch(tex_jtjd_pmp, index);  // first
  int idx2 = tex1Dfetch(tex_jtjd_pmp, index + 1);  // last + 1

  float M00 = 0, M01 = 0, M02 = 0, M11 = 0, M12 = 0, M22 = 0;
  float jxp[3], jyp[3];
  float* rp = value + threadIdx.x * 9;

  float4 sj;
  if (scaling && pd) sj = tex1Dfetch(tex_jacobian_sj, index + ptx);

  for (int i = idx1; i < idx2; ++i) {
    int2 proj = tex1Dfetch(tex_jacobian_idx, i);
    jacobian_point_internal<pd>(proj.x << 1, proj.y, i, rp, jxp, jyp);

    if (scaling && pd) {
      jxp[0] *= sj.x;
      jxp[1] *= sj.y;
      jxp[2] *= sj.z;
      jyp[0] *= sj.x;
      jyp[1] *= sj.y;
      jyp[2] *= sj.z;
    }
    M00 += (jxp[0] * jxp[0] + jyp[0] * jyp[0]);
    M01 += (jxp[0] * jxp[1] + jyp[0] * jyp[1]);
    M02 += (jxp[0] * jxp[2] + jyp[0] * jyp[2]);
    M11 += (jxp[1] * jxp[1] + jyp[1] * jyp[1]);
    M12 += (jxp[1] * jxp[2] + jyp[1] * jyp[2]);
    M22 += (jxp[2] * jxp[2] + jyp[2] * jyp[2]);
  }

  if (scaling && !pd) {
    sj = tex1Dfetch(tex_jacobian_sj, index + ptx);
    M00 *= (sj.x * sj.x);
    M01 *= (sj.x * sj.y);
    M02 *= (sj.x * sj.z);
    M11 *= (sj.y * sj.y);
    M12 *= (sj.y * sj.z);
    M22 *= (sj.z * sj.z);
  }

  diag[index] = make_float4(M00, M11, M22, 0);

  M00 = lambda2 * M00 + lambda1;
  M11 = lambda2 * M11 + lambda1;
  M22 = lambda2 * M22 + lambda1;

  // invert the 3x3 matrix.
  float det = (M00 * M11 - M01 * M01) * M22 + 2.0 * M01 * M12 * M02 -
              M02 * M02 * M11 - M12 * M12 * M00;
  if (det >= FLT_MAX || det <= FLT_MIN * 2.0f) {
    int write_pos = index * 3;
    blocks[write_pos] = make_float4(0, 0, 0, 0);
    blocks[write_pos + 1] = make_float4(0, 0, 0, 0);
    blocks[write_pos + 2] = make_float4(0, 0, 0, 0);
  } else {
    float m00 = (M11 * M22 - M12 * M12) / det;
    float m01 = -(M01 * M22 - M12 * M02) / det;
    float m02 = (M01 * M12 - M02 * M11) / det;
    int write_pos = index * 3;
    blocks[write_pos] = make_float4(m00, m01, m02, 0);

    float m11 = (M00 * M22 - M02 * M02) / det;
    float m12 = -(M00 * M12 - M01 * M02) / det;
    blocks[write_pos + 1] = make_float4(m01, m11, m12, 0);

    float m22 = (M00 * M11 - M01 * M01) / det;
    blocks[write_pos + 2] = make_float4(m02, m12, m22, 0);
  }
}

void ProgramCU::ComputeDiagonalBlock_(
    float lambda, bool dampd, CuTexImage& camera, CuTexImage& point,
    CuTexImage& meas, CuTexImage& cmap, CuTexImage& cmlist, CuTexImage& pmap,
    CuTexImage& jmap, CuTexImage& jp, CuTexImage& sj, CuTexImage& diag,
    CuTexImage& blocks, bool intrinsic_fixed, int radial_distortion,
    bool add_existing_diagc, int mode) {
  float lambda1 = dampd ? 0.0f : lambda;
  float lambda2 = dampd ? (1.0f + lambda) : 1.0f;
  float jfc = intrinsic_fixed ? 0.0f : 1.0f;

  //////////////////////////////////
  jmap.BindTexture(tex_jacobian_idx);
  camera.BindTexture(tex_jacobian_cam);
  point.BindTexture(tex_jacobian_pts);
  cmap.BindTexture(tex_jtjd_cmp);
  cmlist.BindTexture(tex_jtjd_cmlist);

  ////////////////////////////////////////////////////
  const unsigned int bsize1 = 32;
  const unsigned int bheight1 = 2;
  unsigned int ncam = camera.GetImgWidth();  // how many cameras
  unsigned int nblock = (ncam + bheight1 - 1) / bheight1;
  unsigned int bw, bh;
  GetBlockConfiguration(nblock, bw, bh);
  dim3 block1(bsize1, bheight1), grid1(bw, bh);

  ///////////////////////////////////////////////////
  if (radial_distortion == -1) meas.BindTexture(tex_jacobian_meas);
  if (mode == 2) {
    // skip the camera part.
  } else if (sj.IsValid()) {
    sj.BindTexture(tex_jacobian_sj);
    if (radial_distortion == -1)
      jtjd_cam_block_noj_kernel<bheight1, true, false, true><<<grid1, block1>>>(
          ncam, bw * bheight1, lambda1, lambda2, jfc, diag.data(),
          blocks.data(), add_existing_diagc);
    else if (radial_distortion)
      jtjd_cam_block_noj_kernel<bheight1, false, true, true><<<grid1, block1>>>(
          ncam, bw * bheight1, lambda1, lambda2, jfc, diag.data(),
          blocks.data(), add_existing_diagc);
    else
      jtjd_cam_block_noj_kernel<bheight1, false, false,
                                true><<<grid1, block1>>>(
          ncam, bw * bheight1, lambda1, lambda2, jfc, diag.data(),
          blocks.data(), add_existing_diagc);
  } else {
    if (radial_distortion == -1)
      jtjd_cam_block_noj_kernel<bheight1, true, false,
                                false><<<grid1, block1>>>(
          ncam, bw * bheight1, lambda1, lambda2, jfc, diag.data(),
          blocks.data(), add_existing_diagc);
    else if (radial_distortion)
      jtjd_cam_block_noj_kernel<bheight1, false, true,
                                false><<<grid1, block1>>>(
          ncam, bw * bheight1, lambda1, lambda2, jfc, diag.data(),
          blocks.data(), add_existing_diagc);
    else
      jtjd_cam_block_noj_kernel<bheight1, false, false,
                                false><<<grid1, block1>>>(
          ncam, bw * bheight1, lambda1, lambda2, jfc, diag.data(),
          blocks.data(), add_existing_diagc);
  }
  CheckErrorCUDA("ComputeDiagonalBlock_<Camera>");

  ////////////////////////////////////////////////////
  const unsigned int bsize2 = 64;
  unsigned int npoint = point.GetImgWidth();
  unsigned int len2 = npoint;
  unsigned int nblock2 = (len2 + bsize2 - 1) / bsize2;
  unsigned int offsetd = 2 * ncam;
  unsigned int offsetb = (radial_distortion ? 16 : 14) * ncam;
  GetBlockConfiguration(nblock2, bw, bh);
  dim3 grid2(bw, bh), block2(bsize2);
  pmap.BindTexture(tex_jtjd_pmp);

  if (mode == 1) {
  } else if (jp.IsValid()) {
    jp.BindTexture2(tex_jtjd_jp, tex_jtjd_jp2);
    if (jp.GetDataSize() > MAX_TEXSIZE)
      jtjd_point_block_kernel<2><<<grid2, block2>>>(
          len2, (bw * bsize2), lambda1, lambda2,
          ((float4*)diag.data()) + offsetd, ((float4*)blocks.data()) + offsetb);
    else
      jtjd_point_block_kernel<1><<<grid2, block2>>>(
          len2, (bw * bsize2), lambda1, lambda2,
          ((float4*)diag.data()) + offsetd, ((float4*)blocks.data()) + offsetb);
  } else {
    if (sj.IsValid()) {
      sj.BindTexture(tex_jacobian_sj);
      if (radial_distortion && radial_distortion != -1)
        jtjd_point_block_noj_kernel<bsize2, true, true><<<grid2, block2>>>(
            len2, (bw * bsize2), lambda1, lambda2,
            ((float4*)diag.data()) + offsetd,
            ((float4*)blocks.data()) + offsetb, offsetd);
      else
        jtjd_point_block_noj_kernel<bsize2, false, true><<<grid2, block2>>>(
            len2, (bw * bsize2), lambda1, lambda2,
            ((float4*)diag.data()) + offsetd,
            ((float4*)blocks.data()) + offsetb, offsetd);
    } else {
      if (radial_distortion && radial_distortion != -1)
        jtjd_point_block_noj_kernel<bsize2, true, false><<<grid2, block2>>>(
            len2, (bw * bsize2), lambda1, lambda2,
            ((float4*)diag.data()) + offsetd,
            ((float4*)blocks.data()) + offsetb, 0);
      else
        jtjd_point_block_noj_kernel<bsize2, false, false><<<grid2, block2>>>(
            len2, (bw * bsize2), lambda1, lambda2,
            ((float4*)diag.data()) + offsetd,
            ((float4*)blocks.data()) + offsetb, 0);
    }
  }
  CheckErrorCUDA("ComputeDiagonalBlock_<Point>");

  ////////////////////////////////////////////////////
  if (mode != 2) {
    const unsigned int bsize3 = JTJD_BLOCK_CAM_INVERT_KWIDTH;
    unsigned int len3 = ncam * 8;
    unsigned int nblock3 = (len3 + bsize3 - 1) / bsize3;
    dim3 grid3(nblock3), block3(bsize3);
    if (radial_distortion)
      jtjd_cam_block_invert_kernel<8><<<grid3, block3>>>(
          len3, (float4*)blocks.data());
    else
      jtjd_cam_block_invert_kernel<7><<<grid3, block3>>>(
          len3, (float4*)blocks.data());
    CheckErrorCUDA("ComputeDiagonalBlockInverse<Camera>");
  }
}

__global__ void projection_q_kernel(int nproj, int rowsz, float2* pj) {
  ////////////////////////////////
  int tidx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
  if (tidx >= nproj) return;
  int2 proj = tex1Dfetch(tex_projection_idx, tidx);
  float2 wq = tex1Dfetch(tex_projection_mea, tidx);
  ///////////////////////////////////
  float f1 = tex1Dfetch(tex_projection_cam, proj.x * 4).x;
  float r1 = tex1Dfetch(tex_projection_cam, proj.x * 4 + 3).w;
  float f2 = tex1Dfetch(tex_projection_cam, proj.y * 4).x;
  float r2 = tex1Dfetch(tex_projection_cam, proj.y * 4 + 3).w;
  pj[tidx] = make_float2(-wq.x * (f1 - f2), -wq.y * (r1 - r2));
}

void ProgramCU::ComputeProjectionQ(CuTexImage& camera, CuTexImage& qmap,
                                   CuTexImage& qw, CuTexImage& proj,
                                   int offset) {
  ///////////////////////////////////////
  unsigned int len = qmap.GetImgWidth();
  unsigned int bsize = PROJECTION_FRT_KWIDTH;
  unsigned int nblock = (len + bsize - 1) / bsize;
  unsigned int bw, bh;
  GetBlockConfiguration(nblock, bw, bh);
  dim3 grid(bw, bh), block(bsize);

  ///////////////////////////////////////////
  camera.BindTexture(tex_projection_cam);
  qmap.BindTexture(tex_projection_idx);
  qw.BindTexture(tex_projection_mea);

  //////////////////////////////
  projection_q_kernel<<<grid, block>>>(len, bw * bsize,
                                       ((float2*)proj.data()) + offset);
}

template <bool SJ>
__global__ void multiply_jqx_kernel(int num, int bwidth, float2* result) {
  int index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  if (index >= num) return;
  ////////////////////////////////////////////
  int2 proj = tex1Dfetch(tex_mjx_idx, index);
  float2 wq = tex1Dfetch(tex_jacobian_meas, index);
  int idx1 = proj.x * 2, idx2 = proj.y * 2;
  float x11 = tex1Dfetch(tex_mjx_x, idx1).x;
  float x17 = tex1Dfetch(tex_mjx_x, idx1 + 1).w;
  float x21 = tex1Dfetch(tex_mjx_x, idx2).x;
  float x27 = tex1Dfetch(tex_mjx_x, idx2 + 1).w;

  if (SJ) {
    float s11 = tex1Dfetch(tex_jacobian_sj, idx1).x;
    float s17 = tex1Dfetch(tex_jacobian_sj, idx1 + 1).w;
    float s21 = tex1Dfetch(tex_jacobian_sj, idx2).x;
    float s27 = tex1Dfetch(tex_jacobian_sj, idx2 + 1).w;
    result[index] = make_float2((x11 * s11 - x21 * s21) * wq.x,
                                (x17 * s17 - x27 * s27) * wq.y);
  } else {
    result[index] = make_float2((x11 - x21) * wq.x, (x17 - x27) * wq.y);
  }
}

void ProgramCU::ComputeJQX(CuTexImage& x, CuTexImage& qmap, CuTexImage& wq,
                           CuTexImage& sj, CuTexImage& jx, int offset) {
  unsigned int nproj = qmap.GetImgWidth();
  unsigned int len = nproj;
  unsigned int bsize = 64;
  unsigned int nblock = (len + bsize - 1) / bsize;
  unsigned int bw, bh;

  /////////////////////////////
  qmap.BindTexture(tex_mjx_idx);
  x.BindTexture(tex_mjx_x);
  wq.BindTexture(tex_jacobian_meas);

  ///////////////////////////////////
  GetBlockConfiguration(nblock, bw, bh);
  dim3 grid(bw, bh), block(bsize);

  if (sj.IsValid()) {
    sj.BindTexture(tex_jacobian_sj);
    multiply_jqx_kernel<true><<<grid, block>>>(len, (bw * bsize),
                                               ((float2*)jx.data()) + offset);
  } else {
    multiply_jqx_kernel<false><<<grid, block>>>(len, (bw * bsize),
                                                ((float2*)jx.data()) + offset);
  }
}

texture<int2, 1, hipReadModeElementType> tex_jte_q_idx;
texture<float2, 1, hipReadModeElementType> tex_jte_q_w;

template <bool SJ>
__global__ void jte_cam_q_kernel(int num, int bwidth, float* jte) {
  // int cam = blockIdx.x * KH + threadIdx.y + blockIdx.y * rowsz ;
  int index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * bwidth;
  if (index >= num) return;
  int2 indexp = tex1Dfetch(tex_jte_q_idx, index);
  if (indexp.x == -1) return;
  float2 wq = tex1Dfetch(tex_jte_q_w, index);
  float2 e1 = tex1Dfetch(tex_jte_pe, indexp.x);
  float2 e2 = tex1Dfetch(tex_jte_pe, indexp.y);
  int index8 = index << 3;
  if (SJ) {
    float s1 = tex1Dfetch(tex_jacobian_sj, index * 2).x;
    jte[index8] += s1 * wq.x * (e1.x - e2.x);
    float s7 = tex1Dfetch(tex_jacobian_sj, index * 2 + 1).w;
    jte[index8 + 7] += s7 * wq.y * (e1.y - e2.y);
  } else {
    jte[index8] += wq.x * (e1.x - e2.x);
    jte[index8 + 7] += wq.y * (e1.y - e2.y);
  }
}

void ProgramCU::ComputeJQtEC(CuTexImage& pe, CuTexImage& qlist, CuTexImage& wq,
                             CuTexImage& sj, CuTexImage& jte) {
  int ncam = qlist.GetImgWidth();
  const int bsize = 32;
  int nblock = (ncam + bsize - 1) / bsize;
  unsigned int bw, bh;
  GetBlockConfiguration(nblock, bw, bh);
  dim3 grid(bw, bh), block(bsize);

  pe.BindTexture(tex_jte_pe);
  qlist.BindTexture(tex_jte_q_idx);
  wq.BindTexture(tex_jte_q_w);

  if (sj.IsValid()) {
    sj.BindTexture(tex_jacobian_sj);
    jte_cam_q_kernel<true><<<grid, block>>>(ncam, (bw * bsize), jte.data());
  } else {
    jte_cam_q_kernel<false><<<grid, block>>>(ncam, (bw * bsize), jte.data());
  }
}

}  // namespace pba
