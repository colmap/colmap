#include "hip/hip_runtime.h"
// Copyright (c) 2023, ETH Zurich and UNC Chapel Hill.
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
//
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//
//     * Neither the name of ETH Zurich and UNC Chapel Hill nor the names of
//       its contributors may be used to endorse or promote products derived
//       from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
// ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDERS OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
// INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
// ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
// POSSIBILITY OF SUCH DAMAGE.
//
// Author: Johannes L. Schoenberger (jsch-at-demuc-dot-de)

#include "colmap/mvs/gpu_mat_ref_image.h"
#include "colmap/util/cudacc.h"

#include <iostream>

namespace colmap {
namespace mvs {
namespace {

__global__ void FilterKernel(const hipTextureObject_t image_texture,
                             GpuMat<uint8_t> image,
                             GpuMat<float> sum_image,
                             GpuMat<float> squared_sum_image,
                             const int window_radius,
                             const int window_step,
                             const float sigma_spatial,
                             const float sigma_color) {
  const size_t row = blockDim.y * blockIdx.y + threadIdx.y;
  const size_t col = blockDim.x * blockIdx.x + threadIdx.x;
  if (row >= image.GetHeight() || col >= image.GetWidth()) {
    return;
  }

  BilateralWeightComputer bilateral_weight_computer(sigma_spatial, sigma_color);

  const float center_color = tex2D<float>(image_texture, col, row);

  float color_sum = 0.0f;
  float color_squared_sum = 0.0f;
  float bilateral_weight_sum = 0.0f;

  for (int window_row = -window_radius; window_row <= window_radius;
       window_row += window_step) {
    for (int window_col = -window_radius; window_col <= window_radius;
         window_col += window_step) {
      const float color =
          tex2D<float>(image_texture, col + window_col, row + window_row);
      const float bilateral_weight = bilateral_weight_computer.Compute(
          window_row, window_col, center_color, color);
      color_sum += bilateral_weight * color;
      color_squared_sum += bilateral_weight * color * color;
      bilateral_weight_sum += bilateral_weight;
    }
  }

  color_sum /= bilateral_weight_sum;
  color_squared_sum /= bilateral_weight_sum;

  image.Set(row, col, static_cast<uint8_t>(255.0f * center_color));
  sum_image.Set(row, col, color_sum);
  squared_sum_image.Set(row, col, color_squared_sum);
}

}  // namespace

GpuMatRefImage::GpuMatRefImage(const size_t width, const size_t height)
    : height_(height), width_(width) {
  image.reset(new GpuMat<uint8_t>(width, height));
  sum_image.reset(new GpuMat<float>(width, height));
  squared_sum_image.reset(new GpuMat<float>(width, height));
}

void GpuMatRefImage::Filter(const uint8_t* image_data,
                            const size_t window_radius,
                            const size_t window_step,
                            const float sigma_spatial,
                            const float sigma_color) {
  hipTextureDesc texture_desc;
  memset(&texture_desc, 0, sizeof(texture_desc));
  texture_desc.addressMode[0] = hipAddressModeBorder;
  texture_desc.addressMode[1] = hipAddressModeBorder;
  texture_desc.addressMode[2] = hipAddressModeBorder;
  texture_desc.filterMode = hipFilterModePoint;
  texture_desc.readMode = hipReadModeNormalizedFloat;
  texture_desc.normalizedCoords = false;
  auto image_texture = CudaArrayLayeredTexture<uint8_t>::FromHostArray(
      texture_desc, width_, height_, 1, image_data);

  const dim3 block_size(kBlockDimX, kBlockDimY);
  const dim3 grid_size((width_ - 1) / block_size.x + 1,
                       (height_ - 1) / block_size.y + 1);

  FilterKernel<<<grid_size, block_size>>>(image_texture->GetObj(),
                                          *image,
                                          *sum_image,
                                          *squared_sum_image,
                                          window_radius,
                                          window_step,
                                          sigma_spatial,
                                          sigma_color);
  CUDA_SYNC_AND_CHECK();
}

}  // namespace mvs
}  // namespace colmap
