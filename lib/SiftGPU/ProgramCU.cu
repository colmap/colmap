
#include <hip/hip_runtime.h>
////////////////////////////////////////////////////////////////////////////
//	File:		ProgramCU.cu
//	Author:		Changchang Wu
//	Description : implementation of ProgramCU and all CUDA kernels
//
//	Copyright (c) 2007 University of North Carolina at Chapel Hill
//	All Rights Reserved
//
//	Permission to use, copy, modify and distribute this software and its
//	documentation for educational, research and non-profit purposes, without
//	fee, and without a written agreement is hereby granted, provided that the
//	above copyright notice and the following paragraph appear in all copies.
//
//	The University of North Carolina at Chapel Hill make no representations
//	about the suitability of this software for any purpose. It is provided
//	'as is' without express or implied warranty.
//
//	Please send BUG REPORTS to ccwu@cs.unc.edu
//
////////////////////////////////////////////////////////////////////////////

#if defined(CUDA_SIFTGPU_ENABLED)

#include "GL/glew.h"
#include "stdio.h"

#include "CuTexImage.h"
#include "ProgramCU.h"
#include "GlobalUtil.h"

//----------------------------------------------------------------
//Begin SiftGPU setting section.
//////////////////////////////////////////////////////////
#define IMUL(X,Y) __mul24(X,Y)
//#define FDIV(X,Y) ((X)/(Y))
#define FDIV(X,Y) __fdividef(X,Y)

/////////////////////////////////////////////////////////
//filter kernel width range (don't change this)
#define KERNEL_MAX_WIDTH 33
#define KERNEL_MIN_WIDTH 5

//////////////////////////////////////////////////////////
//horizontal filter block size (32, 64, 128, 256, 512)
#define FILTERH_TILE_WIDTH 128
//thread block for vertical filter. FILTERV_BLOCK_WIDTH can be (4, 8 or 16)
#define FILTERV_BLOCK_WIDTH 16
#define FILTERV_BLOCK_HEIGHT 32
//The corresponding image patch for a thread block
#define FILTERV_PIXEL_PER_THREAD 4
#define FILTERV_TILE_WIDTH FILTERV_BLOCK_WIDTH
#define FILTERV_TILE_HEIGHT (FILTERV_PIXEL_PER_THREAD * FILTERV_BLOCK_HEIGHT)


//////////////////////////////////////////////////////////
//thread block size for computing Difference of Gaussian
#define DOG_BLOCK_LOG_DIMX 7
#define DOG_BLOCK_LOG_DIMY 0
#define DOG_BLOCK_DIMX (1 << DOG_BLOCK_LOG_DIMX)
#define DOG_BLOCK_DIMY (1 << DOG_BLOCK_LOG_DIMY)

//////////////////////////////////////////////////////////
//thread block size for keypoint detection
#define KEY_BLOCK_LOG_DIMX 3
#define KEY_BLOCK_LOG_DIMY 3
#define KEY_BLOCK_DIMX (1<<KEY_BLOCK_LOG_DIMX)
#define KEY_BLOCK_DIMY (1<<KEY_BLOCK_LOG_DIMY)
//#define KEY_OFFSET_ONE
//make KEY_BLOCK_LOG_DIMX 4 will make the write coalesced..
//but it seems uncoalesced writes don't affect the speed

//////////////////////////////////////////////////////////
//thread block size for initializing list generation (64, 128, 256, 512 ...)
#define HIST_INIT_WIDTH 128
//thread block size for generating feature list (32, 64, 128, 256, 512, ...)
#define LISTGEN_BLOCK_DIM 128


/////////////////////////////////////////////////////////
//how many keypoint orientations to compute in a block
#define ORIENTATION_COMPUTE_PER_BLOCK 64
//how many keypoint descriptor to compute in a block (2, 4, 8, 16, 32)
#define DESCRIPTOR_COMPUTE_PER_BLOCK	4
#define DESCRIPTOR_COMPUTE_BLOCK_SIZE	(16 * DESCRIPTOR_COMPUTE_PER_BLOCK)
//how many keypoint descriptor to normalized in a block (32, ...)
#define DESCRIPTOR_NORMALIZ_PER_BLOCK	32



///////////////////////////////////////////
//Thread block size for visualization
//(This doesn't affect the speed of computation)
#define BLOCK_LOG_DIM 4
#define BLOCK_DIM (1 << BLOCK_LOG_DIM)

//End SiftGPU setting section.
//----------------------------------------------------------------


__device__ __constant__ float d_kernel[KERNEL_MAX_WIDTH];
texture<float, 1, cudaReadModeElementType> texData;
texture<unsigned char, 1, cudaReadModeNormalizedFloat> texDataB;
texture<float2, 2, cudaReadModeElementType> texDataF2;
texture<float4, 1, cudaReadModeElementType> texDataF4;
texture<int4, 1, cudaReadModeElementType> texDataI4;
texture<int4, 1, cudaReadModeElementType> texDataList;

//template<int i>	 __device__ float Conv(float *data)		{    return Conv<i-1>(data) + data[i]*d_kernel[i];}
//template<>		__device__ float Conv<0>(float *data)	{    return data[0] * d_kernel[0];					}


//////////////////////////////////////////////////////////////
template<int FW> __global__ void FilterH( float* d_result, int width)
{

	const int HALF_WIDTH = FW >> 1;
	const int CACHE_WIDTH = FILTERH_TILE_WIDTH + FW -1;
	const int CACHE_COUNT = 2 + (CACHE_WIDTH - 2)/ FILTERH_TILE_WIDTH;
	__shared__ float data[CACHE_WIDTH];
	const int bcol = IMUL(blockIdx.x, FILTERH_TILE_WIDTH);
	const int col =  bcol + threadIdx.x;
	const int index_min = IMUL(blockIdx.y, width);
	const int index_max = index_min + width - 1;
	int src_index = index_min + bcol - HALF_WIDTH + threadIdx.x;
	int cache_index = threadIdx.x;
	float value = 0;
#pragma unroll
	for(int j = 0; j < CACHE_COUNT; ++j)
	{
		if(cache_index < CACHE_WIDTH)
		{
			int fetch_index = src_index < index_min? index_min : (src_index > index_max ? index_max : src_index);
			data[cache_index] = tex1Dfetch(texData,fetch_index);
			src_index += FILTERH_TILE_WIDTH;
			cache_index += FILTERH_TILE_WIDTH;
		}
	}
	__syncthreads();
	if(col >= width) return;
#pragma unroll
	for(int i = 0; i < FW; ++i)
	{
		value += (data[threadIdx.x + i]* d_kernel[i]);
	}
//	value = Conv<FW-1>(data + threadIdx.x);
	d_result[index_min + col] = value;
}



////////////////////////////////////////////////////////////////////
template<int  FW>  __global__ void FilterV(float* d_result, int width, int height)
{
	const int HALF_WIDTH = FW >> 1;
	const int CACHE_WIDTH = FW + FILTERV_TILE_HEIGHT - 1;
	const int TEMP = CACHE_WIDTH & 0xf;
//add some extra space to avoid bank conflict
#if FILTERV_TILE_WIDTH == 16
	//make the stride 16 * n +/- 1
	const int EXTRA = (TEMP == 1 || TEMP == 0) ? 1 - TEMP : 15 - TEMP;
#elif FILTERV_TILE_WIDTH == 8
	//make the stride 16 * n +/- 2
	const int EXTRA = (TEMP == 2 || TEMP == 1 || TEMP == 0) ? 2 - TEMP : (TEMP == 15? 3 : 14 - TEMP);
#elif FILTERV_TILE_WIDTH == 4
	//make the stride 16 * n +/- 4
	const int EXTRA = (TEMP >=0 && TEMP <=4) ? 4 - TEMP : (TEMP > 12? 20 - TEMP : 12 - TEMP);
#else
#error
#endif
	const int CACHE_TRUE_WIDTH = CACHE_WIDTH + EXTRA;
	const int CACHE_COUNT = (CACHE_WIDTH + FILTERV_BLOCK_HEIGHT - 1) / FILTERV_BLOCK_HEIGHT;
	const int WRITE_COUNT = (FILTERV_TILE_HEIGHT + FILTERV_BLOCK_HEIGHT -1) / FILTERV_BLOCK_HEIGHT;
	__shared__ float data[CACHE_TRUE_WIDTH * FILTERV_TILE_WIDTH];
	const int row_block_first = IMUL(blockIdx.y, FILTERV_TILE_HEIGHT);
	const int col = IMUL(blockIdx.x, FILTERV_TILE_WIDTH) + threadIdx.x;
	const int row_first = row_block_first - HALF_WIDTH;
	const int data_index_max = IMUL(height - 1, width) + col;
	const int cache_col_start = threadIdx.y;
	const int cache_row_start = IMUL(threadIdx.x, CACHE_TRUE_WIDTH);
	int cache_index = cache_col_start + cache_row_start;
	int data_index = IMUL(row_first + cache_col_start, width) + col;

	if(col < width)
	{
#pragma unroll
		for(int i = 0; i < CACHE_COUNT; ++i)
		{
			if(cache_col_start < CACHE_WIDTH - i * FILTERV_BLOCK_HEIGHT)
			{
				int fetch_index = data_index < col ? col : (data_index > data_index_max? data_index_max : data_index);
				data[cache_index + i * FILTERV_BLOCK_HEIGHT] = tex1Dfetch(texData,fetch_index);
				data_index += IMUL(FILTERV_BLOCK_HEIGHT, width);
			}
		}
	}
	__syncthreads();

	if(col >= width) return;

	int row = row_block_first + threadIdx.y;
	int index_start = cache_row_start + threadIdx.y;
#pragma unroll
	for(int i = 0; i < WRITE_COUNT;		++i,
			row += FILTERV_BLOCK_HEIGHT, index_start += FILTERV_BLOCK_HEIGHT)
	{
		if(row < height)
		{
			int index_dest = IMUL(row, width) + col;
			float value = 0;
#pragma unroll
			for(int i = 0; i < FW; ++i)
			{
				value += (data[index_start + i] * d_kernel[i]);
			}
			d_result[index_dest] = value;
		}
	}
}


template<int LOG_SCALE> __global__ void UpsampleKernel(float* d_result, int width)
{
	const int SCALE = (1 << LOG_SCALE), SCALE_MASK = (SCALE - 1);
	const float INV_SCALE = 1.0f / (float(SCALE));
	int col = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	if(col >= width) return;

	int row = blockIdx.y >> LOG_SCALE;
	int index = row * width + col;
	int dst_row = blockIdx.y;
	int dst_idx= (width * dst_row + col) * SCALE;
	int helper = blockIdx.y & SCALE_MASK;
	if (helper)
	{
		float v11 = tex1Dfetch(texData, index);
		float v12 = tex1Dfetch(texData, index + 1);
		index += width;
		float v21 = tex1Dfetch(texData, index);
		float v22 = tex1Dfetch(texData, index + 1);
		float w1 = INV_SCALE * helper, w2 = 1.0 - w1;
		float v1 = (v21 * w1  + w2 * v11);
		float v2 = (v22 * w1  + w2 * v12);
		d_result[dst_idx] = v1;
#pragma unroll
		for(int i = 1; i < SCALE; ++i)
		{
			const float r2 = i * INV_SCALE;
			const float r1 = 1.0f - r2;
			d_result[dst_idx +i] = v1 * r1 + v2 * r2;
		}
	}else
	{
		float v1 = tex1Dfetch(texData, index);
		float v2 = tex1Dfetch(texData, index + 1);
		d_result[dst_idx] = v1;
#pragma unroll
		for(int i = 1; i < SCALE; ++i)
		{
			const float r2 = i * INV_SCALE;
			const float r1 = 1.0f - r2;
			d_result[dst_idx +i] = v1 * r1 + v2 * r2;
		}
	}

}

////////////////////////////////////////////////////////////////////////////////////////
void ProgramCU::SampleImageU(CuTexImage *dst, CuTexImage *src, int log_scale)
{
	int width = src->GetImgWidth(), height = src->GetImgHeight();
	src->BindTexture(texData);
	dim3 grid((width +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH, height << log_scale);
	dim3 block(FILTERH_TILE_WIDTH);
	switch(log_scale)
	{
	case 1 : 	UpsampleKernel<1> <<< grid, block>>> ((float*) dst->_cuData, width);	break;
	case 2 : 	UpsampleKernel<2> <<< grid, block>>> ((float*) dst->_cuData, width);	break;
	case 3 : 	UpsampleKernel<3> <<< grid, block>>> ((float*) dst->_cuData, width);	break;
	default:	break;
	}
}

template<int LOG_SCALE> __global__ void DownsampleKernel(float* d_result, int src_width, int dst_width)
{
	const int dst_col = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	if(dst_col >= dst_width) return;
	const int src_col = min((dst_col << LOG_SCALE), (src_width - 1));
	const int dst_row = blockIdx.y;
	const int src_row = blockIdx.y << LOG_SCALE;
	const int src_idx = IMUL(src_row, src_width) + src_col;
	const int dst_idx = IMUL(dst_width, dst_row) + dst_col;
	d_result[dst_idx] = tex1Dfetch(texData, src_idx);

}

__global__ void DownsampleKernel(float* d_result, int src_width, int dst_width, const int log_scale)
{
	const int dst_col = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	if(dst_col >= dst_width) return;
	const int src_col = min((dst_col << log_scale), (src_width - 1));
	const int dst_row = blockIdx.y;
	const int src_row = blockIdx.y << log_scale;
	const int src_idx = IMUL(src_row, src_width) + src_col;
	const int dst_idx = IMUL(dst_width, dst_row) + dst_col;
	d_result[dst_idx] = tex1Dfetch(texData, src_idx);

}

void ProgramCU::SampleImageD(CuTexImage *dst, CuTexImage *src, int log_scale)
{
	int src_width = src->GetImgWidth(), dst_width = dst->GetImgWidth() ;

	src->BindTexture(texData);
	dim3 grid((dst_width +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH, dst->GetImgHeight());
	dim3 block(FILTERH_TILE_WIDTH);
	switch(log_scale)
	{
	case 1 : 	DownsampleKernel<1> <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width);	break;
	case 2 :	DownsampleKernel<2> <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width);	break;
	case 3 : 	DownsampleKernel<3> <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width);	break;
	default:	DownsampleKernel    <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width, log_scale);
	}
}

__global__ void ChannelReduce_Kernel(float* d_result)
{
	int index = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	d_result[index] = tex1Dfetch(texData, index*4);
}

__global__ void ChannelReduce_Convert_Kernel(float* d_result)
{
	int index = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	float4 rgba = tex1Dfetch(texDataF4, index);
	d_result[index] = 0.299f * rgba.x + 0.587f* rgba.y + 0.114f * rgba.z;
}

void ProgramCU::ReduceToSingleChannel(CuTexImage* dst, CuTexImage* src, int convert_rgb)
{
	int width = src->GetImgWidth(), height = dst->GetImgHeight() ;

	dim3 grid((width * height +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH);
	dim3 block(FILTERH_TILE_WIDTH);
	if(convert_rgb)
	{
		src->BindTexture(texDataF4);
		ChannelReduce_Convert_Kernel<<<grid, block>>>((float*)dst->_cuData);
	}else
	{
		src->BindTexture(texData);
		ChannelReduce_Kernel<<<grid, block>>>((float*)dst->_cuData);
	}
}

__global__ void ConvertByteToFloat_Kernel(float* d_result)
{
	int index = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
	d_result[index] = tex1Dfetch(texDataB, index);
}

void ProgramCU::ConvertByteToFloat(CuTexImage*src, CuTexImage* dst)
{
	int width = src->GetImgWidth(), height = dst->GetImgHeight() ;
	dim3 grid((width * height +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH);
	dim3 block(FILTERH_TILE_WIDTH);
	src->BindTexture(texDataB);
	ConvertByteToFloat_Kernel<<<grid, block>>>((float*)dst->_cuData);
}

void ProgramCU::CreateFilterKernel(float sigma, float* kernel, int& width)
{
	int i, sz = int( ceil( GlobalUtil::_FilterWidthFactor * sigma -0.5) ) ;//
	width = 2*sz + 1;

	if(width > KERNEL_MAX_WIDTH)
	{
		//filter size truncation
		sz = KERNEL_MAX_WIDTH >> 1;
		width =KERNEL_MAX_WIDTH;
	}else if(width < KERNEL_MIN_WIDTH)
	{
		sz = KERNEL_MIN_WIDTH >> 1;
		width =KERNEL_MIN_WIDTH;
	}

	float   rv = 1.0f/(sigma*sigma), v, ksum =0;

	// pre-compute filter
	for( i = -sz ; i <= sz ; ++i)
	{
		kernel[i+sz] =  v = exp(-0.5f * i * i *rv) ;
		ksum += v;
	}

	//normalize the kernel
	rv = 1.0f/ksum;
	for(i = 0; i< width ;i++) kernel[i]*=rv;
}


template<int FW> void ProgramCU::FilterImage(CuTexImage *dst, CuTexImage *src, CuTexImage* buf)
{
	int width = src->GetImgWidth(), height = src->GetImgHeight();

	//horizontal filtering
	src->BindTexture(texData);
	dim3 gridh((width +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH, height);
	dim3 blockh(FILTERH_TILE_WIDTH);
	FilterH<FW><<<gridh, blockh>>>((float*)buf->_cuData, width);
	CheckErrorCUDA("FilterH");

	///vertical filtering
	buf->BindTexture(texData);
	dim3 gridv((width + FILTERV_TILE_WIDTH - 1)/ FILTERV_TILE_WIDTH,  (height + FILTERV_TILE_HEIGHT - 1)/FILTERV_TILE_HEIGHT);
	dim3 blockv(FILTERV_TILE_WIDTH, FILTERV_BLOCK_HEIGHT);
	FilterV<FW><<<gridv, blockv>>>((float*)dst->_cuData, width, height);
	CheckErrorCUDA("FilterV");
}

//////////////////////////////////////////////////////////////////////
// tested on 2048x1500 image, the time on pyramid construction is
// OpenGL version : 18ms
// CUDA version: 28 ms
void ProgramCU::FilterImage(CuTexImage *dst, CuTexImage *src, CuTexImage* buf, float sigma)
{
	float filter_kernel[KERNEL_MAX_WIDTH]; int width;
	CreateFilterKernel(sigma, filter_kernel, width);
	cudaMemcpyToSymbol(d_kernel, filter_kernel, width * sizeof(float), 0, cudaMemcpyHostToDevice);

	switch(width)
	{
		case 5:		FilterImage< 5>(dst, src, buf);	break;
		case 7:		FilterImage< 7>(dst, src, buf);	break;
		case 9:		FilterImage< 9>(dst, src, buf);	break;
		case 11:	FilterImage<11>(dst, src, buf);	break;
		case 13:	FilterImage<13>(dst, src, buf);	break;
		case 15:	FilterImage<15>(dst, src, buf);	break;
		case 17:	FilterImage<17>(dst, src, buf);	break;
		case 19:	FilterImage<19>(dst, src, buf);	break;
		case 21:	FilterImage<21>(dst, src, buf);	break;
		case 23:	FilterImage<23>(dst, src, buf);	break;
		case 25:	FilterImage<25>(dst, src, buf);	break;
		case 27:	FilterImage<27>(dst, src, buf);	break;
		case 29:	FilterImage<29>(dst, src, buf);	break;
		case 31:	FilterImage<31>(dst, src, buf);	break;
		case 33:	FilterImage<33>(dst, src, buf);	break;
		default:	break;
	}

}


texture<float, 1, cudaReadModeElementType> texC;
texture<float, 1, cudaReadModeElementType> texP;
texture<float, 1, cudaReadModeElementType> texN;

void __global__ ComputeDOG_Kernel(float* d_dog, float2* d_got, int width, int height)
{
	int row = (blockIdx.y << DOG_BLOCK_LOG_DIMY) + threadIdx.y;
	int col = (blockIdx.x << DOG_BLOCK_LOG_DIMX) + threadIdx.x;
	if(col < width && row < height)
	{
		int index = IMUL(row, width) + col;
		float vp = tex1Dfetch(texP, index);
		float v = tex1Dfetch(texC, index);
		d_dog[index] = v - vp;
		float vxn = tex1Dfetch(texC, index + 1);
		float vxp = tex1Dfetch(texC, index - 1);
		float vyp = tex1Dfetch(texC, index - width);
		float vyn = tex1Dfetch(texC, index + width);
		float dx = vxn - vxp, dy = vyn - vyp;
		float grd = 0.5f * sqrt(dx * dx  + dy * dy);
		float rot = (grd == 0.0f? 0.0f : atan2(dy, dx));
		d_got[index] = make_float2(grd, rot);
	}
}

void __global__ ComputeDOG_Kernel(float* d_dog, int width, int height)
{
	int row = (blockIdx.y << DOG_BLOCK_LOG_DIMY) + threadIdx.y;
	int col = (blockIdx.x << DOG_BLOCK_LOG_DIMX) + threadIdx.x;
	if(col < width && row < height)
	{
		int index = IMUL(row, width) + col;
		float vp = tex1Dfetch(texP, index);
		float v = tex1Dfetch(texC, index);
		d_dog[index] = v - vp;
	}
}

void ProgramCU::ComputeDOG(CuTexImage* gus, CuTexImage* dog, CuTexImage* got)
{
	int width = gus->GetImgWidth(), height = gus->GetImgHeight();
	dim3 grid((width + DOG_BLOCK_DIMX - 1)/ DOG_BLOCK_DIMX,  (height + DOG_BLOCK_DIMY - 1)/DOG_BLOCK_DIMY);
	dim3 block(DOG_BLOCK_DIMX, DOG_BLOCK_DIMY);
	gus->BindTexture(texC);
	(gus -1)->BindTexture(texP);
	if(got->_cuData)
		ComputeDOG_Kernel<<<grid, block>>>((float*) dog->_cuData, (float2*) got->_cuData, width, height);
	else
		ComputeDOG_Kernel<<<grid, block>>>((float*) dog->_cuData, width, height);
}


#define READ_CMP_DOG_DATA(datai, tex, idx) \
		datai[0] = tex1Dfetch(tex, idx - 1);\
		datai[1] = tex1Dfetch(tex, idx);\
		datai[2] = tex1Dfetch(tex, idx + 1);\
		if(v > nmax)\
		{\
			   nmax = max(nmax, datai[0]);\
			   nmax = max(nmax, datai[1]);\
			   nmax = max(nmax, datai[2]);\
			   if(v < nmax) goto key_finish;\
		}else\
		{\
			   nmin = min(nmin, datai[0]);\
			   nmin = min(nmin, datai[1]);\
			   nmin = min(nmin, datai[2]);\
			   if(v > nmin) goto key_finish;\
		}


void __global__ ComputeKEY_Kernel(float4* d_key, int width, int colmax, int rowmax,
					float dog_threshold0,  float dog_threshold, float edge_threshold, int subpixel_localization)
{
       float data[3][3], v;
       float datap[3][3], datan[3][3];
#ifdef KEY_OFFSET_ONE
       int row = (blockIdx.y << KEY_BLOCK_LOG_DIMY) + threadIdx.y + 1;
       int col = (blockIdx.x << KEY_BLOCK_LOG_DIMX) + threadIdx.x + 1;
#else
       int row = (blockIdx.y << KEY_BLOCK_LOG_DIMY) + threadIdx.y;
       int col = (blockIdx.x << KEY_BLOCK_LOG_DIMX) + threadIdx.x;
#endif
       int index = IMUL(row, width) + col;
	   int idx[3] ={index - width, index, index + width};
       int in_image =0;
       float nmax, nmin, result = 0.0f;
	   float dx = 0, dy = 0, ds = 0;
	   bool offset_test_passed = true;
#ifdef KEY_OFFSET_ONE
       if(row < rowmax && col < colmax)
#else
       if(row > 0 && col > 0 && row < rowmax && col < colmax)
#endif
       {
			in_image = 1;
			data[1][1] = v = tex1Dfetch(texC, idx[1]);
			if(fabs(v) <= dog_threshold0) goto key_finish;

			data[1][0] = tex1Dfetch(texC, idx[1] - 1);
			data[1][2] = tex1Dfetch(texC, idx[1] + 1);
			nmax = max(data[1][0], data[1][2]);
			nmin = min(data[1][0], data[1][2]);

			if(v <=nmax && v >= nmin) goto key_finish;
			//if((v > nmax && v < 0 )|| (v < nmin && v > 0)) goto key_finish;
			READ_CMP_DOG_DATA(data[0], texC, idx[0]);
			READ_CMP_DOG_DATA(data[2], texC, idx[2]);

			//edge supression
			float vx2 = v * 2.0f;
			float fxx = data[1][0] + data[1][2] - vx2;
			float fyy = data[0][1] + data[2][1] - vx2;
			float fxy = 0.25f * (data[2][2] + data[0][0] - data[2][0] - data[0][2]);
			float temp1 = fxx * fyy - fxy * fxy;
			float temp2 = (fxx + fyy) * (fxx + fyy);
			if(temp1 <=0 || temp2 > edge_threshold * temp1) goto key_finish;


			//read the previous level
			READ_CMP_DOG_DATA(datap[0], texP, idx[0]);
			READ_CMP_DOG_DATA(datap[1], texP, idx[1]);
			READ_CMP_DOG_DATA(datap[2], texP, idx[2]);


			//read the next level
			READ_CMP_DOG_DATA(datan[0], texN, idx[0]);
			READ_CMP_DOG_DATA(datan[1], texN, idx[1]);
			READ_CMP_DOG_DATA(datan[2], texN, idx[2]);

			if(subpixel_localization)
			{
				//subpixel localization
				float fx = 0.5f * (data[1][2] - data[1][0]);
				float fy = 0.5f * (data[2][1] - data[0][1]);
				float fs = 0.5f * (datan[1][1] - datap[1][1]);

				float fss = (datan[1][1] + datap[1][1] - vx2);
				float fxs = 0.25f* (datan[1][2] + datap[1][0] - datan[1][0] - datap[1][2]);
				float fys = 0.25f* (datan[2][1] + datap[0][1] - datan[0][1] - datap[2][1]);

				//need to solve dx, dy, ds;
				// |-fx|     | fxx fxy fxs |   |dx|
				// |-fy|  =  | fxy fyy fys | * |dy|
				// |-fs|     | fxs fys fss |   |ds|
				float4 A0 = fxx > 0? make_float4(fxx, fxy, fxs, -fx) : make_float4(-fxx, -fxy, -fxs, fx);
				float4 A1 = fxy > 0? make_float4(fxy, fyy, fys, -fy) : make_float4(-fxy, -fyy, -fys, fy);
				float4 A2 = fxs > 0? make_float4(fxs, fys, fss, -fs) : make_float4(-fxs, -fys, -fss, fs);
				float maxa = max(max(A0.x, A1.x), A2.x);
				if(maxa >= 1e-10)
				{
					if(maxa == A1.x)
					{
						float4 TEMP = A1; A1 = A0; A0 = TEMP;
					}else if(maxa == A2.x)
					{
						float4 TEMP = A2; A2 = A0; A0 = TEMP;
					}
					A0.y /= A0.x;	A0.z /= A0.x;	A0.w/= A0.x;
					A1.y -= A1.x * A0.y;	A1.z -= A1.x * A0.z;	A1.w -= A1.x * A0.w;
					A2.y -= A2.x * A0.y;	A2.z -= A2.x * A0.z;	A2.w -= A2.x * A0.w;
					if(abs(A2.y) > abs(A1.y))
					{
						float4 TEMP = A2;	A2 = A1; A1 = TEMP;
					}
					if(abs(A1.y) >= 1e-10)
					{
						A1.z /= A1.y;	A1.w /= A1.y;
						A2.z -= A2.y * A1.z;	A2.w -= A2.y * A1.w;
						if(abs(A2.z) >= 1e-10)
						{
							ds = A2.w / A2.z;
							dy = A1.w - ds * A1.z;
							dx = A0.w - ds * A0.z - dy * A0.y;

							offset_test_passed =
								fabs(data[1][1] + 0.5f * (dx * fx + dy * fy + ds * fs)) > dog_threshold
								&&fabs(ds) < 1.0f && fabs(dx) < 1.0f && fabs(dy) < 1.0f;
						}
					}
				}
			}
			if(offset_test_passed) result = v > nmax ? 1.0 : -1.0;
       }
key_finish:
       if(in_image) d_key[index] = make_float4(result, dx, dy, ds);
}


void ProgramCU::ComputeKEY(CuTexImage* dog, CuTexImage* key, float Tdog, float Tedge)
{
	int width = dog->GetImgWidth(), height = dog->GetImgHeight();
	float Tdog1 = (GlobalUtil::_SubpixelLocalization? 0.8f : 1.0f) * Tdog;
	CuTexImage* dogp = dog - 1;
	CuTexImage* dogn = dog + 1;
#ifdef KEY_OFFSET_ONE
	dim3 grid((width - 1 + KEY_BLOCK_DIMX - 1)/ KEY_BLOCK_DIMX,  (height - 1 + KEY_BLOCK_DIMY - 1)/KEY_BLOCK_DIMY);
#else
	dim3 grid((width + KEY_BLOCK_DIMX - 1)/ KEY_BLOCK_DIMX,  (height + KEY_BLOCK_DIMY - 1)/KEY_BLOCK_DIMY);
#endif
	dim3 block(KEY_BLOCK_DIMX, KEY_BLOCK_DIMY);
	dogp->BindTexture(texP);
	dog ->BindTexture(texC);
	dogn->BindTexture(texN);
	Tedge = (Tedge+1)*(Tedge+1)/Tedge;
	ComputeKEY_Kernel<<<grid, block>>>((float4*) key->_cuData, width,
        width -1, height -1, Tdog1, Tdog, Tedge, GlobalUtil::_SubpixelLocalization);

}



void __global__ InitHist_Kernel(int4* hist, int ws, int wd, int height)
{
       int row = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;
       int col = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	   if(row < height && col < wd)
	   {
			int hidx = IMUL(row, wd) + col;
			int scol = col << 2;
			int sidx = IMUL(row, ws) + scol;
			int v[4] = {0, 0, 0, 0};
			if(row > 0 && row < height -1)
			{
#pragma unroll
				for(int i = 0; i < 4 ; ++i, ++scol)
				{
					float4 temp = tex1Dfetch(texDataF4, sidx +i);
					v[i] = (scol < ws -1 && scol > 0 && temp.x!=0) ? 1 : 0;
				}
			}
			hist[hidx] = make_int4(v[0], v[1], v[2], v[3]);

	   }
}



void ProgramCU::InitHistogram(CuTexImage* key, CuTexImage* hist)
{
	int ws = key->GetImgWidth(), hs = key->GetImgHeight();
	int wd = hist->GetImgWidth(), hd = hist->GetImgHeight();
	dim3 grid((wd  + HIST_INIT_WIDTH - 1)/ HIST_INIT_WIDTH,  hd);
	dim3 block(HIST_INIT_WIDTH, 1);
	key->BindTexture(texDataF4);
	InitHist_Kernel<<<grid, block>>>((int4*) hist->_cuData, ws, wd, hd);
}



void __global__ ReduceHist_Kernel(int4* d_hist, int ws, int wd, int height)
{
       int row = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;
       int col = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	   if(row < height && col < wd)
	   {
			int hidx = IMUL(row, wd) + col;
			int scol = col << 2;
			int sidx = IMUL(row, ws) + scol;
			int v[4] = {0, 0, 0, 0};
#pragma unroll
			for(int i = 0; i < 4 && scol < ws; ++i, ++scol)
			{
				int4 temp = tex1Dfetch(texDataI4, sidx + i);
				v[i] = temp.x + temp.y + temp.z + temp.w;
			}
			d_hist[hidx] = make_int4(v[0], v[1], v[2], v[3]);
	   }
}

void ProgramCU::ReduceHistogram(CuTexImage*hist1, CuTexImage* hist2)
{
	int ws = hist1->GetImgWidth(), hs = hist1->GetImgHeight();
	int wd = hist2->GetImgWidth(), hd = hist2->GetImgHeight();
	int temp = (int)floorf(logf(float(wd * 2/ 3)) / logf(2.0f));
	const int wi = min(7, max(temp , 0));
	hist1->BindTexture(texDataI4);

	const int BW = 1 << wi, BH =  1 << (7 - wi);
	dim3 grid((wd  + BW - 1)/ BW,  (hd + BH -1) / BH);
	dim3 block(BW, BH);
	ReduceHist_Kernel<<<grid, block>>>((int4*)hist2->_cuData, ws, wd, hd);
}


void __global__ ListGen_Kernel(int4* d_list, int list_len, int width)
{
	int idx1 = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
    int4 pos = tex1Dfetch(texDataList, idx1);
	int idx2 = IMUL(pos.y, width) + pos.x;
	int4 temp = tex1Dfetch(texDataI4, idx2);
	int  sum1 = temp.x + temp.y;
	int  sum2 = sum1 + temp.z;
	pos.x <<= 2;
	if(pos.z >= sum2)
	{
		pos.x += 3;
		pos.z -= sum2;
	}else if(pos.z >= sum1)
	{
		pos.x += 2;
		pos.z -= sum1;
	}else if(pos.z >= temp.x)
	{
		pos.x += 1;
		pos.z -= temp.x;
	}
  if (idx1 < list_len) {
    d_list[idx1] = pos;
  }
}

//input list (x, y) (x, y) ....
void ProgramCU::GenerateList(CuTexImage* list, CuTexImage* hist)
{
	int len = list->GetImgWidth();
	list->BindTexture(texDataList);
	hist->BindTexture(texDataI4);
	dim3  grid((len + LISTGEN_BLOCK_DIM -1) /LISTGEN_BLOCK_DIM);
	dim3  block(LISTGEN_BLOCK_DIM);
	ListGen_Kernel<<<grid, block>>>((int4*) list->_cuData, len,
                                  hist->GetImgWidth());
}

void __global__ ComputeOrientation_Kernel(float4* d_list,
										  int list_len,
										  int width, int height,
										  float sigma, float sigma_step,
										  float gaussian_factor, float sample_factor,
										  int num_orientation,
										  int existing_keypoint,
										  int subpixel,
										  int keepsign)
{
	const float ten_degree_per_radius = 5.7295779513082320876798154814105;
	const float radius_per_ten_degrees = 1.0 / 5.7295779513082320876798154814105;
	int idx = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	if(idx >= list_len) return;
	float4 key;
	if(existing_keypoint)
	{
		key = tex1Dfetch(texDataF4, idx);
	}else
	{
		int4 ikey = tex1Dfetch(texDataList, idx);
		key.x = ikey.x + 0.5f;
		key.y = ikey.y + 0.5f;
		key.z = sigma;
		if(subpixel || keepsign)
		{
			float4 offset = tex1Dfetch(texDataF4, IMUL(width, ikey.y) + ikey.x);
			if(subpixel)
			{
				key.x += offset.y;
				key.y += offset.z;
				key.z *= pow(sigma_step, offset.w);
			}
			if(keepsign) key.z *= offset.x;
		}
	}
	if(num_orientation == 0)
	{
		key.w = 0;
		d_list[idx] = key;
		return;
	}
	float vote[37];
	float gsigma = key.z * gaussian_factor;
	float win = fabs(key.z) * sample_factor;
	float dist_threshold = win * win + 0.5;
	float factor = -0.5f / (gsigma * gsigma);
	float xmin = max(1.5f, floorf(key.x - win) + 0.5f);
	float ymin = max(1.5f, floorf(key.y - win) + 0.5f);
	float xmax = min(width - 1.5f, floorf(key.x + win) + 0.5f);
	float ymax = min(height -1.5f, floorf(key.y + win) + 0.5f);
#pragma unroll
	for(int i = 0; i < 36; ++i) vote[i] = 0.0f;
	for(float y = ymin; y <= ymax; y += 1.0f)
	{
		for(float x = xmin; x <= xmax; x += 1.0f)
		{
			float dx = x - key.x;
			float dy = y - key.y;
			float sq_dist  = dx * dx + dy * dy;
			if(sq_dist >= dist_threshold) continue;
			float2 got = tex2D(texDataF2, x, y);
			float weight = got.x * exp(sq_dist * factor);
			float fidx = floorf(got.y * ten_degree_per_radius);
			int oidx = fidx;
			if(oidx < 0) oidx += 36;
			vote[oidx] += weight;
		}
	}

	//filter the vote

	const float one_third = 1.0 /3.0;
#pragma unroll
	for(int i = 0; i < 6; ++i)
	{
		vote[36] = vote[0];
		float pre = vote[35];
#pragma unroll
		for(int j = 0; j < 36; ++j)
		{
			float temp = one_third * (pre + vote[j] + vote[j + 1]);
			pre = vote[j];			vote[j] = temp;
		}
	}

	vote[36] = vote[0];
	if(num_orientation == 1 || existing_keypoint)
	{
		int index_max = 0;
		float max_vote = vote[0];
#pragma unroll
		for(int i = 1; i < 36; ++i)
		{
			index_max =  vote[i] > max_vote? i : index_max;
			max_vote = max(max_vote, vote[i]);
		}
		float pre = vote[index_max == 0? 35 : index_max -1];
		float next = vote[index_max + 1];
		float weight = max_vote;
		float off =  0.5f * FDIV(next - pre, weight + weight - next - pre);
		key.w = radius_per_ten_degrees * (index_max + 0.5f + off);
		d_list[idx] = key;

	}else
	{
		float max_vote = vote[0];
#pragma unroll
		for(int i = 1; i < 36; ++i)		max_vote = max(max_vote, vote[i]);

		float vote_threshold = max_vote * 0.8f;
		float pre = vote[35];
		float max_rot[2], max_vot[2] = {0, 0};
		int  ocount = 0;
#pragma unroll
		for(int i =0; i < 36; ++i)
		{
			float next = vote[i + 1];
			if(vote[i] > vote_threshold && vote[i] > pre && vote[i] > next)
			{
				float di = 0.5f * FDIV(next - pre, vote[i] + vote[i] - next - pre);
				float rot = i + di + 0.5f;
				float weight = vote[i];
				///
				if(weight > max_vot[1])
				{
					if(weight > max_vot[0])
					{
						max_vot[1] = max_vot[0];
						max_rot[1] = max_rot[0];
						max_vot[0] = weight;
						max_rot[0] = rot;
					}
					else
					{
						max_vot[1] = weight;
						max_rot[1] = rot;
					}
					ocount ++;
				}
			}
			pre = vote[i];
		}
		float fr1 = max_rot[0] / 36.0f;
		if(fr1 < 0) fr1 += 1.0f;
		unsigned short us1 = ocount == 0? 65535 : ((unsigned short )floorf(fr1 * 65535.0f));
		unsigned short us2 = 65535;
		if(ocount > 1)
		{
			float fr2 = max_rot[1] / 36.0f;
			if(fr2 < 0) fr2 += 1.0f;
			us2 = (unsigned short ) floorf(fr2 * 65535.0f);
		}
		unsigned int uspack = (us2 << 16) | us1;
		key.w = __int_as_float(uspack);
		d_list[idx] = key;
	}

}




void ProgramCU::ComputeOrientation(CuTexImage* list, CuTexImage* got, CuTexImage*key,
								   float sigma, float sigma_step, int existing_keypoint)
{
	int len = list->GetImgWidth();
	if(len <= 0) return;
	int width = got->GetImgWidth(), height = got->GetImgHeight();
	if(existing_keypoint)
	{
		list->BindTexture(texDataF4);
	}else
	{
		list->BindTexture(texDataList);
		if(GlobalUtil::_SubpixelLocalization) key->BindTexture(texDataF4);
	}
	got->BindTexture2D(texDataF2);

	const int block_width = len < ORIENTATION_COMPUTE_PER_BLOCK ? 16 : ORIENTATION_COMPUTE_PER_BLOCK;
	dim3 grid((len + block_width -1) / block_width);
	dim3 block(block_width);

	ComputeOrientation_Kernel<<<grid, block>>>((float4*) list->_cuData,
		len, width, height, sigma, sigma_step,
		GlobalUtil::_OrientationGaussianFactor,
		GlobalUtil::_OrientationGaussianFactor * GlobalUtil::_OrientationWindowFactor,
		GlobalUtil::_FixedOrientation? 0 : GlobalUtil::_MaxOrientation,
		existing_keypoint, GlobalUtil::_SubpixelLocalization, GlobalUtil::_KeepExtremumSign);

	ProgramCU::CheckErrorCUDA("ComputeOrientation");
}

template <bool DYNAMIC_INDEXING> void __global__ ComputeDescriptor_Kernel(float4* d_des, int num,
											 int width, int height, float window_factor)
{
	const float rpi = 4.0/ 3.14159265358979323846;
	int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	int fidx = idx >> 4;
	if(fidx >= num) return;
	float4 key = tex1Dfetch(texDataF4, fidx);
	int bidx = idx& 0xf, ix = bidx & 0x3, iy = bidx >> 2;
	float spt = fabs(key.z * window_factor);
	float s, c; __sincosf(key.w, &s, &c);
	float anglef = key.w > 3.14159265358979323846? key.w - (2.0 * 3.14159265358979323846) : key.w ;
	float cspt = c * spt, sspt = s * spt;
	float crspt = c / spt, srspt = s / spt;
	float2 offsetpt, pt;
	float xmin, ymin, xmax, ymax, bsz;
	offsetpt.x = ix - 1.5f;
	offsetpt.y = iy - 1.5f;
	pt.x = cspt * offsetpt.x - sspt * offsetpt.y + key.x;
	pt.y = cspt * offsetpt.y + sspt * offsetpt.x + key.y;
	bsz =  fabs(cspt) + fabs(sspt);
	xmin = max(1.5f, floorf(pt.x - bsz) + 0.5f);
	ymin = max(1.5f, floorf(pt.y - bsz) + 0.5f);
	xmax = min(width - 1.5f, floorf(pt.x + bsz) + 0.5f);
	ymax = min(height - 1.5f, floorf(pt.y + bsz) + 0.5f);
	float des[9];
#pragma unroll
	for(int i =0; i < 9; ++i) des[i] = 0.0f;
	for(float y = ymin; y <= ymax; y += 1.0f)
	{
		for(float x = xmin; x <= xmax; x += 1.0f)
		{
			float dx = x - pt.x;
			float dy = y - pt.y;
			float nx = crspt * dx + srspt * dy;
			float ny = crspt * dy - srspt * dx;
			float nxn = fabs(nx);
			float nyn = fabs(ny);
			if(nxn < 1.0f && nyn < 1.0f)
			{
				float2 cc = tex2D(texDataF2, x, y);
				float dnx = nx + offsetpt.x;
				float dny = ny + offsetpt.y;
				float ww = exp(-0.125f * (dnx * dnx + dny * dny));
				float wx = 1.0 - nxn;
				float wy = 1.0 - nyn;
				float weight = ww * wx * wy * cc.x;
				float theta = (anglef - cc.y) * rpi;
				if(theta < 0) theta += 8.0f;
				float fo = floorf(theta);
				int fidx = fo;
				float weight1 = fo + 1.0f  - theta;
				float weight2 = theta - fo;
				if(DYNAMIC_INDEXING)
				{
					des[fidx] += (weight1 * weight);
					des[fidx + 1] += (weight2 * weight);
					//this dynamic indexing part might be slow
				}else
				{
					#pragma unroll
					for(int k = 0; k < 8; ++k)
					{
						if(k == fidx)
						{
							des[k] += (weight1 * weight);
							des[k+1] += (weight2 * weight);
						}
					}
				}
			}
		}
	}
	des[0] += des[8];

	int didx = idx << 1;
	d_des[didx] = make_float4(des[0], des[1], des[2], des[3]);
	d_des[didx+1] = make_float4(des[4], des[5], des[6], des[7]);
}


template <bool DYNAMIC_INDEXING> void __global__ ComputeDescriptorRECT_Kernel(float4* d_des, int num,
											 int width, int height, float window_factor)
{
	const float rpi = 4.0/ 3.14159265358979323846;
	int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	int fidx = idx >> 4;
	if(fidx >= num) return;
	float4 key = tex1Dfetch(texDataF4, fidx);
	int bidx = idx& 0xf, ix = bidx & 0x3, iy = bidx >> 2;
    //float aspect_ratio = key.w / key.z;
    //float aspect_sq = aspect_ratio * aspect_ratio;
	float sptx = key.z * 0.25, spty = key.w * 0.25;
	float xmin, ymin, xmax, ymax; float2 pt;
	pt.x = sptx * (ix + 0.5f)  + key.x;
	pt.y = spty * (iy + 0.5f)  + key.y;
	xmin = max(1.5f, floorf(pt.x - sptx) + 0.5f);
	ymin = max(1.5f, floorf(pt.y - spty) + 0.5f);
	xmax = min(width - 1.5f, floorf(pt.x + sptx) + 0.5f);
	ymax = min(height - 1.5f, floorf(pt.y + spty) + 0.5f);
	float des[9];
#pragma unroll
	for(int i =0; i < 9; ++i) des[i] = 0.0f;
	for(float y = ymin; y <= ymax; y += 1.0f)
	{
		for(float x = xmin; x <= xmax; x += 1.0f)
		{
			float nx = (x - pt.x) / sptx;
			float ny = (y - pt.y) / spty;
			float nxn = fabs(nx);
			float nyn = fabs(ny);
			if(nxn < 1.0f && nyn < 1.0f)
			{
				float2 cc = tex2D(texDataF2, x, y);
				float wx = 1.0 - nxn;
				float wy = 1.0 - nyn;
				float weight =  wx * wy * cc.x;
				float theta = (- cc.y) * rpi;
				if(theta < 0) theta += 8.0f;
				float fo = floorf(theta);
				int fidx = fo;
				float weight1 = fo + 1.0f  - theta;
				float weight2 = theta - fo;
				if(DYNAMIC_INDEXING)
				{
					des[fidx] += (weight1 * weight);
					des[fidx + 1] += (weight2 * weight);
					//this dynamic indexing part might be slow
				}else
				{
					#pragma unroll
					for(int k = 0; k < 8; ++k)
					{
						if(k == fidx)
						{
							des[k] += (weight1 * weight);
							des[k+1] += (weight2 * weight);
						}
					}
				}
			}
		}
	}
	des[0] += des[8];

	int didx = idx << 1;
	d_des[didx] = make_float4(des[0], des[1], des[2], des[3]);
	d_des[didx+1] = make_float4(des[4], des[5], des[6], des[7]);
}

void __global__ NormalizeDescriptor_Kernel(float4* d_des, int num)
{
	float4 temp[32];
	int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	if(idx >= num) return;
	int sidx = idx << 5;
	float norm1 = 0, norm2 = 0;
#pragma unroll
	for(int i = 0; i < 32; ++i)
	{
		temp[i] = tex1Dfetch(texDataF4, sidx +i);
		norm1 += (temp[i].x * temp[i].x + temp[i].y * temp[i].y +
				 temp[i].z * temp[i].z + temp[i].w * temp[i].w);
	}
	norm1 = rsqrt(norm1);

#pragma unroll
	for(int i = 0; i < 32; ++i)
	{
		temp[i].x = min(0.2f, temp[i].x * norm1);
		temp[i].y = min(0.2f, temp[i].y * norm1);
		temp[i].z = min(0.2f, temp[i].z * norm1);
		temp[i].w = min(0.2f, temp[i].w * norm1);
		norm2 += (temp[i].x * temp[i].x + temp[i].y * temp[i].y +
				 temp[i].z * temp[i].z + temp[i].w * temp[i].w);
	}

	norm2 = rsqrt(norm2);
#pragma unroll
	for(int i = 0; i < 32; ++i)
	{
		temp[i].x *= norm2;		temp[i].y *= norm2;
		temp[i].z *= norm2;		temp[i].w *= norm2;
		d_des[sidx + i] = temp[i];
	}
}

void ProgramCU::ComputeDescriptor(CuTexImage*list, CuTexImage* got, CuTexImage* dtex, int rect, int stream)
{
	int num = list->GetImgWidth();
	int width = got->GetImgWidth();
	int height = got->GetImgHeight();

    dtex->InitTexture(num * 128, 1, 1);
	got->BindTexture2D(texDataF2);
	list->BindTexture(texDataF4);
	int block_width = DESCRIPTOR_COMPUTE_BLOCK_SIZE;
	dim3 grid((num * 16 + block_width -1) / block_width);
	dim3 block(block_width);

    if(rect)
    {
	    if(GlobalUtil::_UseDynamicIndexing)
	    	ComputeDescriptorRECT_Kernel<true><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
	    else
	    	ComputeDescriptorRECT_Kernel<false><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);

    }else
    {
	    if(GlobalUtil::_UseDynamicIndexing)
	    	ComputeDescriptor_Kernel<true><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
	    else
	    	ComputeDescriptor_Kernel<false><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
    }
	if(GlobalUtil::_NormalizedSIFT)
	{
		dtex->BindTexture(texDataF4);
		const int block_width = DESCRIPTOR_NORMALIZ_PER_BLOCK;
		dim3 grid((num + block_width -1) / block_width);
		dim3 block(block_width);
		NormalizeDescriptor_Kernel<<<grid, block>>>((float4*) dtex->_cuData, num);
	}
	CheckErrorCUDA("ComputeDescriptor");
}

//////////////////////////////////////////////////////
void ProgramCU::FinishCUDA()
{
	cudaThreadSynchronize();
}

int ProgramCU::CheckErrorCUDA(const char* location)
{
	cudaError_t e = cudaGetLastError();
	if(e)
	{
        if(location) fprintf(stderr, "%s:\t",  location);
		fprintf(stderr, "%s\n",  cudaGetErrorString(e));
		//assert(0);
        return 1;
	}else
    {
        return 0;
    }
}

void __global__ ConvertDOG_Kernel(float* d_result, int width, int height)
{
	int row = (blockIdx.y << BLOCK_LOG_DIM) + threadIdx.y;
	int col = (blockIdx.x << BLOCK_LOG_DIM) + threadIdx.x;
	if(col < width && row < height)
	{
		int index = row * width  + col;
		float v = tex1Dfetch(texData, index);
		d_result[index] = (col == 0 || row == 0 || col == width -1 || row == height -1)?
			0.5 : saturate(0.5+20.0*v);
	}
}
///
void ProgramCU::DisplayConvertDOG(CuTexImage* dog, CuTexImage* out)
{
	if(out->_cuData == NULL) return;
	int width = dog->GetImgWidth(), height = dog ->GetImgHeight();
	dog->BindTexture(texData);
	dim3 grid((width + BLOCK_DIM - 1)/ BLOCK_DIM,  (height + BLOCK_DIM - 1)/BLOCK_DIM);
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	ConvertDOG_Kernel<<<grid, block>>>((float*) out->_cuData, width, height);
	ProgramCU::CheckErrorCUDA("DisplayConvertDOG");
}

void __global__ ConvertGRD_Kernel(float* d_result, int width, int height)
{
	int row = (blockIdx.y << BLOCK_LOG_DIM) + threadIdx.y;
	int col = (blockIdx.x << BLOCK_LOG_DIM) + threadIdx.x;
	if(col < width && row < height)
	{
		int index = row * width  + col;
		float v = tex1Dfetch(texData, index << 1);
		d_result[index] = (col == 0 || row == 0 || col == width -1 || row == height -1)?
				0 : saturate(5 * v);

	}
}


void ProgramCU::DisplayConvertGRD(CuTexImage* got, CuTexImage* out)
{
	if(out->_cuData == NULL) return;
	int width = got->GetImgWidth(), height = got ->GetImgHeight();
	got->BindTexture(texData);
	dim3 grid((width + BLOCK_DIM - 1)/ BLOCK_DIM,  (height + BLOCK_DIM - 1)/BLOCK_DIM);
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	ConvertGRD_Kernel<<<grid, block>>>((float*) out->_cuData, width, height);
	ProgramCU::CheckErrorCUDA("DisplayConvertGRD");
}

void __global__ ConvertKEY_Kernel(float4* d_result, int width, int height)
{

	int row = (blockIdx.y << BLOCK_LOG_DIM) + threadIdx.y;
	int col = (blockIdx.x << BLOCK_LOG_DIM) + threadIdx.x;
	if(col < width && row < height)
	{
		int index = row * width + col;
		float4 keyv = tex1Dfetch(texDataF4, index);
		int is_key = (keyv.x == 1.0f || keyv.x == -1.0f);
		int inside = col > 0 && row > 0 && row < height -1 && col < width - 1;
		float v = inside? saturate(0.5 + 20 * tex1Dfetch(texData, index)) : 0.5;
		d_result[index] = is_key && inside ?
			(keyv.x > 0? make_float4(1.0f, 0, 0, 1.0f) : make_float4(0.0f, 1.0f, 0.0f, 1.0f)):
			make_float4(v, v, v, 1.0f) ;
	}
}
void ProgramCU::DisplayConvertKEY(CuTexImage* key, CuTexImage* dog, CuTexImage* out)
{
	if(out->_cuData == NULL) return;
	int width = key->GetImgWidth(), height = key ->GetImgHeight();
	dog->BindTexture(texData);
	key->BindTexture(texDataF4);
	dim3 grid((width + BLOCK_DIM - 1)/ BLOCK_DIM,  (height + BLOCK_DIM - 1)/BLOCK_DIM);
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	ConvertKEY_Kernel<<<grid, block>>>((float4*) out->_cuData, width, height);
}


void __global__ DisplayKeyPoint_Kernel(float4 * d_result, int num)
{
	int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	if(idx >= num) return;
	float4 v = tex1Dfetch(texDataF4, idx);
	d_result[idx] = make_float4(v.x, v.y, 0, 1.0f);
}

void ProgramCU::DisplayKeyPoint(CuTexImage* ftex, CuTexImage* out)
{
	int num = ftex->GetImgWidth();
	int block_width = 64;
	dim3 grid((num + block_width -1) /block_width);
	dim3 block(block_width);
	ftex->BindTexture(texDataF4);
	DisplayKeyPoint_Kernel<<<grid, block>>>((float4*) out->_cuData, num);
	ProgramCU::CheckErrorCUDA("DisplayKeyPoint");
}

void __global__ DisplayKeyBox_Kernel(float4* d_result, int num)
{
	int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
	if(idx >= num) return;
	int  kidx = idx / 10, vidx = idx - IMUL(kidx , 10);
	float4 v = tex1Dfetch(texDataF4, kidx);
	float sz = fabs(v.z * 3.0f);
	///////////////////////
	float s, c;	__sincosf(v.w, &s, &c);
	///////////////////////
	float dx = vidx == 0? 0 : ((vidx <= 4 || vidx >= 9)? sz : -sz);
	float dy = vidx <= 1? 0 : ((vidx <= 2 || vidx >= 7)? -sz : sz);
	float4 pos;
	pos.x = v.x + c * dx - s * dy;
	pos.y = v.y + c * dy + s * dx;
	pos.z = 0;	pos.w = 1.0f;
	d_result[idx]  = pos;
}

void ProgramCU::DisplayKeyBox(CuTexImage* ftex, CuTexImage* out)
{
	int len = ftex->GetImgWidth();
	int block_width = 32;
	dim3 grid((len * 10 + block_width -1) / block_width);
	dim3 block(block_width);
	ftex->BindTexture(texDataF4);
	DisplayKeyBox_Kernel<<<grid, block>>>((float4*) out->_cuData, len * 10);
}
///////////////////////////////////////////////////////////////////
inline void CuTexImage:: BindTexture(textureReference& texRef)
{
	 cudaBindTexture(NULL, &texRef, _cuData, &texRef.channelDesc, _numBytes);
}

inline void CuTexImage::BindTexture2D(textureReference& texRef)
{
#if defined(SIFTGPU_ENABLE_LINEAR_TEX2D)
	cudaBindTexture2D(0, &texRef, _cuData, &texRef.channelDesc, _imgWidth, _imgHeight, _imgWidth* _numChannel* sizeof(float));
#else
	cudaChannelFormatDesc desc;
	cudaGetChannelDesc(&desc, _cuData2D);
	cudaBindTextureToArray(&texRef, _cuData2D, &desc);
#endif
}

int ProgramCU::CheckCudaDevice(int device)
{
    int count = 0, device_used;
    if(cudaGetDeviceCount(&count) != cudaSuccess  || count <= 0)
    {
        ProgramCU::CheckErrorCUDA("CheckCudaDevice");
        return 0;
    }else if(count == 1)
    {
        cudaDeviceProp deviceProp;
        if ( cudaGetDeviceProperties(&deviceProp, 0) != cudaSuccess  ||
		  (deviceProp.major == 9999 && deviceProp.minor == 9999))
        {
            fprintf(stderr, "CheckCudaDevice: no device supporting CUDA.\n");
            return 0;
        }else
		{
			GlobalUtil::_MemCapGPU = deviceProp.totalGlobalMem / 1024;
			GlobalUtil::_texMaxDimGL = 32768;
			if(GlobalUtil::_verbose)
				fprintf(stdout, "NOTE: changing maximum texture dimension to %d\n", GlobalUtil::_texMaxDimGL);

		}
    }
    if(device >0 && device < count)
    {
        cudaSetDevice(device);
        CheckErrorCUDA("cudaSetDevice\n");
    }
    cudaGetDevice(&device_used);
    if(device != device_used)
        fprintf(stderr,  "\nERROR:   Cannot set device to %d\n"
        "\nWARNING: Use # %d device instead (out of %d)\n", device, device_used, count);
    return 1;
}

////////////////////////////////////////////////////////////////////////////////////////
// siftmatch funtions
//////////////////////////////////////////////////////////////////////////////////////////

#define MULT_TBLOCK_DIMX 128
#define MULT_TBLOCK_DIMY 1
#define MULT_BLOCK_DIMX (MULT_TBLOCK_DIMX)
#define MULT_BLOCK_DIMY (8 * MULT_TBLOCK_DIMY)


texture<uint4, 1, cudaReadModeElementType> texDes1;
texture<uint4, 1, cudaReadModeElementType> texDes2;

void __global__ MultiplyDescriptor_Kernel(int* d_result, int num1, int num2, int3* d_temp)
{
	int idx01 = (blockIdx.y  * MULT_BLOCK_DIMY),  idx02 = (blockIdx.x  * MULT_BLOCK_DIMX);

	int idx1 = idx01 + threadIdx.y, idx2 = idx02 + threadIdx.x;
	__shared__ int data1[17 * 2 * MULT_BLOCK_DIMY];
	int read_idx1 = idx01 * 8 +  threadIdx.x, read_idx2 = idx2 * 8;
	int col4 = threadIdx.x & 0x3, row4 = threadIdx.x >> 2;
	int cache_idx1 = IMUL(row4, 17) + (col4 << 2);

	///////////////////////////////////////////////////////////////
	//Load feature descriptors
	///////////////////////////////////////////////////////////////
#if MULT_BLOCK_DIMY == 16
	uint4 v = tex1Dfetch(texDes1, read_idx1);
	data1[cache_idx1]   = v.x;	data1[cache_idx1+1] = v.y;
	data1[cache_idx1+2] = v.z;	data1[cache_idx1+3] = v.w;
#elif MULT_BLOCK_DIMY == 8
	if(threadIdx.x < 64)
	{
		uint4 v = tex1Dfetch(texDes1, read_idx1);
		data1[cache_idx1]   = v.x;		data1[cache_idx1+1] = v.y;
		data1[cache_idx1+2] = v.z;		data1[cache_idx1+3] = v.w;
	}
#else
#error
#endif
	__syncthreads();

	///
	if(idx2 >= num2) return;
	///////////////////////////////////////////////////////////////////////////
	//compare descriptors

	int results[MULT_BLOCK_DIMY];
#pragma unroll
	for(int i = 0; i < MULT_BLOCK_DIMY; ++i) results[i] = 0;

#pragma unroll
	for(int i = 0; i < 8; ++i)
	{
		uint4 v = tex1Dfetch(texDes2, read_idx2 + i);
		unsigned char* p2 = (unsigned char*)(&v);
#pragma unroll
		for(int k = 0; k < MULT_BLOCK_DIMY; ++k)
		{
			unsigned char* p1 = (unsigned char*) (data1 + k * 34 + i *  4 + (i/4));
			results[k] += 	 ( IMUL(p1[0], p2[0])	+ IMUL(p1[1], p2[1])
							 + IMUL(p1[2], p2[2])  	+ IMUL(p1[3], p2[3])
							 + IMUL(p1[4], p2[4])  	+ IMUL(p1[5], p2[5])
							 + IMUL(p1[6], p2[6])  	+ IMUL(p1[7], p2[7])
							 + IMUL(p1[8], p2[8])  	+ IMUL(p1[9], p2[9])
							 + IMUL(p1[10], p2[10])	+ IMUL(p1[11], p2[11])
							 + IMUL(p1[12], p2[12])	+ IMUL(p1[13], p2[13])
							 + IMUL(p1[14], p2[14])	+ IMUL(p1[15], p2[15]));
		}
	}

	int dst_idx = IMUL(idx1, num2)  + idx2;
	if(d_temp)
	{
		int3 cmp_result = make_int3(0, -1, 0);

#pragma unroll
		for(int i = 0; i < MULT_BLOCK_DIMY; ++i)
		{
			if(idx1 + i < num1)
			{
				cmp_result = results[i] > cmp_result.x?
				make_int3(results[i], idx1 + i, cmp_result.x) :
				make_int3(cmp_result.x, cmp_result.y, max(cmp_result.z, results[i]));
				d_result[dst_idx + IMUL(i, num2)] = results[i];
			}
		}
		d_temp[ IMUL(blockIdx.y, num2) + idx2] = cmp_result;
	}else
	{
#pragma unroll
		for(int i = 0; i < MULT_BLOCK_DIMY; ++i)
		{
			if(idx1 + i < num1) d_result[dst_idx + IMUL(i, num2)] = results[i];
		}
	}

}


void ProgramCU::MultiplyDescriptor(CuTexImage* des1, CuTexImage* des2, CuTexImage* texDot, CuTexImage* texCRT)
{
	int num1 = des1->GetImgWidth() / 8;
	int num2 = des2->GetImgWidth() / 8;
	dim3 grid(	(num2 + MULT_BLOCK_DIMX - 1)/ MULT_BLOCK_DIMX,
		(num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY);
	dim3 block(MULT_TBLOCK_DIMX, MULT_TBLOCK_DIMY);
	texDot->InitTexture( num2,num1);
	if(texCRT) texCRT->InitTexture(num2, (num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY, 32);
	des1->BindTexture(texDes1);
	des2->BindTexture(texDes2);

	MultiplyDescriptor_Kernel<<<grid, block>>>((int*)texDot->_cuData, num1, num2,
												(texCRT? (int3*)texCRT->_cuData : NULL));
}

texture<float, 1, cudaReadModeElementType> texLoc1;
texture<float2, 1, cudaReadModeElementType> texLoc2;
struct Matrix33{float mat[3][3];};



void __global__ MultiplyDescriptorG_Kernel(int* d_result, int num1, int num2, int3* d_temp,
										   Matrix33 H, float hdistmax, Matrix33 F, float fdistmax)
{
	int idx01 = (blockIdx.y  * MULT_BLOCK_DIMY);
	int idx02 = (blockIdx.x  * MULT_BLOCK_DIMX);

	int idx1 = idx01 + threadIdx.y;
	int idx2 = idx02 + threadIdx.x;
	__shared__ int data1[17 * 2 * MULT_BLOCK_DIMY];
	__shared__ float loc1[MULT_BLOCK_DIMY * 2];
	int read_idx1 = idx01 * 8 +  threadIdx.x ;
	int read_idx2 = idx2 * 8;
	int col4 = threadIdx.x & 0x3, row4 = threadIdx.x >> 2;
	int cache_idx1 = IMUL(row4, 17) + (col4 << 2);
#if MULT_BLOCK_DIMY == 16
	uint4 v = tex1Dfetch(texDes1, read_idx1);
	data1[cache_idx1]   = v.x;
	data1[cache_idx1+1] = v.y;
	data1[cache_idx1+2] = v.z;
	data1[cache_idx1+3] = v.w;
#elif MULT_BLOCK_DIMY == 8
	if(threadIdx.x < 64)
	{
		uint4 v = tex1Dfetch(texDes1, read_idx1);
		data1[cache_idx1]   = v.x;
		data1[cache_idx1+1] = v.y;
		data1[cache_idx1+2] = v.z;
		data1[cache_idx1+3] = v.w;
	}
#else
#error
#endif
	__syncthreads();
	if(threadIdx.x < MULT_BLOCK_DIMY * 2)
	{
		loc1[threadIdx.x] = tex1Dfetch(texLoc1, 2 * idx01 + threadIdx.x);
	}
	__syncthreads();
	if(idx2 >= num2) return;
	int results[MULT_BLOCK_DIMY];
	/////////////////////////////////////////////////////////////////////////////////////////////
	//geometric verification
	/////////////////////////////////////////////////////////////////////////////////////////////
	int good_count = 0;
	float2 loc2 = tex1Dfetch(texLoc2, idx2);
#pragma unroll
	for(int i = 0; i < MULT_BLOCK_DIMY; ++i)
	{

		if(idx1 + i < num1)
		{
			float* loci = loc1 + i * 2;
			float locx = loci[0], locy = loci[1];
			//homography
			float x[3], diff[2];
			x[0] = H.mat[0][0] * locx + H.mat[0][1] * locy + H.mat[0][2];
			x[1] = H.mat[1][0] * locx + H.mat[1][1] * locy + H.mat[1][2];
			x[2] = H.mat[2][0] * locx + H.mat[2][1] * locy + H.mat[2][2];
			diff[0] = FDIV(x[0], x[2]) - loc2.x;
			diff[1] = FDIV(x[1], x[2]) - loc2.y;
      float hdist = diff[0] * diff[0] + diff[1] * diff[1];
			if(hdist < hdistmax)
			{
				//check fundamental matrix
				float fx1[3], ftx2[3], x2fx1, se;
				fx1[0] = F.mat[0][0] * locx + F.mat[0][1] * locy + F.mat[0][2];
				fx1[1] = F.mat[1][0] * locx + F.mat[1][1] * locy + F.mat[1][2];
				fx1[2] = F.mat[2][0] * locx + F.mat[2][1] * locy + F.mat[2][2];

				ftx2[0] = F.mat[0][0] * loc2.x + F.mat[1][0] * loc2.y + F.mat[2][0];
				ftx2[1] = F.mat[0][1] * loc2.x + F.mat[1][1] * loc2.y + F.mat[2][1];
				//ftx2[2] = F.mat[0][2] * loc2.x + F.mat[1][2] * loc2.y + F.mat[2][2];

				x2fx1 = loc2.x * fx1[0]  + loc2.y * fx1[1] + fx1[2];
				se = FDIV(x2fx1 * x2fx1, fx1[0] * fx1[0] + fx1[1] * fx1[1] + ftx2[0] * ftx2[0] + ftx2[1] * ftx2[1]);
				results[i] = se < fdistmax? 0: -262144;
			}else
			{
				results[i] = -262144;
			}
		}else
		{
			results[i] = -262144;
		}
		good_count += (results[i] >=0);
	}
	/////////////////////////////////////////////////////////////////////////////////////////////
	///compare feature descriptors anyway
	/////////////////////////////////////////////////////////////////////////////////////////////
	if(good_count > 0)
	{
#pragma unroll
		for(int i = 0; i < 8; ++i)
		{
			uint4 v = tex1Dfetch(texDes2, read_idx2 + i);
			unsigned char* p2 = (unsigned char*)(&v);
#pragma unroll
			for(int k = 0; k < MULT_BLOCK_DIMY; ++k)
			{
				unsigned char* p1 = (unsigned char*) (data1 + k * 34 + i *  4 + (i/4));
				results[k] += 	 ( IMUL(p1[0], p2[0])	+ IMUL(p1[1], p2[1])
								 + IMUL(p1[2], p2[2])  	+ IMUL(p1[3], p2[3])
								 + IMUL(p1[4], p2[4])  	+ IMUL(p1[5], p2[5])
								 + IMUL(p1[6], p2[6])  	+ IMUL(p1[7], p2[7])
								 + IMUL(p1[8], p2[8])  	+ IMUL(p1[9], p2[9])
								 + IMUL(p1[10], p2[10])	+ IMUL(p1[11], p2[11])
								 + IMUL(p1[12], p2[12])	+ IMUL(p1[13], p2[13])
								 + IMUL(p1[14], p2[14])	+ IMUL(p1[15], p2[15]));
			}
		}
	}
	int dst_idx = IMUL(idx1, num2)  + idx2;
	if(d_temp)
	{
		int3 cmp_result = make_int3(0, -1, 0);
#pragma unroll
		for(int i= 0; i < MULT_BLOCK_DIMY; ++i)
		{
			if(idx1 + i < num1)
			{
				cmp_result = results[i] > cmp_result.x?
				make_int3(results[i], idx1 + i, cmp_result.x) :
				make_int3(cmp_result.x, cmp_result.y, max(cmp_result.z, results[i]));
				d_result[dst_idx + IMUL(i, num2)] = max(results[i], 0);
			}else
			{
				break;
			}
		}
		d_temp[ IMUL(blockIdx.y, num2) + idx2] = cmp_result;
	}else
	{
#pragma unroll
		for(int i = 0; i < MULT_BLOCK_DIMY; ++i)
		{
			if(idx1 + i < num1) d_result[dst_idx + IMUL(i, num2)] = max(results[i], 0);
			else break;
		}
	}

}


void ProgramCU::MultiplyDescriptorG(CuTexImage* des1, CuTexImage* des2,
		CuTexImage* loc1, CuTexImage* loc2, CuTexImage* texDot, CuTexImage* texCRT,
		float* H, float hdistmax, float* F, float fdistmax)
{
	int num1 = des1->GetImgWidth() / 8;
	int num2 = des2->GetImgWidth() / 8;
	Matrix33 MatF, MatH;
	//copy the matrix
	memcpy(MatF.mat, F, 9 * sizeof(float));
	memcpy(MatH.mat, H, 9 * sizeof(float));
	//thread blocks
	dim3 grid(	(num2 + MULT_BLOCK_DIMX - 1)/ MULT_BLOCK_DIMX,
		(num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY);
	dim3 block(MULT_TBLOCK_DIMX, MULT_TBLOCK_DIMY);
	//intermediate results
	texDot->InitTexture( num2,num1);
	if(texCRT) texCRT->InitTexture( num2, (num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY, 3);
	loc1->BindTexture(texLoc1);
	loc2->BindTexture(texLoc2);
	des1->BindTexture(texDes1);
	des2->BindTexture(texDes2);
	MultiplyDescriptorG_Kernel<<<grid, block>>>((int*)texDot->_cuData, num1, num2,
												(texCRT? (int3*)texCRT->_cuData : NULL),
												MatH, hdistmax, MatF, fdistmax);
}


texture<int,  1, cudaReadModeElementType> texDOT;

#define ROWMATCH_BLOCK_WIDTH 32
#define ROWMATCH_BLOCK_HEIGHT 1

void __global__  RowMatch_Kernel(int*d_dot, int* d_result, int num2, float distmax, float ratiomax)
{
#if ROWMATCH_BLOCK_HEIGHT == 1
	__shared__ int dotmax[ROWMATCH_BLOCK_WIDTH];
	__shared__ int dotnxt[ROWMATCH_BLOCK_WIDTH];
	__shared__ int dotidx[ROWMATCH_BLOCK_WIDTH];
	int	row = blockIdx.y;
#else
	__shared__ int x_dotmax[ROWMATCH_BLOCK_HEIGHT][ROWMATCH_BLOCK_WIDTH];
	__shared__ int x_dotnxt[ROWMATCH_BLOCK_HEIGHT][ROWMATCH_BLOCK_WIDTH];
	__shared__ int x_dotidx[ROWMATCH_BLOCK_HEIGHT][ROWMATCH_BLOCK_WIDTH];
	int*	dotmax = x_dotmax[threadIdx.y];
	int*	dotnxt = x_dotnxt[threadIdx.y];
	int*	dotidx = x_dotidx[threadIdx.y];
	int row = IMUL(blockIdx.y, ROWMATCH_BLOCK_HEIGHT) + threadIdx.y;
#endif

	int base_address = IMUL(row , num2);
	int t_dotmax = 0, t_dotnxt = 0, t_dotidx = -1;
	for(int i = 0; i < num2; i += ROWMATCH_BLOCK_WIDTH)
	{
		if(threadIdx.x + i < num2)
		{
			int v = d_dot[base_address + threadIdx.x + i];  // tex1Dfetch(texDOT, base_address + threadIdx.x + i);
			bool test = v > t_dotmax;
			t_dotnxt = test? t_dotmax : max(t_dotnxt, v);
			t_dotidx = test? (threadIdx.x + i) : t_dotidx;
			t_dotmax = test? v: t_dotmax;
		}
		__syncthreads();
	}
	dotmax[threadIdx.x] = t_dotmax;
	dotnxt[threadIdx.x] = t_dotnxt;
	dotidx[threadIdx.x] = t_dotidx;
	__syncthreads();

#pragma unroll
	for(int step = ROWMATCH_BLOCK_WIDTH/2; step >0; step /= 2)
	{
		if(threadIdx.x < step)
		{
			int v1 = dotmax[threadIdx.x], v2 = dotmax[threadIdx.x + step];
			bool test =  v2 > v1;
			dotnxt[threadIdx.x] = test? max(v1, dotnxt[threadIdx.x + step]) :max(dotnxt[threadIdx.x], v2);
			dotidx[threadIdx.x] = test? dotidx[threadIdx.x + step] : dotidx[threadIdx.x];
			dotmax[threadIdx.x] = test? v2 : v1;
		}
		__syncthreads();
	}
	if(threadIdx.x == 0)
	{
		float dist =  acos(min(dotmax[0] * 0.000003814697265625f, 1.0));
		float distn = acos(min(dotnxt[0] * 0.000003814697265625f, 1.0));
		//float ratio = dist / distn;
		d_result[row] = (dist < distmax) && (dist < distn * ratiomax) ? dotidx[0] : -1;//?  : -1;
	}

}


void ProgramCU::GetRowMatch(CuTexImage* texDot, CuTexImage* texMatch, float distmax, float ratiomax)
{
	int num1 = texDot->GetImgHeight();
	int num2 = texDot->GetImgWidth();
	dim3 grid(1, num1/ROWMATCH_BLOCK_HEIGHT);
	dim3 block(ROWMATCH_BLOCK_WIDTH, ROWMATCH_BLOCK_HEIGHT);
	// texDot->BindTexture(texDOT);
	RowMatch_Kernel<<<grid, block>>>((int*)texDot->_cuData,
		(int*)texMatch->_cuData, num2, distmax, ratiomax);
}

#define COLMATCH_BLOCK_WIDTH 32

//texture<int3,  1, cudaReadModeElementType> texCT;

void __global__  ColMatch_Kernel(int3*d_crt, int* d_result, int height, int num2, float distmax, float ratiomax)
{
	int col = COLMATCH_BLOCK_WIDTH * blockIdx.x + threadIdx.x;
	if(col >= num2) return;
	int3 result = d_crt[col];//tex1Dfetch(texCT, col);
	int read_idx = col + num2;
	for(int i = 1; i < height; ++i, read_idx += num2)
	{
		int3 temp = d_crt[read_idx];//tex1Dfetch(texCT, read_idx);
		result = result.x < temp.x?
			make_int3(temp.x, temp.y, max(result.x, temp.z)) :
			make_int3(result.x, result.y, max(result.z, temp.x));
	}

	float dist =  acos(min(result.x * 0.000003814697265625f, 1.0));
	float distn = acos(min(result.z * 0.000003814697265625f, 1.0));
		//float ratio = dist / distn;
	d_result[col] = (dist < distmax) && (dist < distn * ratiomax) ? result.y : -1;//?  : -1;

}

void ProgramCU::GetColMatch(CuTexImage* texCRT, CuTexImage* texMatch, float distmax, float ratiomax)
{
	int height = texCRT->GetImgHeight();
	int num2 = texCRT->GetImgWidth();
	//texCRT->BindTexture(texCT);
    dim3 grid((num2 + COLMATCH_BLOCK_WIDTH -1) / COLMATCH_BLOCK_WIDTH);
    dim3 block(COLMATCH_BLOCK_WIDTH);
	ColMatch_Kernel<<<grid, block>>>((int3*)texCRT->_cuData, (int*) texMatch->_cuData, height, num2, distmax, ratiomax);
}

#endif
